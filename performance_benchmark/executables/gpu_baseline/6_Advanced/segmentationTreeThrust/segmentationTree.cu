#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This application demonstrates an approach to the image segmentation
 * trees construction. It is based on Boruvka's MST algorithm.
 * Here's the complete list of references:
 * 1) V. Vineet et al, "Fast Minimum Spanning Tree for
 *    Large Graphs on the GPU";
 * 2) P. Felzenszwalb et al, "Efficient Graph-Based Image Segmentation";
 * 3) A. Ion et al, "Considerations Regarding the Minimum Spanning
 *    Tree Pyramid Segmentation Method".
 */

// System includes.
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <chrono>  
#include <unistd.h>

// STL includes.
#include <iostream>
#include <fstream>
#include <iterator>
#include <vector>
#include <list>
#include <deque>
#include <algorithm>

// Thrust library includes.
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/for_each.h>
#include <thrust/reduce.h>
#include <thrust/unique.h>
#include <thrust/scan.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/fill.h>
#include <thrust/sort.h>
#include <thrust/adjacent_difference.h>
#include <thrust/find.h>

#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

// Sample framework includes.
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// Project includes.
#include "common.cuh"

// Kernels.
#include "kernels.cuh"

using std::cin;
using std::cout;
using std::cerr;
using std::endl;
using std::vector;
using std::list;
using std::deque;

vector<uchar3> in_image;
std::string in_imageName;
std::string outputPath;
int w;
int b;
bool partial = false;
bool no_output = false;


// Very simple von Neumann middle-square prng.  rand() is different across
// various OS platforms, which makes testing and the output inconsistent.
int myrand(void)
{
    static int seed = 72191;
    char sq[22];

    seed *= seed;
    sprintf(sq, "%010d", seed);
    // pull the middle 5 digits out of sq
    sq[8] = 0;
    seed = atoi(&sq[3]);

    return seed;
}

// Simple memory pool class. It is nothing more than array of fixed-sized
// arrays.
template <typename T>
class DeviceMemoryPool
{
    public:
        // The parameters of the constructor are as follows:
        // 1) uint chunkSize --- size of the particular array;
        // 2) uint chunksCount --- number of fixed-sized arrays.
        DeviceMemoryPool(uint chunkSize, uint chunksCount) :
            chunkSize_(chunkSize)
        {
            chunkRawSize_ = (chunkSize * sizeof(T) + 511) & ~511;

            try
            {
                basePtr_ =
                    thrust::device_malloc(chunkRawSize_ * chunksCount);
            }
            catch (thrust::system_error &e)
            {
                cerr << "Pool memory allocation failed (" << e.what() << ")"
                     << endl;
                exit(EXIT_FAILURE);
            }

            for (uint chunkIndex = 0; chunkIndex < chunksCount; ++chunkIndex)
            {
                chunks_.push_back(
                    thrust::device_ptr<T>(
                        reinterpret_cast<T *>(
                            static_cast<char *>(basePtr_.get()) +
                            chunkRawSize_ * chunkIndex)));
            }
        }

        ~DeviceMemoryPool()
        {
            try
            {
                thrust::device_free(basePtr_);
            }
            catch (thrust::system_error &e)
            {
                cerr << "Pool memory allocation failed (" << e.what() << ")"
                     << endl;
                exit(EXIT_FAILURE);
            }
        }

        // Returns an address of the first available array
        // in the memory pool.
        thrust::device_ptr<T> get()
        {
            thrust::device_ptr<T> ptr(chunks_.back());
            chunks_.pop_back();

            return ptr;
        }

        // Pushes an address stored in "ptr" to the list
        // of available arrays of the memory pool.
        // It should be noted that it is user who is responsible for returning
        // the previously requested memory to the appropriate pool.
        inline void put(const thrust::device_ptr<T> &ptr)
        {
            chunks_.push_back(ptr);
        }

        uint totalFreeChunks() const
        {
            return chunks_.size();
        }

    private:
        uint chunkSize_, chunkRawSize_;
        thrust::device_ptr<void> basePtr_;

        list< thrust::device_ptr<T> > chunks_;
};

// Graph structure.
struct Graph
{
    Graph() {}

    Graph(uint verticesCount, uint edgesCount) :
        vertices(verticesCount),
        edges(edgesCount),
        weights(edgesCount)
    {}

    // This vector stores offsets for each vertex in "edges" and "weights"
    // vectors. For example:
    // "vertices[0]" is an index of the first outgoing edge of vertex #0,
    // "vertices[1]" is an index of the first outgoing edge of vertex #1, etc.
    vector<uint> vertices;

    // This vector stores indices of endpoints of the corresponding edges.
    // For example, "edges[vertices[0]]" is the first neighbouring vertex
    // of vertex #0.
    vector<uint> edges;

    // This vector stores weights of the corresponding edges.
    vector<float> weights;
};

// Simple segmentation tree class.
// Each level of the tree corresponds to the segmentation.
// See "Level" class for the details.
class Pyramid
{
    public:
        void addLevel(uint totalSuperNodes,
                      uint totalNodes,
                      thrust::device_ptr<uint> superVerticesOffsets,
                      thrust::device_ptr<uint> verticesIDs)
        {
            levels_.push_back(Level(totalSuperNodes, totalNodes));
            levels_.back().buildFromDeviceData(superVerticesOffsets,
                                               verticesIDs);
        }

        uint levelsCount() const
        {
            return static_cast<uint>(levels_.size());
        }

        void dump(uint width, uint height, bool no_output_local) const
        {
            uint levelIndex = 0;

            uint requiredDigitsCount =
                static_cast<uint>(log10(static_cast<float>(levelsCount()))) +
                1;

            std::string outFile = outputPath;
            size_t lastindex = outFile.find_last_of("."); 
            std::string rawOutName = outFile.substr(0, lastindex);

            
            for (LevelsIterator level = levels_.rbegin();
                 level != levels_.rend();
                 ++level, ++levelIndex)
            {
                std::string outfilename = rawOutName + std::string("_")  + std::to_string(levelIndex) + std::string(".ppm");

                dumpLevel(level, width, height, outfilename.c_str(), no_output_local);
            }
        }

    private:
        // Level of the segmentation tree.
        class Level
        {
            public:
                Level(uint totalSuperNodes, uint totalNodes) :
                    superNodesOffsets_(totalSuperNodes), nodes_(totalNodes)
                {
                }

                void buildFromDeviceData(
                    thrust::device_ptr<uint> superVerticesOffsets,
                    thrust::device_ptr<uint> verticesIDs)
                {
                    checkCudaErrors(
                        hipMemcpy(&(superNodesOffsets_[0]),
                                   superVerticesOffsets.get(),
                                   sizeof(uint) * superNodesOffsets_.size(),
                                   hipMemcpyDeviceToHost));

                    checkCudaErrors(
                        hipMemcpy(&(nodes_[0]),
                                   verticesIDs.get(),
                                   sizeof(uint) * nodes_.size(),
                                   hipMemcpyDeviceToHost));
                }

            private:
                friend class Pyramid;

                // The pair of the following vectors describes the
                // relation between the consecutive levels.
                // Consider an example. Let the index of the current level be n.
                // Then nodes of level #(n-1) with indices stored in
                // "nodes[superNodesOffsets_[0]]",
                // "nodes[superNodesOffsets_[0] + 1]",
                // ...,
                // "nodes[superNodesOffsets_[1] - 1]"
                // correspond to vertex #0 of level #n. An so on.
                vector<uint> superNodesOffsets_;
                vector<uint> nodes_;
        };

        typedef list<Level>::const_reverse_iterator LevelsIterator;

        // Dumps level to the file "level_n.ppm" where n
        // is index of the level. Segments are drawn in random colors.
        void dumpLevel(LevelsIterator level,
                       uint width,
                       uint height,
                       const char *filename, bool no_output_local) const
        {
            deque< std::pair<uint, uint> > nodesQueue;

            uint totalSegments;

            {
                const vector<uint> &superNodesOffsets =
                    level->superNodesOffsets_;
                const vector<uint> &nodes =
                    level->nodes_;

                totalSegments = static_cast<uint>(superNodesOffsets.size());

                for (uint superNodeIndex = 0, nodeIndex = 0;
                     superNodeIndex < superNodesOffsets.size();
                     ++superNodeIndex)
                {

                    uint superNodeEnd =
                        superNodeIndex + 1 < superNodesOffsets.size() ?
                        superNodesOffsets[superNodeIndex + 1] :
                        static_cast<uint>(nodes.size());

                    for (; nodeIndex < superNodeEnd; ++nodeIndex)
                    {
                        nodesQueue.push_back(std::make_pair(nodes[nodeIndex],
                                                            superNodeIndex));
                    }
                }
            }

            ++level;

            while (level != levels_.rend())
            {
                uint superNodesCount = static_cast<uint>(nodesQueue.size());

                const vector<uint> &superNodesOffsets =
                    level->superNodesOffsets_;
                const vector<uint> &nodes =
                    level->nodes_;

                while (superNodesCount--)
                {
                    std::pair<uint, uint> currentNode = nodesQueue.front();
                    nodesQueue.pop_front();

                    uint superNodeBegin = superNodesOffsets[currentNode.first];

                    uint superNodeEnd =
                        currentNode.first + 1 < superNodesOffsets.size() ?
                        superNodesOffsets[currentNode.first + 1] :
                        static_cast<uint>(nodes.size());

                    for (uint nodeIndex = superNodeBegin;
                         nodeIndex < superNodeEnd;
                         ++nodeIndex)
                    {

                        nodesQueue.push_back(
                            std::make_pair(nodes[nodeIndex],
                                           currentNode.second));
                    }
                }

                ++level;
            }

            vector<uint> colors(3 * totalSegments);

            for (uint colorIndex = 0; colorIndex < totalSegments; ++colorIndex)
            {
                colors[colorIndex * 3    ] = myrand() % 256;
                colors[colorIndex * 3 + 1] = myrand() % 256;
                colors[colorIndex * 3 + 2] = myrand() % 256;
            }

            uchar *image = new uchar[width * height * 3];

            while (!nodesQueue.empty())
            {
                std::pair<uint, uint> currentNode = nodesQueue.front();
                nodesQueue.pop_front();

                uint pixelIndex = currentNode.first;
                uint pixelSegment = currentNode.second;

                image[pixelIndex * 3    ] = colors[pixelSegment * 3    ];
                image[pixelIndex * 3 + 1] = colors[pixelSegment * 3 + 1];
                image[pixelIndex * 3 + 2] = colors[pixelSegment * 3 + 2];
            }

            if (!no_output_local) {
                __savePPM(filename, image, width, height, 3);
            }

            delete[] image;
        }

        list<Level> levels_;
};

// The class that encapsulates the main algorithm.
class SegmentationTreeBuilder
{
    public:
        SegmentationTreeBuilder():verticesCount_(0),edgesCount_(0)  {}

        ~SegmentationTreeBuilder() {}

        // Repeatedly invokes the step of the algorithm
        // until the limiting segmentation is found.
        // Returns time (in ms) spent on building the tree.
        float run(const Graph &graph, Pyramid &segmentations)
        {
            hipEvent_t start, stop;

            hipEventCreate(&start);
            hipEventCreate(&stop);

            hipEventRecord(start, 0);

            // Allocate required memory pools. We need just 4 types of arrays.
            MemoryPoolsCollection pools =
            {
                DeviceMemoryPool<uint>(
                    static_cast<uint>(graph.vertices.size()),
                    kUintVerticesPoolsRequired),
                DeviceMemoryPool<float>(
                    static_cast<uint>(graph.vertices.size()),
                    kFloatVerticesPoolsRequired),
                DeviceMemoryPool<uint>(
                    static_cast<uint>(graph.edges.size()),
                    kUintEdgesPoolsRequired),
                DeviceMemoryPool<float>(
                    static_cast<uint>(graph.edges.size()),
                    kFloatEdgesPoolsRequired)
            };

            // Initialize internal variables
            try
            {
                initalizeData(graph, pools);
            }
            catch (thrust::system_error &e)
            {
                cerr << "Initialization failed (" << e.what() << ")" << endl;
                exit(EXIT_FAILURE);
            }

            // Run steps
            AlgorithmStatus status;

            try
            {
                do
                {
                    status = invokeStep(pools, segmentations);
                }
                while (status != ALGORITHM_FINISHED);
            }
            catch (thrust::system_error &e)
            {
                cerr << "Algorithm failed (" << e.what() << ")" << endl;
                exit(EXIT_FAILURE);
            }

            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);

            float elapsedTime;
            hipEventElapsedTime(&elapsedTime, start, stop);

            return elapsedTime;
        }

    private:
        struct MemoryPoolsCollection
        {
            DeviceMemoryPool<uint> uintVertices;
            DeviceMemoryPool<float> floatVertices;
            DeviceMemoryPool<uint> uintEdges;
            DeviceMemoryPool<float> floatEdges;
        };

        static const uint kUintVerticesPoolsRequired = 8;
        static const uint kFloatVerticesPoolsRequired = 3;
        static const uint kUintEdgesPoolsRequired = 8;
        static const uint kFloatEdgesPoolsRequired = 4;

        void initalizeData(const Graph &graph, MemoryPoolsCollection &pools)
        {
            // Get memory for the internal variables
            verticesCount_ = static_cast<uint>(graph.vertices.size());
            edgesCount_ = static_cast<uint>(graph.edges.size());

            dVertices_ = pools.uintVertices.get();
            dEdges_ = pools.uintEdges.get();
            dWeights_ = pools.floatEdges.get();

            dOutputEdgesFlags_ = pools.uintEdges.get();

            // Copy graph to the device memory
            checkCudaErrors(hipMemcpy(dVertices_.get(),
                                       &(graph.vertices[0]),
                                       sizeof(uint) * verticesCount_,
                                       hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(dEdges_.get(),
                                       &(graph.edges[0]),
                                       sizeof(uint) * edgesCount_,
                                       hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(dWeights_.get(),
                                       &(graph.weights[0]),
                                       sizeof(float) * edgesCount_,
                                       hipMemcpyHostToDevice));


            thrust::fill(dOutputEdgesFlags_,
                         dOutputEdgesFlags_ + edgesCount_,
                         0);
        }

        static const uint kMaxThreadsPerBlock = 256;

        // Calculates grid parameters of the consecutive kernel calls
        // based on the number of elements in the array.
        void calculateThreadsDistribution(uint totalElements,
                                          uint &blocksCount,
                                          uint &threadsPerBlockCount)
        {
            if (totalElements > kMaxThreadsPerBlock)
            {
                blocksCount =
                    (totalElements + kMaxThreadsPerBlock - 1) /
                    kMaxThreadsPerBlock;

                threadsPerBlockCount = kMaxThreadsPerBlock;
            }
            else
            {
                blocksCount = 1;
                threadsPerBlockCount = totalElements;
            }
        }

        enum AlgorithmStatus { ALGORITHM_NOT_FINISHED, ALGORITHM_FINISHED };

        AlgorithmStatus invokeStep(MemoryPoolsCollection &pools,
                                   Pyramid &segmentations)
        {
            uint blocksCount, threadsPerBlockCount;

            calculateThreadsDistribution(edgesCount_,
                                         blocksCount,
                                         threadsPerBlockCount);
            dim3 gridDimsForEdges(blocksCount, 1, 1);
            dim3 blockDimsForEdges(threadsPerBlockCount, 1, 1);

            calculateThreadsDistribution(verticesCount_,
                                         blocksCount,
                                         threadsPerBlockCount);
            dim3 gridDimsForVertices(blocksCount, 1, 1);
            dim3 blockDimsForVertices(threadsPerBlockCount, 1, 1);

            thrust::device_ptr<uint> dEdgesFlags = pools.uintEdges.get();

            thrust::fill(dEdgesFlags, dEdgesFlags + edgesCount_, 0);

            // Mark the first edge for each vertex in "dEdgesFlags"
            markSegments<<< gridDimsForVertices, blockDimsForVertices, 0 >>>
            (dVertices_.get(), dEdgesFlags.get(), verticesCount_);
            getLastCudaError("markSegments launch failed.");

            // Now find minimum edges for each vertex.
            thrust::device_ptr<uint> dMinScannedEdges =
                pools.uintEdges.get();
            thrust::device_ptr<float> dMinScannedWeights =
                pools.floatEdges.get();

            thrust::inclusive_scan_by_key(
                dEdgesFlags,
                dEdgesFlags + edgesCount_,
                thrust::make_zip_iterator(
                    thrust::make_tuple(dWeights_, dEdges_)),
                thrust::make_zip_iterator(
                    thrust::make_tuple(dMinScannedWeights, dMinScannedEdges)),
                thrust::greater_equal<uint>(),
                thrust::minimum< thrust::tuple<float, uint> >());

            // To make things clear.
            // Let "dEdgesFlags" denote groups of edges that
            // correspond to the same vertices. Then the last edge of each group
            // (in "dMinScannedEdges" and "dMinScannedWeights") is now minimal.

            // Calculate a successor vertex for each vertex. A successor of the
            // vertex v is a neighbouring vertex connected to v
            // by the minimal edge.
            thrust::device_ptr<uint> dSuccessors = pools.uintVertices.get();

            getSuccessors<<< gridDimsForVertices, blockDimsForVertices, 0 >>>
            (dVertices_.get(),
             dMinScannedEdges.get(),
             dSuccessors.get(),
             verticesCount_,
             edgesCount_);
            getLastCudaError("getSuccessors launch failed.");

            pools.uintEdges.put(dMinScannedEdges);
            pools.floatEdges.put(dMinScannedWeights);

            // Remove cyclic successor dependencies. Note that there can be only
            // two vertices in a cycle. See [1] for details.
            removeCycles<<< gridDimsForVertices, blockDimsForVertices, 0 >>>
            (dSuccessors.get(), verticesCount_);
            getLastCudaError("removeCycles launch failed.");

            // Build up an array of startpoints for edges. As already stated,
            // each group of edges denoted by "dEdgesFlags"
            // has the same startpoint.
            thrust::device_ptr<uint> dStartpoints = pools.uintEdges.get();

            thrust::inclusive_scan(dEdgesFlags,
                                   dEdgesFlags + edgesCount_,
                                   dStartpoints);

            addScalar<<< gridDimsForEdges, blockDimsForEdges, 0 >>>
            (dStartpoints.get(), -1, edgesCount_);
            getLastCudaError("addScalar launch failed.");

            // Shrink the chains of successors. New successors will eventually
            // represent superpixels of the new level.
            thrust::device_ptr<uint> dRepresentatives =
                pools.uintVertices.get();

            getRepresentatives
            <<< gridDimsForVertices, blockDimsForVertices, 0 >>>
            (dSuccessors.get(),
             dRepresentatives.get(),
             verticesCount_);
            getLastCudaError("getRepresentatives launch failed.");

            swap(dSuccessors, dRepresentatives);

            pools.uintVertices.put(dRepresentatives);

            // Group vertices by successors' indices.
            thrust::device_ptr<uint> dClusteredVerticesIDs =
                pools.uintVertices.get();

            thrust::sequence(dClusteredVerticesIDs,
                             dClusteredVerticesIDs + verticesCount_);

            thrust::sort(
                thrust::make_zip_iterator(
                    thrust::make_tuple(
                        thrust::device_ptr<uint> (dSuccessors),
                        thrust::device_ptr<uint> (dClusteredVerticesIDs))),
                thrust::make_zip_iterator(
                    thrust::make_tuple(
                        thrust::device_ptr<uint> (dSuccessors +
                                                  verticesCount_),
                        thrust::device_ptr<uint> (dClusteredVerticesIDs +
                                                  verticesCount_))));

            // Mark those groups.
            thrust::device_ptr<uint> dVerticesFlags_ = pools.uintVertices.get();

            thrust::fill(dVerticesFlags_, dVerticesFlags_ + verticesCount_, 0);

            thrust::adjacent_difference(dSuccessors,
                                        dSuccessors + verticesCount_,
                                        dVerticesFlags_,
                                        thrust::not_equal_to<uint>());

            hipMemset((void *) dVerticesFlags_.get(), 0, sizeof(uint));

            // Assign new indices to the successors (the indices of vertices
            // at the new level).
            thrust::device_ptr<uint> dNewVerticesIDs_ =
                pools.uintVertices.get();

            thrust::inclusive_scan(dVerticesFlags_,
                                   dVerticesFlags_ + verticesCount_,
                                   dNewVerticesIDs_);

            pools.uintVertices.put(dVerticesFlags_);

            // Now we can calculate number of resulting superpixels easily.
            uint newVerticesCount;
            hipMemcpy(&newVerticesCount,
                       (dNewVerticesIDs_ + verticesCount_ - 1).get(),
                       sizeof(uint),
                       hipMemcpyDeviceToHost);
            ++newVerticesCount;

            // There are two special cases when we can stop our algorithm:
            // 1) number of vertices in the graph remained unchanged;
            // 2) only one vertex remains.
            if (newVerticesCount == verticesCount_)
            {
                return ALGORITHM_FINISHED;
            }
            else if (newVerticesCount == 1)
            {
                thrust::device_ptr<uint> dDummyVerticesOffsets =
                    pools.uintVertices.get();

                hipMemset((void *) dDummyVerticesOffsets.get(),
                           0,
                           sizeof(uint));

                thrust::device_ptr<uint> dDummyVerticesIDs =
                    pools.uintVertices.get();

                thrust::sequence(dDummyVerticesIDs,
                                 dDummyVerticesIDs + verticesCount_);

                segmentations.addLevel(1,
                                       verticesCount_,
                                       dDummyVerticesOffsets,
                                       dDummyVerticesIDs);

                return ALGORITHM_FINISHED;
            }

            // Calculate how old vertices IDs map to new vertices IDs.
            thrust::device_ptr<uint> dVerticesMapping =
                pools.uintVertices.get();

            getVerticesMapping
            <<< gridDimsForVertices, blockDimsForVertices, 0 >>>
            (dClusteredVerticesIDs.get(),
             dNewVerticesIDs_.get(),
             dVerticesMapping.get(),
             verticesCount_);
            getLastCudaError("getVerticesMapping launch failed.");

            pools.uintVertices.put(dNewVerticesIDs_);
            pools.uintVertices.put(dClusteredVerticesIDs);
            pools.uintVertices.put(dSuccessors);

            // Invalidate self-loops in the reduced graph (the graph
            // produced by merging all old vertices that have
            // the same successor).
            invalidateLoops<<< gridDimsForEdges, blockDimsForEdges, 0 >>>
            (dStartpoints.get(),
             dVerticesMapping.get(),
             dEdges_.get(),
             edgesCount_);
            getLastCudaError("invalidateLoops launch failed.");

            // Calculate various information about the surviving
            // (new startpoints IDs and IDs of edges) and
            // non-surviving/contracted edges (their weights).
            thrust::device_ptr<uint> dNewStartpoints = pools.uintEdges.get();
            thrust::device_ptr<uint> dSurvivedEdgesIDs = pools.uintEdges.get();

            calculateEdgesInfo<<< gridDimsForEdges, blockDimsForEdges, 0 >>>
            (dStartpoints.get(),
             dVerticesMapping.get(),
             dEdges_.get(),
             dWeights_.get(),
             dNewStartpoints.get(),
             dSurvivedEdgesIDs.get(),
             edgesCount_,
             newVerticesCount);
            getLastCudaError("calculateEdgesInfo launch failed.");

            pools.uintEdges.put(dStartpoints);

            // Group that information by the new startpoints IDs.
            // Keep in mind that we want to build new (reduced) graph and apply
            // the step of the algorithm to that one. Hence we need to
            // preserve the structure of the original graph: neighbours and
            // weights should be grouped by vertex.
            thrust::sort(
                thrust::make_zip_iterator(
                    thrust::make_tuple(dNewStartpoints,
                                       dSurvivedEdgesIDs)),
                thrust::make_zip_iterator(
                    thrust::make_tuple(dNewStartpoints + edgesCount_,
                                       dSurvivedEdgesIDs + edgesCount_)));

            // Find the group of contracted edges.
            uint *invalidEdgesPtr =
                thrust::find_if(
                    dNewStartpoints,
                    dNewStartpoints + edgesCount_,
                    IsGreaterEqualThan<uint>(newVerticesCount)).get();

            // Calculate how many edges there are in the reduced graph.
            uint validEdgesCount =
                static_cast<uint>(invalidEdgesPtr - dNewStartpoints.get());

            // Mark groups of edges corresponding to the same vertex in the
            // reduced graph.
            thrust::adjacent_difference(dNewStartpoints,
                                        dNewStartpoints + edgesCount_,
                                        dEdgesFlags,
                                        thrust::not_equal_to<uint>());

            hipMemset((void *) dEdgesFlags.get(), 0, sizeof(uint));
            hipMemset((void *) dEdgesFlags.get(), 1, 1);

            pools.uintEdges.put(dNewStartpoints);

            // Now we are able to build the reduced graph. See "Graph"
            // class for the details on the graph's internal structure.

            // Calculate vertices' offsets for the reduced graph.
            thrust::copy_if(thrust::make_counting_iterator(0U),
                            thrust::make_counting_iterator(validEdgesCount),
                            dEdgesFlags,
                            dVertices_,
                            thrust::identity<uint>()).get();

            pools.uintEdges.put(dEdgesFlags);

            // Build up a neighbourhood for each vertex in the reduced graph
            // (this includes recalculating edges' weights).
            calculateThreadsDistribution(validEdgesCount,
                                         blocksCount,
                                         threadsPerBlockCount);
            dim3 newGridDimsForEdges(blocksCount, 1, 1);
            dim3 newBlockDimsForEdges(threadsPerBlockCount, 1, 1);

            thrust::device_ptr<uint> dNewEdges = pools.uintEdges.get();
            thrust::device_ptr<float> dNewWeights = pools.floatEdges.get();

            makeNewEdges<<< newGridDimsForEdges,
                         newBlockDimsForEdges,
                         0 >>>
                         (dSurvivedEdgesIDs.get(),
                          dVerticesMapping.get(),
                          dEdges_.get(),
                          dWeights_.get(),
                          dNewEdges.get(),
                          dNewWeights.get(),
                          validEdgesCount);
            getLastCudaError("makeNewEdges launch failed.");

            swap(dEdges_, dNewEdges);
            swap(dWeights_, dNewWeights);

            pools.uintEdges.put(dNewEdges);
            pools.floatEdges.put(dNewWeights);

            pools.uintEdges.put(dSurvivedEdgesIDs);

            // The graph's reconstruction is now finished.

            // Build new level of the segmentation tree. It is a trivial task
            // as we already have "dVerticesMapping" that contains all
            // sufficient information about the vertices' transformations.
            thrust::device_ptr<uint> dVerticesIDs =
                pools.uintVertices.get();
            thrust::device_ptr<uint> dNewVerticesOffsets =
                pools.uintVertices.get();

            thrust::sequence(dVerticesIDs, dVerticesIDs + verticesCount_);

            thrust::sort_by_key(dVerticesMapping,
                                dVerticesMapping + verticesCount_,
                                dVerticesIDs);

            thrust::unique_by_key_copy(dVerticesMapping,
                                       dVerticesMapping + verticesCount_,
                                       thrust::make_counting_iterator(0),
                                       thrust::make_discard_iterator(),
                                       dNewVerticesOffsets);

            segmentations.addLevel(newVerticesCount,
                                   verticesCount_,
                                   dNewVerticesOffsets,
                                   dVerticesIDs);

            pools.uintVertices.put(dVerticesIDs);
            pools.uintVertices.put(dNewVerticesOffsets);
            pools.uintVertices.put(dVerticesMapping);

            // We can now safely set new counts for vertices and edges.
            verticesCount_ = newVerticesCount;
            edgesCount_ = validEdgesCount;

            return ALGORITHM_NOT_FINISHED;
        }

        uint verticesCount_;
        uint edgesCount_;

        thrust::device_ptr<uint> dVertices_;
        thrust::device_ptr<uint> dEdges_;
        thrust::device_ptr<float> dWeights_;

        thrust::device_ptr<uint> dOutputEdgesFlags_;
};

// Loads PPM image.
int loadImage(const char *filename,
              const char *executablePath,
              vector<uchar3> &data,
              uint &width,
              uint &height)
{
    const char *imagePath = sdkFindFilePath(filename, executablePath);

    if (imagePath == NULL)
    {
        return -1;
    }

    uchar *dataHandle = NULL;
    unsigned int channels;

    if (!__loadPPM(imagePath, &dataHandle, &width, &height, &channels))
    {
        return -1;
    }

    data.assign(reinterpret_cast<uchar3 *>(dataHandle),
                reinterpret_cast<uchar3 *>(dataHandle) + width * height);

    free(reinterpret_cast<void *>(dataHandle));

    return 0;
}

inline float distance(const uchar3 &first, const uchar3 &second)
{
    int dx = static_cast<int>(first.x) - static_cast<int>(second.x);
    int dy = static_cast<int>(first.y) - static_cast<int>(second.y);
    int dz = static_cast<int>(first.z) - static_cast<int>(second.z);

    uint sqrResult = dx * dx + dy * dy + dz * dz;

    return sqrt(static_cast<float>(sqrResult));
}

// Builds a net-graph for the image with 4-connected pixels.
void buildGraph(const vector<uchar3> &image,
                uint width,
                uint height,
                Graph &graph)
{
    uint totalNodes = static_cast<uint>(image.size());

    graph.vertices.resize(totalNodes);
    graph.edges.reserve(4 * totalNodes - 2 * (width + height));
    graph.weights.reserve(graph.edges.size());

    uint edgesProcessed = 0;

    for (uint y = 0; y < height; ++y)
    {
        for (uint x = 0; x < width; ++x)
        {
            uint nodeIndex = y * width + x;
            const uchar3 &centerPixel = image[nodeIndex];

            graph.vertices[nodeIndex] = edgesProcessed;

            if (y > 0)
            {
                uint lowerNodeIndex = (y - 1) * width + x;
                const uchar3 &lowerPixel = image[lowerNodeIndex];

                graph.edges.push_back(lowerNodeIndex);
                graph.weights.push_back(distance(centerPixel, lowerPixel));

                ++edgesProcessed;
            }

            if (y + 1 < height)
            {
                uint upperNodeIndex = (y + 1) * width + x;
                const uchar3 &upperPixel = image[upperNodeIndex];

                graph.edges.push_back(upperNodeIndex);
                graph.weights.push_back(distance(centerPixel, upperPixel));

                ++edgesProcessed;
            }

            if (x > 0)
            {
                uint leftNodeIndex = y * width + x - 1;
                const uchar3 &leftPixel = image[leftNodeIndex];

                graph.edges.push_back(leftNodeIndex);
                graph.weights.push_back(distance(centerPixel, leftPixel));

                ++edgesProcessed;
            }

            if (x + 1 < width)
            {
                uint rightNodeIndex = y * width + x + 1;
                const uchar3 &rightPixel = image[rightNodeIndex];

                graph.edges.push_back(rightNodeIndex);
                graph.weights.push_back(distance(centerPixel, rightPixel));

                ++edgesProcessed;
            }
        }
    }
}

void printUsage() {
    puts("Usage: ./segmentationTreeThrust -i [input image path] -o [output image path]");
    puts("Options:");
    puts("\t-i: Path to input file (eg: data/beach.ppm)");
    puts("\t-o: Path to output file (eg: segmented.ppm)");
    puts("Benchmarking options");
    puts("\t-w: Number of iterations to perform during warmup");
    puts("\t-b: Number of iterations to perform during benchmarking");
    puts("\t-p: If want to do partial timings");
    puts("\t-n: Don't write images to disk (for benchmarking purposes)");
    exit(1);
}


void algo_wrapper(uint imageWidth, uint imageHeight, bool no_output_local) {
    std::chrono::high_resolution_clock::time_point start, end;

    if (!partial) {
        start = std::chrono::high_resolution_clock::now();
    }

    if (partial) {
        start = std::chrono::high_resolution_clock::now();
    }

    Graph graph;
    buildGraph(in_image, imageWidth, imageHeight, graph);

    if (partial) {
        hipDeviceSynchronize();
        end = std::chrono::high_resolution_clock::now();
        int time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
        printf("0, %d", time);
    }

    if (partial) {
        start = std::chrono::high_resolution_clock::now();
    }

    Pyramid segmentations;

    cerr << "* Building segmentation tree... ";
    cerr.flush();

    SegmentationTreeBuilder algo;
    float elapsedTime = algo.run(graph, segmentations);

    if (partial) {
        hipDeviceSynchronize();
        end = std::chrono::high_resolution_clock::now();
        int time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
        printf(", %d", time);
    }

    cerr << "* Dumping levels for each tree..." << endl << endl;

    if (partial) {
        start = std::chrono::high_resolution_clock::now();
    }

    segmentations.dump(imageWidth, imageHeight, no_output_local);

    if (partial) {
        hipDeviceSynchronize();
        end = std::chrono::high_resolution_clock::now();
        int time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
        printf(", %d\n", time);
    }

    if (!partial) {
        hipDeviceSynchronize();
        end = std::chrono::high_resolution_clock::now();
        int time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
        printf("%d\n", time);
    }
}

void handleParams(int argc, char **argv) {
    for(;;)
    {
        switch(getopt(argc, argv, "pnhi:o:w:b:"))
        {
            case 'i': {
                in_imageName = std::string(optarg);
                continue;
            }
            case 'o': {
                outputPath = std::string(optarg);
                continue;
            }
            case 'w': {
                w = atoi(optarg);
                continue;
            }
            case 'b': {
                b = atoi(optarg);
                continue;
            }
            case 'p': {
                partial = true;
                continue;
            }
            case 'n': {
                no_output = true;
                continue;
            }
            case '?':
            case 'h':
            default : {
                printUsage();
                break;
            }

            case -1:  {
                break;
            }
        }
        break;
    }
}

int main(int argc, char **argv)
{
    uint imageWidth, imageHeight;
    handleParams(argc, argv);
    


    fprintf(stderr,"%s Starting...\n\n", argv[0]);


    if (loadImage(in_imageName.c_str(), argv[0], in_image, imageWidth, imageHeight) != 0)
    {
        fprintf(stderr, "Failed to open <%s>, program exit...\n", in_imageName.c_str());
        exit(EXIT_FAILURE);
    }

    if (partial) {
        printf("gaussian, graph, segmentation, output\n");
    } else {
        printf("total\n");
    }

    for (int i = 0; i < w; i++) {
        algo_wrapper(imageWidth, imageHeight, true);
    }

    for (int i = 0; i < b; i++) {
        if (i == b-1) {
            algo_wrapper(imageWidth, imageHeight, no_output);
        } else {
            algo_wrapper(imageWidth, imageHeight, true);
        }
    }
    
}
