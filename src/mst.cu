#include "hip/hip_runtime.h"
//
// Created by gyorgy on 16/11/2020.
//

#include <stdio.h>
#include <iostream>

#include "mst.h"

#define CHANNEL_SIZE 3
#define K 100
#define NUM_NEIGHBOURS 8

/*
 * Matrix structure:
 *      - vertices array of type uint4, where
 *          * x = id (starting at 1, so that 1 can represent null node)
 *          * y = component id
 *          * z = component size
 *          * w = component internal difference // This probably needs to be a float?
 *
 *      - edges 2D array of type uint2, where
 *          * x = destination id
 *          * y = weight (dissimilarity)
 *
 * Min edges:
 *      - x = weight
 *      - y = source id
 *      - z = destination id
 *      - w = component id
 */

typedef struct {
    uint weight;
    uint src_id;
    uint src_comp;
    uint dest_id;
    uint dest_comp;
    uint new_int_diff;
} min_edge;

// Kernel to encode graph
__global__
void encode(u_char *image, uint4 vertices[], uint2 edges[], uint x_len, uint y_len, size_t pitch) {
    uint x_pos = blockDim.x * blockIdx.x + threadIdx.x;
    if (x_pos >= x_len) return;
    uint y_pos = blockDim.y * blockIdx.y + threadIdx.y;
    if (y_pos >= y_len) return;

    uint this_id = (x_pos * y_len + y_pos);
    uint4 *this_vertice = &vertices[this_id];
    this_vertice->x = this_id + 1;
    this_vertice->y = this_id + 1;
    this_vertice->z = 1;
    this_vertice->w = 0;

    uint this_start = x_pos * pitch + y_pos * CHANNEL_SIZE;
    u_char this_r = image[this_start];
    u_char this_g = image[this_start + 1];
    u_char this_b = image[this_start + 2];

    // Maybe could have 4 edges instead of 8?
    uint2 *edge;
    uint edge_id;
    uint other_start;
    u_char other_r;
    u_char other_g;
    u_char other_b;
    bool is_first_col = y_pos <= 0;
    bool is_last_col = y_pos >= y_len - 1;

    if (x_pos > 0) {
        uint prev_row = this_id - y_len;
        if (!is_first_col) {
            edge_id = prev_row - 1;
            other_start = (x_pos - 1) * pitch + (y_pos - 1) * CHANNEL_SIZE;
            other_r = image[other_start];
            other_g = image[other_start + 1];
            other_b = image[other_start + 2];
            edge = &edges[this_id * NUM_NEIGHBOURS];
            edge->x = edge_id + 1;
            edge->y = sqrtf(powf(this_r-other_r, 2.0f) + powf(this_g-other_g, 2.0f) + powf(this_b-other_b, 2.0f));
        }

        edge_id = prev_row;
        other_start = (x_pos - 1) * pitch + (y_pos) * CHANNEL_SIZE;
        other_r = image[other_start];
        other_g = image[other_start + 1];
        other_b = image[other_start + 2];
        edge = &edges[this_id * NUM_NEIGHBOURS + 1];
        edge->x = edge_id + 1;
        edge->y = sqrtf(powf(this_r-other_r, 2.0f) + powf(this_g-other_g, 2.0f) + powf(this_b-other_b, 2.0f));

        if (!is_last_col) {
            edge_id = prev_row + 1;
            other_start = (x_pos - 1) * pitch + (y_pos + 1) * CHANNEL_SIZE;
            other_r = image[other_start];
            other_g = image[other_start + 1];
            other_b = image[other_start + 2];
            edge = &edges[this_id * NUM_NEIGHBOURS + 2];
            edge->x = edge_id + 1;
            edge->y = sqrtf(powf(this_r-other_r, 2.0f) + powf(this_g-other_g, 2.0f) + powf(this_b-other_b, 2.0f));
        }
    }

    if (x_pos < x_len - 1) {
        uint next_row = this_id + y_len;
        if (!is_first_col) {
            edge_id = next_row - 1;
            other_start = (x_pos + 1) * pitch + (y_pos - 1) * CHANNEL_SIZE;
            other_r = image[other_start];
            other_g = image[other_start + 1];
            other_b = image[other_start + 2];
            edge = &edges[this_id * NUM_NEIGHBOURS + 3];
            edge->x = edge_id + 1;
            edge->y = sqrtf(powf(this_r-other_r, 2.0f) + powf(this_g-other_g, 2.0f) + powf(this_b-other_b, 2.0f));
        }

        edge_id = next_row;
        other_start = (x_pos + 1) * pitch + (y_pos) * CHANNEL_SIZE;
        other_r = image[other_start];
        other_g = image[other_start + 1];
        other_b = image[other_start + 2];
        edge = &edges[this_id * NUM_NEIGHBOURS + 4];
        edge->x = edge_id + 1;
        edge->y = sqrtf(powf(this_r-other_r, 2.0f) + powf(this_g-other_g, 2.0f) + powf(this_b-other_b, 2.0f));

        if (!is_last_col) {
            edge_id = next_row + 1;
            other_start = (x_pos + 1) * pitch + (y_pos + 1) * CHANNEL_SIZE;
            other_r = image[other_start];
            other_g = image[other_start + 1];
            other_b = image[other_start + 2];
            edge = &edges[this_id * NUM_NEIGHBOURS + 5];
            edge->x = edge_id + 1;
            edge->y = sqrtf(powf(this_r-other_r, 2.0f) + powf(this_g-other_g, 2.0f) + powf(this_b-other_b, 2.0f));
        }
    }

    if (!is_first_col) {
        edge_id = this_id - 1;
        other_start = (x_pos) * pitch + (y_pos - 1) * CHANNEL_SIZE;
        other_r = image[other_start];
        other_g = image[other_start + 1];
        other_b = image[other_start + 2];
        edge = &edges[this_id * NUM_NEIGHBOURS + 6];
        edge->x = edge_id + 1;
        edge->y = sqrtf(powf(this_r-other_r, 2.0f) + powf(this_g-other_g, 2.0f) + powf(this_b-other_b, 2.0f));
    }

    if (!is_last_col) {
        edge_id = this_id + 1;
        other_start = (x_pos) * pitch + (y_pos + 1) * CHANNEL_SIZE;
        other_r = image[other_start];
        other_g = image[other_start + 1];
        other_b = image[other_start + 2];
        edge = &edges[this_id * NUM_NEIGHBOURS + 7];
        edge->x = edge_id + 1;
        edge->y = sqrtf(powf(this_r-other_r, 2.0f) + powf(this_g-other_g, 2.0f) + powf(this_b-other_b, 2.0f));
    }
}

// Kernel to decode graph
__global__
void decode(uint4 vertices[], char *image, char* colours, uint num_vertices) {
    uint pos = blockDim.x * blockIdx.x + threadIdx.x;
    if (pos >= num_vertices) return;

    uint img_pos = pos * CHANNEL_SIZE;
    uint colour_start = (vertices[pos].y - 1) * CHANNEL_SIZE;
    image[img_pos] = colours[colour_start];
    image[img_pos + 1] = colours[colour_start + 1];
    image[img_pos + 2] = colours[colour_start + 2];
}

__global__
void find_min_edges_sort(uint4 vertices[], uint2 edges[], min_edge min_edges[], uint vertices_length) {
    uint index = blockDim.x * blockIdx.x + threadIdx.x;
    uint num_threads = gridDim.x * blockDim.x;
    for (uint tid = index; tid < vertices_length; tid += num_threads) {
        uint4 vertice = vertices[tid];
        min_edge min;
        min.weight = UINT_MAX;
        min.src_id = 0;
        min.src_comp = 0;
        for (int j = tid * NUM_NEIGHBOURS; j < tid * NUM_NEIGHBOURS + NUM_NEIGHBOURS; j++) {
            uint2 edge = edges[j];
            // Maybe it would be better to just check if it's not in the same component? We would not need to remove internal edges
            if (edge.x != 0) {
                if (edge.y < min.weight) {
                    min.src_id = vertice.x;
                    min.src_comp = vertice.y;
                    min.dest_id = edge.x;
                    min.dest_comp = vertices[edge.x - 1].y; //edge.z;
                    min.weight = edge.y;
                }
            }
        }
        min_edges[tid] = min;
    }
}

__device__ __forceinline__
int compare_min_edges(min_edge left, min_edge right) {
    //printf("Compare %d with %d\n", left.src_comp, right.src_comp);
    if (left.src_comp == 0 && right.src_comp == 0) return 0;
    if (left.src_comp == 0) return 1;
    if (right.src_comp == 0) return -1;
    uint component_diff = left.src_comp - right.src_comp;
    if (component_diff != 0) return component_diff;
    return left.weight - right.weight;
}

__global__
void sort_min_edges(min_edge min_edges[], uint vertices_length, uint offset, uint *not_sorted) {
    uint index = (blockDim.x * blockIdx.x + threadIdx.x) * 2 + offset;
    uint num_threads = gridDim.x * blockDim.x;

    for (uint tid = index; tid < vertices_length - 1; tid += (num_threads * 2)) {
        //printf("tid: %d\n", tid);
        min_edge left = min_edges[tid];
        min_edge right = min_edges[tid + 1];
        if (compare_min_edges(left, right) > 0) {
            //printf("Swap: %d with %d\n", tid, tid+1);
            min_edges[tid] = right;
            min_edges[tid+1] = left;
            *not_sorted = 1;
        }
    }
}

__device__ __forceinline__
void sort_min_edges_wrapper(min_edge min_edges[], uint n_vertices, uint *did_change) {
    *did_change = 1;
    uint offset = 0;
    uint wanted_threads = n_vertices / 2;
    uint threads;
    uint blocks;
    if (wanted_threads <= 1024) {
        threads = wanted_threads;
        blocks = 1;
    } else {
        threads = 1024;
        blocks = wanted_threads / 1024 + 1;
    }
    while (*did_change == 1) {
        *did_change = 0;
        sort_min_edges<<<blocks, threads>>>(min_edges, n_vertices, offset, did_change);
        hipDeviceSynchronize();
        offset ^= 1;
        sort_min_edges<<<blocks, threads>>>(min_edges, n_vertices, offset, did_change);
        hipDeviceSynchronize();
        offset ^= 0;
    }
}

__global__
void compact_min_edges(min_edge min_edges[], uint n_vertices, uint *pos_counter) {
    uint tid = blockDim.x * blockIdx.x + threadIdx.x;
    uint num_threads = gridDim.x * blockDim.x;
    for (int index = tid + 1; index < n_vertices - 1; index += num_threads) {
        uint pos;
        min_edge left = min_edges[index];
        min_edge right = min_edges[index + 1];
        bool write = right.src_comp != left.src_comp && right.src_comp != 0;
        if (write) {
            pos = atomicAdd_system(pos_counter, 1);
        }
        __syncthreads();
        if (write) {
            min_edges[pos] = right;
        }
    }
}

__global__
void construct_sources(min_edge min_edges[], uint num_components, uint2 sources[]) {
    uint component_id = blockDim.x * blockIdx.x + threadIdx.x;
    uint num_threads = gridDim.x * blockDim.x;
    for (uint comp_id = component_id; comp_id < num_components; comp_id += num_threads) {
        min_edge *edge = &min_edges[comp_id];
        //if (comp_id == 0) printf("Src comp %d\n", min_edges[comp_id].src_comp);
        sources[edge->src_comp - 1].x = edge->dest_comp;
        sources[edge->src_comp - 1].y = edge->weight;
    }
}

__global__
void update_destinations(min_edge min_edges[], uint num_components, uint2 sources[], uint *did_change) {
    uint component_id = blockDim.x * blockIdx.x + threadIdx.x;
    uint num_threads = gridDim.x * blockDim.x;
    for (uint comp_id = component_id; comp_id < num_components; comp_id += num_threads) {
        min_edge *edge = &min_edges[comp_id];
        uint src = edge->src_comp;
        uint dest = edge->dest_comp;
        uint weight = edge->weight;
        uint new_dest = sources[dest - 1].x;
        uint new_weight = sources[dest - 1].y;
        if (((new_dest == src) || (new_dest != src && new_dest != dest && weight == new_weight)) && src < dest) {
            edge->dest_comp = new_dest;
            *did_change = 1;
        }
    }
}

__device__ __forceinline__
void remove_deps(min_edge min_edges[], uint num_components, uint2 sources[], uint blocks, uint threads, uint* did_change) {
    *did_change = 1;
    while (*did_change == 1) {
        *did_change = 0;
        construct_sources<<<blocks, threads>>>(min_edges, num_components, sources);
        hipDeviceSynchronize();
        update_destinations<<<blocks, threads>>>(min_edges, num_components, sources, did_change);
        hipDeviceSynchronize();
    }
}

// Kernel to remove cycles
__global__
void remove_cycles(min_edge min_edges[], uint num_components) {
    uint component_id_x = blockDim.x * blockIdx.x + threadIdx.x;
    uint num_threads_x = gridDim.x * blockDim.x;

    uint component_id_y = blockDim.y * blockIdx.y + threadIdx.y;
    uint num_threads_y = gridDim.y * blockDim.y;

    if (component_id_x == component_id_y) return;

    for (uint comp_x = component_id_x; comp_x < num_components; comp_x += num_threads_x) {
        for (uint comp_y = component_id_y; comp_y < num_components; comp_y += num_threads_y) {
            min_edge *x_edge = &min_edges[comp_x];
            min_edge *y_edge = &min_edges[comp_y];

            uint x_src = x_edge->src_comp;
            uint x_dest = x_edge->dest_comp;

            uint y_src = y_edge->src_comp;
            uint y_dest = y_edge->dest_comp;

            bool not_root = x_src != x_dest;
            bool has_dep = x_src == y_dest;
            bool can_update = comp_x > comp_y && x_dest == y_src;

            __syncthreads();
            if (not_root && has_dep && can_update) {
                y_edge->dest_comp = x_dest;
            }
        }
    }
}

__device__ __forceinline__
void remove_cycles_wrapper(min_edge min_edges[], uint curr_n_comp, dim3 cycle_blocks, dim3 cycle_threads) {
    //printf("Cycles: (%d, %d), (%d, %d)\n", cycle_blocks.x, cycle_blocks.y, cycle_threads.x, cycle_threads.y);
    remove_cycles<<<cycle_blocks, cycle_threads>>>(min_edges, curr_n_comp);
    hipDeviceSynchronize();
}

// Kernel to update the whole matrix
__global__
void update_whole_matrix(uint4 vertices[], min_edge min_edges[], uint num_components, uint num_vertices) {
    uint component_id = blockDim.x * blockIdx.x + threadIdx.x;
    uint comp_threads = gridDim.x * blockDim.x;
    uint vertice_id = blockDim.y * blockIdx.y + threadIdx.y;
    uint v_threads = gridDim.y * blockDim.y;

    for (int comp_id = component_id; comp_id < num_components; comp_id += comp_threads) {
        min_edge current_comp = min_edges[comp_id];
        if (current_comp.src_comp == current_comp.dest_comp || current_comp.weight > 0) continue;
        //printf("Merge %d into %d\n", current_comp.src_comp, current_comp.dest_comp);

        // If we merge
        for (int v_id = vertice_id; v_id < num_vertices; v_id += v_threads) {
            uint4 *vertice = &vertices[v_id];
            // Change just the parent and then path compression takes care of it?
            if (vertice->y == current_comp.src_comp) {
                vertice->y = current_comp.dest_comp;
            }
            else if (vertice->x == current_comp.dest_comp) {
                atomicMax_system(&(vertice->w), current_comp.new_int_diff);
            }
        }
    }
}

__device__
void print_vertice(uint4 vertices[], uint pos) {
    uint component = vertices[pos].y;
    uint4 parent = vertices[component - 1];
    printf("[%d] (%d, %d) -> (%d, %d)\n", pos, vertices[pos].x, component, parent.x, parent.y);
}

__global__
void path_compression(uint4 vertices[], uint num_vertices) {
    uint vertice_id = blockDim.x * blockIdx.x + threadIdx.x;
    uint comp_threads = gridDim.x * blockDim.x;

    for (int v_id = vertice_id; v_id < num_vertices; v_id += comp_threads) {
        uint4 *vertice = &vertices[v_id];

        if (vertice->x != vertice->y) {
            uint4 *parent = &vertices[vertice->y - 1];
            while(parent->y != parent->x) {parent = &vertices[parent->y - 1]; /*printf("%d -> %d\n", parent->x, parent->y);*/ /*if (parent->x == 17978) print_vertice(vertices, 17977);*/}
            //printf("%d has root: %d\n", v_id, parent->x);

            vertice->y = parent->x;
            atomicAdd_system(&(parent->z), vertice->z);
            atomicMax_system(&(parent->w), vertice->w);
        }
    }
}

__global__
void update_new_size(uint4 vertices[], uint num_vertices, uint2 edges[]) {
    uint vertice_id = blockDim.x * blockIdx.x + threadIdx.x;
    uint comp_threads = gridDim.x * blockDim.x;

    for (int v_id = vertice_id; v_id < num_vertices; v_id += comp_threads) {
        uint4 *vertice = &vertices[v_id];

        if (vertice->x != vertice->y) {
            uint4 *parent = &vertices[vertice->y - 1];
            vertice->z = parent->z;
            vertice->w = parent->w;
        }

        for (int j = v_id * NUM_NEIGHBOURS; j < v_id * NUM_NEIGHBOURS + NUM_NEIGHBOURS; j++) {
            uint2 *neighbour_edge = &edges[j];
            if (neighbour_edge->x != 0) {
                if (vertices[neighbour_edge->x - 1].y == vertice->y) {
                    neighbour_edge->x = 0; // Remove internal edges
                }
            }
        }
    }
}

// Kernel to merge components
__global__
void merge(uint4 vertices[], min_edge min_edges[], uint *num_components, uint update_threads, uint update_blocks, uint vertices_length, uint comp_count) {
    uint component_id = blockDim.x * blockIdx.x + threadIdx.x;
    uint num_threads = gridDim.x * blockDim.x;
    for (uint comp_id = component_id; comp_id < comp_count; comp_id += num_threads) {

        min_edge min_edge = min_edges[comp_id];
        if (min_edge.src_comp == min_edge.dest_comp || min_edge.src_comp == 0) return;
        uint4 src = vertices[min_edge.src_id - 1];
        uint4 dest = vertices[min_edge.dest_id - 1];
        uint src_diff = src.w + (K / src.z);
        uint dest_diff = dest.w + (K / dest.z);
        __syncthreads();

        if (min_edge.weight <= min(src_diff, dest_diff)) {
            //printf("Merge %d into %d\n", min_edge.src_comp, min_edge.dest_comp);
            atomicSub_system(num_components, 1); // Is this horribly inefficient?
            uint new_int_diff = max(max(dest.w, src.w), min_edge.weight);
            min_edges[comp_id].weight = 0;
            min_edges[comp_id].new_int_diff = new_int_diff;
        }
    }
}

__global__
void debug_print_min_edges(min_edge min_edges[], uint length) {
    for (int i = 0; i < length; i++) {
        if (min_edges[i].src_comp == 0) continue;
        printf("[%d]: %d(%d) -(%d)-> %d (%d)\n", i, min_edges[i].src_comp, min_edges[i].src_id, min_edges[i].weight, min_edges[i].dest_comp, min_edges[i].dest_id);
    }
    printf("\n");
}

__global__
void debug_print_vertices(uint4 vertices[], uint length, uint2 edges[]) {
    for (int v_id = 0; v_id < length; v_id++) {
        printf("vertices[%d] = %d %d %d | ", v_id, vertices[v_id].x, vertices[v_id].y, vertices[v_id].z);
        for (int j = v_id * NUM_NEIGHBOURS; j < v_id * NUM_NEIGHBOURS + NUM_NEIGHBOURS; j++) {
            printf("%d(%d), ", edges[j].x, edges[j].y);
        }
        printf("\n");
    }
}

// Kernel to orchestrate
__global__
void segment(uint4 vertices[], uint2 edges[], min_edge min_edges[], uint2 sources[], uint *n_components, uint *did_change) {
    uint counter = 0;
    uint prev_n_components = 0;
    uint n_vertices = *n_components;
    uint curr_n_comp = *n_components;
    dim3 threads;
    dim3 blocks;
    dim3 cycle_blocks;
    dim3 cycle_threads;
    dim3 update_blocks;
    dim3 update_threads;
    if (n_vertices < 1024) {
        threads.y = n_vertices;
        blocks.y = 1;
    } else {
        threads.y = 1024;
        blocks.y = min(n_vertices / 1024 + 1, 65535);
    }

    printf("N components: %d\n", curr_n_comp);
    while (curr_n_comp != prev_n_components) {
        if (curr_n_comp < 1024) {
            threads.x = curr_n_comp;
            blocks.x = 1;
        } else {
            threads.x = 1024;
            blocks.x = min(curr_n_comp / 1024 + 1, 65535);
        }

        if (curr_n_comp < 32) {
            cycle_threads.x = curr_n_comp;
            cycle_threads.y = cycle_threads.x;

            cycle_blocks.x = 1;
            cycle_blocks.y = 1;
        } else {
            cycle_threads.x = 32;
            cycle_blocks.x = min(curr_n_comp / 32 + 1, 65535);

            cycle_threads.y = cycle_threads.x;
            cycle_blocks.y = cycle_blocks.x;
        }

        if (curr_n_comp * n_vertices < 1024 && curr_n_comp < 1024 && n_vertices < 1024) {
            update_threads.x = curr_n_comp;
            update_threads.y = n_vertices;

            update_blocks.x = 1;
            update_blocks.y = 1;
        } else {
            update_threads.x = 32;
            update_threads.y = 32;

            update_blocks.x = min(curr_n_comp / 32 + 1, 65535);
            update_blocks.y = min(n_vertices / 32 + 1, 65535);
        }
        //printf("Update: (%d, %d) (%d, %d)\n", update_blocks.x, update_blocks.y, update_threads.x, update_threads.y);

        printf("Find min edges\n");
        find_min_edges_sort<<<blocks.y, threads.y>>>(vertices, edges, min_edges, n_vertices);
        hipDeviceSynchronize();
        // First time there is no point in doing these, since n_vertices == n_components
        if (counter > 0) {
            printf("Sort\n");
            sort_min_edges_wrapper(min_edges, n_vertices, did_change);
            hipDeviceSynchronize();
            printf("Compact\n");
            *did_change = 1;
            compact_min_edges<<<blocks.y, threads.y>>>(min_edges, n_vertices, did_change);
            hipDeviceSynchronize();
        }

        // Need to enhance this so runs faster
        // Somehow reduce the number of threads
        // Only detect circular merges and apply path compression at the end of iteration?
        printf("Remove cycles\n");
        if (false) {
            debug_print_min_edges<<<1, 1>>>(min_edges, curr_n_comp);
            hipDeviceSynchronize();
        }
        //remove_cycles_wrapper(min_edges, curr_n_comp, cycle_blocks, cycle_threads);
        remove_deps(min_edges, curr_n_comp, sources ,blocks.x, threads.x, did_change);
        if (false) {
            debug_print_min_edges<<<1, 1>>>(min_edges, curr_n_comp);
            hipDeviceSynchronize();
            return;
        }

        printf("Merge\n");
        merge<<<blocks.x, threads.x>>>(vertices, min_edges, n_components, threads.y, blocks.y, n_vertices, curr_n_comp);
        hipDeviceSynchronize();
        //debug_print_min_edges<<<1, 1>>>(min_edges, curr_n_comp);
        //hipDeviceSynchronize();
        //printf("Update whole matrix: (%d, %d), (%d, %d)\n", update_blocks.x, update_blocks.y, update_threads.x, update_threads.y);
        printf("Update\n");
        update_whole_matrix<<<update_blocks, update_threads>>>(vertices, min_edges, curr_n_comp, n_vertices);
        hipDeviceSynchronize();
        printf("Path compress\n");
        path_compression<<<blocks.y, threads.y>>>(vertices, n_vertices);
        hipDeviceSynchronize();
        printf("New size\n");
        update_new_size<<<blocks.y, threads.y>>>(vertices, n_vertices, edges);
        hipDeviceSynchronize();

        //debug_print_vertices<<<1, 1>>>(vertices, n_vertices, edges);
        //hipDeviceSynchronize();

        prev_n_components = curr_n_comp;
        curr_n_comp = *n_components;
        printf("N components: %d\n", curr_n_comp);
        counter++;
        //return;
    }
    printf("Iterations: %d\n", counter);

}

void get_component_colours(char colours[], uint num_colours) {
    srand(123456789);
    for (int i = 0; i < num_colours * CHANNEL_SIZE; i++) {
        colours[i] = rand() % 256;
    }
}

void checkErrors(const char *identifier) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << " " << identifier << std::endl;
}

char *compute_segments(void *input, uint x, uint y, size_t pitch) {
    uint4 *vertices;
    uint2 *edges;
    uint2 *sources;
    min_edge *min_edges;
    uint num_vertices = (x) * (y);
    uint *num_components;
    uint *did_change;

    hipMalloc(&vertices, num_vertices*sizeof(uint4));
    checkErrors("Malloc vertices");
    hipMalloc(&edges, num_vertices*NUM_NEIGHBOURS*sizeof(uint2));
    checkErrors("Malloc edges");
    hipMalloc(&min_edges, num_vertices*sizeof(min_edge)); // max(min_edges) == vertices.length
    checkErrors("Malloc min_edges");
    hipMalloc(&sources, num_vertices*sizeof(uint2));
    checkErrors("Malloc sources");
    hipMalloc(&num_components, sizeof(uint));
    checkErrors("Malloc num components");
    hipMalloc(&did_change, sizeof(uint));
    checkErrors("Malloc did change");

    hipMemcpyAsync(num_components, &num_vertices, sizeof(uint), hipMemcpyHostToDevice);
    checkErrors("Memcpy num_vertices");

    // Write to the matrix from image
    // cudaOccupancyScheduler?
    dim3 encode_threads;
    dim3 encode_blocks;
    if (num_vertices < 1024) {
        encode_threads.x = x;
        encode_threads.y = y;
        encode_blocks.x = 1;
        encode_blocks.y = 1;
    } else {
        encode_threads.x = 32;
        encode_threads.y = 32;
        encode_blocks.x = x / 32 + 1;
        encode_blocks.y = y / 32 + 1;
    }

    encode<<<encode_blocks, encode_threads>>>((u_char*)input, vertices, edges, x, y, pitch);
    checkErrors("encode()");

    // Segment matrix
    //hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    segment<<<1, 1>>>(vertices, edges, min_edges, sources, num_components, did_change);
    hipDeviceSynchronize();
    checkErrors("segment()");

    // Setup random colours for components
    dim3 decode_threads;
    dim3 decode_blocks;
    if (num_vertices <= 1024) {
        decode_threads.x = num_vertices;
        decode_blocks.x = 1;
    } else {
        decode_threads.x = 1024;
        decode_blocks.x = num_vertices / 1024 + 1;
    }

    //char component_colours[num_vertices * CHANNEL_SIZE];
    char *component_colours = (char *) malloc(num_vertices * CHANNEL_SIZE * sizeof(char));
    get_component_colours(component_colours, num_vertices);
    char *component_colours_dev;
    hipMalloc(&component_colours_dev, num_vertices * CHANNEL_SIZE * sizeof(char));
    hipMemcpyAsync(component_colours_dev, component_colours, num_vertices * CHANNEL_SIZE * sizeof(char), hipMemcpyHostToDevice);
    char *output_dev;
    hipMalloc(&output_dev, num_vertices * CHANNEL_SIZE * sizeof(char ));

    // Write image back from segmented matrix
    decode<<<decode_blocks, decode_threads>>>(vertices, output_dev, component_colours_dev, num_vertices);
    hipDeviceSynchronize();
    checkErrors("decode()");

    // Clean up matrix
    hipFree(vertices);
    checkErrors("Free vertices");
    hipFree(edges);
    checkErrors("Free edges");
    hipFree(min_edges);
    checkErrors("Free min_edges");
    hipFree(component_colours_dev);
    checkErrors("Free component_colours_dev");

    //Copy image data back from GPU
    char *output = (char*) malloc(x*y*CHANNEL_SIZE*sizeof(char));

    hipMemcpy(output, output_dev, x*y*CHANNEL_SIZE*sizeof(char), hipMemcpyDeviceToHost);
    checkErrors("Memcpy output");

    return output;
}
