#include "hip/hip_runtime.h"
//
// Created by gyorgy on 16/11/2020.
//

#include <stdio.h>
#include <iostream>

#include "mst.h"

#define CHANNEL_SIZE 3
#define K 250
#define NUM_NEIGHBOURS 8

/*
 * Matrix structure:
 *      - vertices array of type uint4, where
 *          * x = id (starting at 1, so that 1 can represent null node)
 *          * y = component id
 *          * z = component size
 *          * w = component internal difference // This probably needs to be a float?
 *
 *      - edges 2D array of type uint3, where
 *          * x = destination id
 *          * y = weight (dissimilarity)
 *          * z = component id
 *
 * Min edges:
 *      - x = weight
 *      - y = source id
 *      - z = destination id
 */

// Kernel to encode graph
__global__
void encode(u_char *image, uint4 vertices[], uint3 edges[], uint x_len, uint y_len, size_t pitch) {
    uint x_pos = blockDim.x * blockIdx.x + threadIdx.x;
    if (x_pos >= x_len) return;
    uint y_pos = blockDim.y * blockIdx.y + threadIdx.y;
    if (y_pos >= y_len) return;

    uint this_id = (x_pos * y_len + y_pos);
    uint4 *this_vertice = &vertices[this_id];
    this_vertice->x = this_id + 1;
    this_vertice->y = this_id + 1;
    this_vertice->z = 1;
    this_vertice->w = 0;

    uint this_start = x_pos * pitch + y_pos * CHANNEL_SIZE;
    u_char this_r = image[this_start];
    u_char this_g = image[this_start + 1];
    u_char this_b = image[this_start + 2];

    // Maybe could have 4 edges instead of 8?
    uint3 *edge;
    uint edge_id;
    uint other_start;
    u_char other_r;
    u_char other_g;
    u_char other_b;
    bool is_first_col = y_pos <= 0;
    bool is_last_col = y_pos >= y_len - 1;

    if (x_pos > 0) {
        uint prev_row = this_id - y_len;
        if (!is_first_col) {
            edge_id = prev_row - 1;
            other_start = (x_pos - 1) * pitch + (y_pos - 1) * CHANNEL_SIZE;
            other_r = image[other_start];
            other_g = image[other_start + 1];
            other_b = image[other_start + 2];
            edge = &edges[this_id * NUM_NEIGHBOURS];
            edge->x = edge_id + 1;
            edge->y = sqrtf(powf(this_r-other_r, 2.0f) + powf(this_g-other_g, 2.0f) + powf(this_b-other_b, 2.0f));
            edge->z = edge_id + 1;
        }

        edge_id = prev_row;
        other_start = (x_pos - 1) * pitch + (y_pos) * CHANNEL_SIZE;
        other_r = image[other_start];
        other_g = image[other_start + 1];
        other_b = image[other_start + 2];
        edge = &edges[this_id * NUM_NEIGHBOURS + 1];
        edge->x = edge_id + 1;
        edge->y = sqrtf(powf(this_r-other_r, 2.0f) + powf(this_g-other_g, 2.0f) + powf(this_b-other_b, 2.0f));
        edge->z = edge_id + 1;

        if (!is_last_col) {
            edge_id = prev_row + 1;
            other_start = (x_pos - 1) * pitch + (y_pos + 1) * CHANNEL_SIZE;
            other_r = image[other_start];
            other_g = image[other_start + 1];
            other_b = image[other_start + 2];
            edge = &edges[this_id * NUM_NEIGHBOURS + 2];
            edge->x = edge_id + 1;
            edge->y = sqrtf(powf(this_r-other_r, 2.0f) + powf(this_g-other_g, 2.0f) + powf(this_b-other_b, 2.0f));
            edge->z = edge_id + 1;
        }
    }

    if (x_pos < x_len - 1) {
        uint next_row = this_id + y_len;
        if (!is_first_col) {
            edge_id = next_row - 1;
            other_start = (x_pos + 1) * pitch + (y_pos - 1) * CHANNEL_SIZE;
            other_r = image[other_start];
            other_g = image[other_start + 1];
            other_b = image[other_start + 2];
            edge = &edges[this_id * NUM_NEIGHBOURS + 3];
            edge->x = edge_id + 1;
            edge->y = sqrtf(powf(this_r-other_r, 2.0f) + powf(this_g-other_g, 2.0f) + powf(this_b-other_b, 2.0f));
            edge->z = edge_id + 1;
        }

        edge_id = next_row;
        other_start = (x_pos + 1) * pitch + (y_pos) * CHANNEL_SIZE;
        other_r = image[other_start];
        other_g = image[other_start + 1];
        other_b = image[other_start + 2];
        edge = &edges[this_id * NUM_NEIGHBOURS + 4];
        edge->x = edge_id + 1;
        edge->y = sqrtf(powf(this_r-other_r, 2.0f) + powf(this_g-other_g, 2.0f) + powf(this_b-other_b, 2.0f));
        edge->z = edge_id + 1;

        if (!is_last_col) {
            edge_id = next_row + 1;
            other_start = (x_pos + 1) * pitch + (y_pos + 1) * CHANNEL_SIZE;
            other_r = image[other_start];
            other_g = image[other_start + 1];
            other_b = image[other_start + 2];
            edge = &edges[this_id * NUM_NEIGHBOURS + 5];
            edge->x = edge_id + 1;
            edge->y = sqrtf(powf(this_r-other_r, 2.0f) + powf(this_g-other_g, 2.0f) + powf(this_b-other_b, 2.0f));
            edge->z = edge_id + 1;
        }
    }

    if (!is_first_col) {
        edge_id = this_id - 1;
        other_start = (x_pos) * pitch + (y_pos - 1) * CHANNEL_SIZE;
        other_r = image[other_start];
        other_g = image[other_start + 1];
        other_b = image[other_start + 2];
        edge = &edges[this_id * NUM_NEIGHBOURS + 6];
        edge->x = edge_id + 1;
        edge->y = sqrtf(powf(this_r-other_r, 2.0f) + powf(this_g-other_g, 2.0f) + powf(this_b-other_b, 2.0f));
        edge->z = edge_id + 1;
    }

    if (!is_last_col) {
        edge_id = this_id + 1;
        other_start = (x_pos) * pitch + (y_pos + 1) * CHANNEL_SIZE;
        other_r = image[other_start];
        other_g = image[other_start + 1];
        other_b = image[other_start + 2];
        edge = &edges[this_id * NUM_NEIGHBOURS + 7];
        edge->x = edge_id + 1;
        edge->y = sqrtf(powf(this_r-other_r, 2.0f) + powf(this_g-other_g, 2.0f) + powf(this_b-other_b, 2.0f));
        edge->z = edge_id + 1;
    }
}

// Kernel to decode graph
__global__
void decode(uint4 vertices[], char *image, char* colours, uint num_vertices) {
    uint pos = blockDim.x * blockIdx.x + threadIdx.x;
    if (pos >= num_vertices) return;

    uint img_pos = pos * CHANNEL_SIZE;
    uint colour_start = (vertices[pos].y - 1) * CHANNEL_SIZE;
    image[img_pos] = colours[colour_start];
    image[img_pos + 1] = colours[colour_start + 1];
    image[img_pos + 2] = colours[colour_start + 2];
}

// Kernel to find min edge
__global__
void find_min_edges(uint4 vertices[], uint3 edges[], uint3 min_edges[], uint num_components, uint vertices_length) {
    uint component_id = blockDim.x * blockIdx.x + threadIdx.x;
    if (component_id >= num_components) return;

    uint3 min;
    min.x = UINT_MAX;
    min.y = 0;
    // Scan all vertices and find the min with component == tid
    for (int i = 0; i < vertices_length; i++) {
        uint4 vertice = vertices[i];
        if (vertice.y - 1 == component_id) {
            for (int j = component_id * NUM_NEIGHBOURS; j < component_id * NUM_NEIGHBOURS + NUM_NEIGHBOURS; j++) {
                uint3 edge = edges[j];
                if (edge.x != 0) {
                    if (edge.y < min.x) {
                        min.x = edge.y;
                        min.y = vertice.x;
                        min.z = edge.x;
                    }
                }
            }
        }
    }
    min_edges[component_id] = min;
}

// Kernel to remove cycles
__global__
void remove_cycles(uint3 min_edges[], uint num_components, uint *did_change) {
    uint component_id_x = blockDim.x * blockIdx.x + threadIdx.x;
    if (component_id_x >= num_components) return;

    uint component_id_y = blockDim.y * blockIdx.y + threadIdx.y;
    if (component_id_y >= num_components) return;

    if (component_id_x == component_id_y) return;

    uint3 *edge = &min_edges[component_id_x];
    uint src = edge->y;
    uint dest = edge->z;
    if (src == dest) return;
    __syncthreads();

    uint3 *curr_edge = &min_edges[component_id_y];
    if (src == curr_edge->z) {
        if (component_id_x > component_id_y || dest != curr_edge->y) {
            curr_edge->z = dest;
            *did_change = 1;
        }
    }
}

// Kernel to update vertices with new components
__global__
void update_matrix(uint4 vertices[], uint3 edges[], uint vertices_length, uint new_component, uint new_size, uint new_int_diff, uint original_id) {
    uint vertice_id = blockDim.x * blockIdx.x + threadIdx.x;
    if (vertice_id >= vertices_length) return;

    uint4 *vertice = &vertices[vertice_id];
    bool is_vertice_new_comp = vertice->y == original_id || vertice->y == new_component;
    if (is_vertice_new_comp) {
        vertice->y = new_component;
        vertice->z = new_size;
        vertice->w = new_int_diff;
    }

    for (int j = vertice_id * NUM_NEIGHBOURS; j < vertice_id * NUM_NEIGHBOURS + NUM_NEIGHBOURS; j++) {
        uint3 *neighbour_edge = &edges[j];
        if (neighbour_edge->x != 0) {
            if (neighbour_edge->z == original_id || neighbour_edge->z == new_component) {
                if (is_vertice_new_comp) neighbour_edge->x = 0; // Remove internal edges
                else neighbour_edge->z = new_component;
            }
        }
    }

}

// Kernel to merge components
__global__
void merge(uint4 vertices[], uint3 edges[], uint3 min_edges[], uint *num_components, uint update_threads, uint update_blocks, uint vertices_length, uint comp_count) {
    uint component_id = blockDim.x * blockIdx.x + threadIdx.x;
    if (component_id >= comp_count)  {
        return;
    }

    uint3 min_edge = min_edges[component_id];
    if (min_edge.y == min_edge.z || min_edge.y == 0) return;
    uint4 src = vertices[min_edge.y - 1];
    uint4 dest = vertices[min_edge.z - 1];
    __syncthreads();
    uint src_diff = src.w + (K / src.z);
    uint dest_diff = dest.w + (K / dest.z);
    if (min_edge.x <= min(src_diff, dest_diff)) {
        atomicSub_system(num_components, 1); // Is this horribly inefficient?
        uint new_int_diff = max(max(dest.w, src.w), min_edge.x);
        uint new_size = src.z + dest.z;
        uint new_component = dest.y;

        update_matrix<<<update_blocks, update_threads>>>(vertices, edges, vertices_length, new_component, new_size, new_int_diff, src.y);
        hipDeviceSynchronize();
    }
}

__global__
void debug_print_min_edges(uint3 min_edges[], uint length) {
    for (int i = 0; i < length; i++) {
        printf("(%d %d)\n", min_edges[i].y, min_edges[i].z);
    }
    printf("\n");
}

// Kernel to orchestrate
__global__
void segment(uint4 vertices[], uint3 edges[], uint3 min_edges[], uint *n_components, uint *did_change) {
    uint prev_n_components = 0;
    uint n_vertices = *n_components;
    uint curr_n_comp = *n_components;
    dim3 threads;
    dim3 blocks;
    dim3 cycle_blocks;
    dim3 cycle_threads;
    if (n_vertices < 1024) {
        threads.y = n_vertices;
        blocks.y = 1;
    } else {
        threads.y = 1024;
        blocks.y = n_vertices / 1024 + 1;
    }

    while (curr_n_comp != prev_n_components) {
        if (curr_n_comp < 1024) {
            threads.x = curr_n_comp;
            blocks.x = 1;
        } else {
            threads.x = 1024;
            blocks.x = curr_n_comp / 1024 + 1;
        }

        if (curr_n_comp < 32) {
            cycle_threads.x = curr_n_comp;
            cycle_threads.y = cycle_threads.x;

            cycle_blocks.x = 1;
            cycle_blocks.y = 1;
        } else {
            cycle_threads.x = 32;
            cycle_blocks.x = curr_n_comp / 32 + 1;

            cycle_threads.y = cycle_threads.x;
            cycle_blocks.y = cycle_blocks.x;
        }

        find_min_edges<<<blocks.x, threads.x>>>(vertices, edges, min_edges, curr_n_comp, n_vertices);
        hipDeviceSynchronize();

        *did_change = 1;
        while (*did_change == 1) {
            *did_change = 0;
            remove_cycles<<<cycle_blocks, cycle_threads>>>(min_edges, curr_n_comp, did_change);
            hipDeviceSynchronize();
        }

        merge<<<blocks.x, threads.x>>>(vertices, edges, min_edges, n_components, threads.y, blocks.y, n_vertices, curr_n_comp);
        hipDeviceSynchronize();

        prev_n_components = curr_n_comp;
        curr_n_comp = *n_components;
        printf("N components: %d\n", curr_n_comp);
    }
}

__global__
void debug_print_vertices(uint4 vertices[], uint length, uint3 edges[]) {
    for (int v_id = 0; v_id < length; v_id++) {
        printf("vertices[%d] = %d %d | ", v_id, vertices[v_id].x, vertices[v_id].y);
        for (int j = v_id * NUM_NEIGHBOURS; j < v_id * NUM_NEIGHBOURS + NUM_NEIGHBOURS; j++) {
            printf("%d(%d), ", edges[j].x, edges[j].y);
        }
        printf("\n");
    }
}

void get_component_colours(char colours[], uint num_colours) {
    srand(123456789);
    for (int i = 0; i < num_colours * CHANNEL_SIZE; i++) {
        colours[i] = rand() % 256;
    }
}

void checkErrors(const char *identifier) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << " " << identifier << std::endl;
}

char *compute_segments(void *input, uint x, uint y, size_t pitch) {
    uint4 *vertices;
    uint3 *edges;
    uint3 *min_edges;
    uint num_vertices = (x) * (y);
    uint *num_components;
    uint *did_change;

    hipMalloc(&vertices, num_vertices*sizeof(uint4));
    checkErrors("Malloc vertices");
    hipMalloc(&edges, num_vertices*NUM_NEIGHBOURS*sizeof(uint3));
    checkErrors("Malloc edges");
    hipMalloc(&min_edges, num_vertices*sizeof(uint3)); // max(min_edges) == vertices.length
    checkErrors("Malloc min_edges");
    hipMalloc(&num_components, sizeof(uint));
    checkErrors("Malloc num components");
    hipMalloc(&did_change, sizeof(uint));
    checkErrors("Malloc did change");

    hipMemcpyAsync(num_components, &num_vertices, sizeof(uint), hipMemcpyHostToDevice);
    checkErrors("Memcpy num_vertices");

    // Write to the matrix from image
    // cudaOccupancyScheduler?
    dim3 encode_threads;
    dim3 encode_blocks;
    if (num_vertices < 1024) {
        encode_threads.x = x;
        encode_threads.y = y;
        encode_blocks.x = 1;
        encode_blocks.y = 1;
    } else {
        encode_threads.x = 32;
        encode_threads.y = 32;
        encode_blocks.x = x / 32 + 1;
        encode_blocks.y = y / 32 + 1;
    }

    encode<<<encode_blocks, encode_threads>>>((u_char*)input, vertices, edges, x, y, pitch);
    checkErrors("encode()");

    // Segment matrix
    segment<<<1, 1>>>(vertices, edges, min_edges, num_components, did_change);
    hipDeviceSynchronize();
    checkErrors("segment()");

    // Setup random colours for components
    dim3 decode_threads;
    dim3 decode_blocks;
    if (num_vertices <= 1024) {
        decode_threads.x = num_vertices;
        decode_blocks.x = 1;
    } else {
        decode_threads.x = 1024;
        decode_blocks.x = num_vertices / 1024 + 1;
    }

    char component_colours[num_vertices * CHANNEL_SIZE];
    get_component_colours(component_colours, num_vertices);
    char *component_colours_dev;
    hipMalloc(&component_colours_dev, num_vertices * CHANNEL_SIZE * sizeof(char));
    hipMemcpyAsync(component_colours_dev, component_colours, num_vertices * CHANNEL_SIZE * sizeof(char), hipMemcpyHostToDevice);
    char *output_dev;
    hipMalloc(&output_dev, num_vertices * CHANNEL_SIZE * sizeof(char ));

    // Write image back from segmented matrix
    decode<<<decode_blocks, decode_threads>>>(vertices, output_dev, component_colours_dev, num_vertices);
    hipDeviceSynchronize();
    checkErrors("decode()");

    // Clean up matrix
    hipFree(vertices);
    checkErrors("Free vertices");
    hipFree(edges);
    checkErrors("Free edges");
    hipFree(min_edges);
    checkErrors("Free min_edges");
    hipFree(component_colours_dev);
    checkErrors("Free component_colours_dev");

    //Copy image data back from GPU
    char *output = (char*) malloc(x*y*CHANNEL_SIZE*sizeof(char));

    hipMemcpy(output, output_dev, x*y*CHANNEL_SIZE*sizeof(char), hipMemcpyDeviceToHost);
    checkErrors("Memcpy output");

    return output;
}
