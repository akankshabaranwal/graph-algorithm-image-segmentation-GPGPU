#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <opencv2/cudafilters.hpp>
#include <opencv2/highgui.hpp>
#include "CreateGraph.h"
#include "moderngpu.cuh"		// Include all MGPU kernels.
#include "FastMST.h"
#include "RecolorImage.h"

using namespace cv;
using namespace cv::cuda;
using namespace mgpu;
// TODO: Add the error handling code from:
//  http://cuda-programming.blogspot.com/2013/01/vector-addition-in-cuda-cuda-cc-program.html

uint64_t mask_32 = 0x00000000FFFFFFFF;//32 bit mask
uint64_t mask_22 = 0x000003FFFFF;//32 bit mask
uint64_t mask_20 = 0x000000FFFFF;//32 bit mask

void segment(Mat image, int argc, char **argv)
{
    Mat output;
    GpuMat dev_image, dev_output;
    dev_image.upload(image);

    Ptr<Filter> filter = createGaussianFilter(CV_8UC3, CV_8UC3, Size(5, 5), 1.0);
    filter->apply(dev_image, dev_output);

    //Graph parameters
    int numVertices = image.rows * image.cols;
    uint numEdges = (image.rows) * (image.cols) * 4;

    //Convert image to graph
    uint32_t *VertexList, *OnlyEdge, *FlagList, *NWE, *Successor, *newSuccessor, *L, *Representative, *VertexIds;
    uint64_t *OnlyWeight, *tempArray, *BitEdgeList, *MinSegmentedList;
    uint32_t *MinMaxScanArray;
    uint32_t *new_E_size, *new_V_size;
    uint32_t *compactLocations, *expanded_u;
    uint32_t *C;
    edge *EdgeList;
    uint *flag;
    uint *Flag2;
    uint32_t *SuperVertexId;
    //Allocating memory
    hipMallocManaged(&flag, numEdges * sizeof(uint32_t));
    hipMallocManaged(&VertexList, numVertices * sizeof(uint32_t));
    hipMallocManaged(&FlagList, numVertices * sizeof(uint32_t));
    hipMallocManaged(&MinSegmentedList, numVertices * sizeof(uint64_t));
    hipMallocManaged(&tempArray, numVertices * sizeof(uint64_t));
    hipMallocManaged(&EdgeList, numEdges * sizeof(edge));
    hipMallocManaged(&BitEdgeList, numEdges * sizeof(uint64_t));
    hipMallocManaged(&NWE, numVertices * sizeof(uint32_t));
    hipMallocManaged(&Successor, numVertices * sizeof(uint32_t));
    hipMallocManaged(&newSuccessor, numVertices * sizeof(uint32_t));
    hipMallocManaged(&OnlyEdge, numEdges * sizeof(uint32_t));
    hipMallocManaged(&OnlyWeight, numEdges * sizeof(uint64_t));
    hipMallocManaged(&L, numVertices * sizeof(uint32_t));
    hipMallocManaged(&Representative, numVertices * sizeof(uint32_t));
    hipMallocManaged(&VertexIds, numVertices * sizeof(uint32_t));
    hipMallocManaged(&new_E_size, numEdges * sizeof(uint32_t));
    hipMallocManaged(&new_V_size, numEdges * sizeof(uint32_t));
    hipMallocManaged(&MinMaxScanArray, numEdges * sizeof(uint32_t));
    hipMallocManaged(&compactLocations, numEdges * sizeof(uint32_t));
    hipMallocManaged(&expanded_u, numEdges * sizeof(uint32_t));
    hipMallocManaged(&C, numEdges * sizeof(uint32_t));
    hipMallocManaged(&Flag2, numEdges * sizeof(uint32_t));

    hipMallocManaged(&SuperVertexId, numVertices * sizeof(uint32_t));

    uint *uid;
    hipMallocManaged(&uid, numVertices * sizeof(uint32_t));
    dim3 threadsPerBlock(1024, 1024);
    uint BlockX = image.rows / threadsPerBlock.x;
    uint BlockY = image.cols / threadsPerBlock.y;
    dim3 numBlocks(BlockX, BlockY);
    hipDeviceSynchronize(); //FIXME: Need to check where all this synchronize call is needed
    ContextPtr context = CreateCudaDevice(argc, argv, true);
    hipError_t err = hipGetLastError();

    uint *flag4; //Same as F4. New flag for creating vertex list. Assigning the new ids.
    hipMallocManaged(&flag4,numEdges * sizeof(uint));

    bool *change;
    hipMallocManaged(&change, sizeof(bool));

    dev_output.download(output);

    uint32_t tmp_V;
    uint64_t tmp_Wt;

    uint numthreads;
    uint numBlock;

    uint64_t *UV, *UVW;
    uint32_t *W;

    hipMallocManaged(&UV,numEdges*sizeof(uint64_t));
    hipMallocManaged(&UVW,numEdges*sizeof(uint64_t));
    hipMallocManaged(&W,numEdges*sizeof(uint32_t));

    uint *flag3;
    hipMallocManaged(&flag3,numEdges*sizeof(uint));
    uint *Flag4;
    hipMallocManaged(&Flag4,numEdges*sizeof(uint));

    numEdges = ImagetoGraphSerial(image, EdgeList, VertexList, BitEdgeList);


//    printf("\nEdge\n");
    for (uint32_t i = 0; i < numEdges; i++)
    {
        tmp_V = BitEdgeList[i] & mask_32;
        tmp_Wt = BitEdgeList[i]>>32;
        OnlyEdge[i] = tmp_V;
        OnlyWeight[i] = (tmp_Wt<<32) | i;

        if (tmp_V != EdgeList[i].Vertex)
        {    printf("ERROR!!!");
            exit(-1);
        }
        if (tmp_Wt != EdgeList[i].Weight)
        {    printf("ERROR!!!");
            exit(-1);
        }
    }

    std::vector<uint32_t*> d_hierarchy_levels;	// Vector containing pointers to all hierarchy levels (don't dereference on CPU, device pointers)
    std::vector<int> hierarchy_level_sizes;			// Size of each hierarchy level

    while(numVertices>1)
    {
        if(numVertices>1024)
            numthreads = 1024;
        else if(numVertices>512)
            numthreads = 512;
        else if(numVertices>256)
            numthreads = 256;
        else if(numVertices>128)
            numthreads = 128;
        else if(numVertices>64)
            numthreads = 64;
        else
            numthreads = min(32, numVertices);

        numBlock = numVertices/numthreads;

        //1. The graph creation step above takes care of this
        SetOnlyWeightArray<<<numBlock, numthreads>>>(BitEdgeList, OnlyWeight, numEdges);
        hipError_t err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error: SetOnlyWeightArray%s\n", hipGetErrorString(err));
            exit(-1);
        }

        ClearFlagArray<<<numBlock, numthreads>>>(flag, numEdges);
        err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error: Flag Array%s\n", hipGetErrorString(err));
            exit(-1);
        }

        MarkSegments<<<numBlock, numthreads>>>(flag, VertexList, numEdges);
        //3. Segmented min scan
        SegmentedReduction(*context, VertexList, OnlyWeight, tempArray, numEdges, numVertices);
        err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error: Segment Reduction%s\n", hipGetErrorString(err));
            exit(-1);
        }
        // Create NWE array
        CreateNWEArray<<<numBlock, numthreads>>>(NWE, tempArray, numVertices);
        err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error: CreateNWEArray %s\n", hipGetErrorString(err));
            exit(-1);
        }

        //4. Find Successor array of each vertex
        FindSuccessorArray<<<numBlock, numthreads>>>(Successor, BitEdgeList, NWE, numVertices);
        err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error: FindSuccessorArray %s\n", hipGetErrorString(err));
            exit(-1);
        }

        RemoveCycles<<<numBlock, numthreads>>>(Successor, numVertices);
        err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error RemoveCycles: %s\n", hipGetErrorString(err));
            exit(-1);
        }
        hipDeviceSynchronize();

        //C. Merging vertices and assigning IDs to supervertices
        //7. Propagate representative vertex IDs using pointer doubling

        PropagateRepresentativeVertices(Successor, numVertices);

        //8, 9 Append appendSuccessorArray
        appendSuccessorArray<<<numBlock, numthreads>>>(Representative, VertexIds, Successor, numVertices);
        err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error: AppendSuccessorArray %s\n", hipGetErrorString(err));
            exit(-1);
            exit(-1);
        }
        thrust::sort_by_key(thrust::device, Representative, Representative + numVertices, VertexIds);

        CreateFlag2Array<<<numBlock, numthreads>>>(Representative, Flag2, numVertices);
        hipDeviceSynchronize();

        thrust::inclusive_scan(Flag2, Flag2 + numVertices, C, thrust::plus<int>());

        //D. Finding the Supervertex ids and storing it in an array
        CreateSuperVertexArray<<<numBlock,numthreads>>>(SuperVertexId, VertexIds, C, numVertices);
        err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error CreateSuperVertexArray: %s\n", hipGetErrorString(err));
            exit(-1);
        }
        hipDeviceSynchronize();

        //Create UID array. 10.2
        CreateUid(uid, flag, numEdges);

        //11. Removing self edges
        RemoveSelfEdges<<<numBlock,numthreads>>>(OnlyEdge, numEdges, uid, SuperVertexId);
        err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error RemoveSelfEdges: %s\n", hipGetErrorString(err));
            exit(-1);
        }

        //E 12.
        CreateUVWArray<<<numBlock,numthreads>>>(BitEdgeList, OnlyEdge, numEdges, uid, SuperVertexId, UV, W, UVW);
        err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error CreateUVWArray: %s\n", hipGetErrorString(err));
            exit(-1);
        }

        //12.2 Sort the UVW Array
        thrust::sort_by_key(thrust::device, UV, UV + numEdges, W);
        thrust::sort_by_key(thrust::device, UVW, UVW + numEdges, W);

        flag3[0]=1;
        CreateFlag3Array<<<numBlock,numthreads>>>(UV, W, numEdges, flag3, MinMaxScanArray);

        uint32_t *new_edge_size = thrust::max_element(thrust::device, MinMaxScanArray, MinMaxScanArray + numEdges);
        hipDeviceSynchronize();
        *new_edge_size = *new_edge_size+1;
        thrust::inclusive_scan(flag3, flag3 + *new_edge_size, compactLocations, thrust::plus<int>());

        ResetCompactLocationsArray<<<numBlock,numthreads>>>(compactLocations, *new_edge_size);
        CreateNewEdgeList<<<numBlock,numthreads>>>( BitEdgeList, compactLocations, OnlyEdge, OnlyWeight, UV, W, UVW, flag3, *new_edge_size, new_E_size, new_V_size, expanded_u);

        uint32_t *new_E_sizeptr = thrust::max_element(thrust::device, new_E_size, new_E_size + *new_edge_size);
        uint32_t *new_V_sizeptr = thrust::max_element(thrust::device, new_V_size, new_V_size + *new_edge_size);
        numVertices = *new_V_sizeptr;
        numEdges = *new_E_sizeptr;

        Flag4[0]=1;
        CreateFlag4Array<<<numBlock,numthreads>>>(expanded_u, Flag4, numEdges);
        CreateNewVertexList<<<numBlock,numthreads>>>(VertexList, Flag4, numEdges, expanded_u);
        err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error: CreateNewVertexList%s\n", hipGetErrorString(err));
            exit(-1);
        }
        hipDeviceSynchronize();
        d_hierarchy_levels.push_back(SuperVertexId);
        hierarchy_level_sizes.push_back(numVertices);
        hipMallocManaged(&SuperVertexId, numVertices * sizeof(uint32_t));
    }
    std::string outFile="test";
    writeComponents(d_hierarchy_levels, image.rows*image.cols, 3, hierarchy_level_sizes, outFile, image.rows, image.cols);
}


int main(int argc, char **argv)
{
    Mat image;


    image = imread("data/bear.jpg", IMREAD_COLOR);

    printf("Size of image obtained is: Rows: %d, Columns: %d, Pixels: %d\n", image.rows, image.cols, image.rows * image.cols);
    segment(image, argc, argv);

    return 0;
}
