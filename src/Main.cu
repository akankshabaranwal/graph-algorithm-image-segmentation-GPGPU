#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <opencv2/cudafilters.hpp>
#include <opencv2/highgui.hpp>
#include "CreateGraph.h"
#include "moderngpu.cuh"		// Include all MGPU kernels.
#include "FastMST.h"

using namespace cv;
using namespace cv::cuda;
using namespace mgpu;
// TODO: Add the error handling code from:
//  http://cuda-programming.blogspot.com/2013/01/vector-addition-in-cuda-cuda-cc-program.html
int main(int argc, char **argv)
{
    Mat image, output;
    GpuMat dev_image, dev_output;

    image = imread("data/beach.png", IMREAD_COLOR);
    int scale_percent = 10; // percent of original size
    cv::resize(image, image, cv::Size(), 0.05, 0.05);//Debugging using beach scaled down by 0.05, 0.05

    printf("Size of image obtained is: Rows: %d, Columns: %d, Pixels: %d\n", image.rows, image.cols, image.rows * image.cols);

    //TODO: Add checker for image size depending on the bits decided for representing edge weight and vertex index
    dev_image.upload(image);

    Ptr<Filter> filter = createGaussianFilter(CV_8UC3, CV_8UC3, Size(5, 5), 1.0);
    filter->apply(dev_image, dev_output);

    //Graph parameters
    int numVertices = image.rows * image.cols;
    int numEdges = (image.rows) * (image.cols) * 4;

    //Convert image to graph
    int32_t *VertexList, *OnlyEdge, *OnlyVertex, *OnlyWeight, *BitEdgeList, *FlagList, *MinSegmentedList, *tempArray, *NWE, *Successor, *newSuccessor, *L, *Representative, *VertexIds;
    int32_t *MinMaxScanArray;
    int32_t *new_E_size, *new_V_size;
    int32_t *compactLocations, *expanded_u;

    edge *EdgeList;

    int *flag;
    hipMallocManaged(&flag, numEdges * sizeof(int32_t));
    //Allocating memory
    hipMallocManaged(&VertexList, numVertices * sizeof(int32_t));
    hipMallocManaged(&FlagList, numVertices * sizeof(int32_t));
    hipMallocManaged(&MinSegmentedList, numVertices * sizeof(int32_t));
    hipMallocManaged(&tempArray, numVertices * sizeof(int32_t));
    hipMallocManaged(&EdgeList, numEdges * sizeof(edge));
    hipMallocManaged(&BitEdgeList, numEdges * sizeof(int32_t));
    hipMallocManaged(&NWE, numVertices * sizeof(int32_t));
    hipMallocManaged(&Successor, numVertices * sizeof(int32_t));
    hipMallocManaged(&newSuccessor, numVertices * sizeof(int32_t));

    hipMallocManaged(&OnlyEdge, numEdges * sizeof(int32_t));
    hipMallocManaged(&OnlyVertex, numEdges * sizeof(int32_t));
    hipMallocManaged(&OnlyWeight, numEdges * sizeof(int32_t));

    hipMallocManaged(&L, numVertices * sizeof(int32_t));
    hipMallocManaged(&Representative, numVertices * sizeof(int32_t));
    hipMallocManaged(&VertexIds, numVertices * sizeof(int32_t));
    hipMallocManaged(&new_E_size, numEdges * sizeof(int32_t));
    hipMallocManaged(&new_V_size, numEdges * sizeof(int32_t));
    hipMallocManaged(&MinMaxScanArray, numEdges * sizeof(int32_t));
    hipMallocManaged(&compactLocations, numEdges * sizeof(int32_t));
    hipMallocManaged(&expanded_u, numEdges * sizeof(int32_t));


    int *Flag2;
    hipMallocManaged(&Flag2, numEdges * sizeof(int32_t));
    int *SuperVertexId;
    hipMallocManaged(&SuperVertexId, numVertices * sizeof(int32_t));

    int *uid;
    hipMallocManaged(&uid, numVertices * sizeof(int32_t));
    dim3 threadsPerBlock(32, 32);
    int BlockX = image.rows / threadsPerBlock.x;
    int BlockY = image.cols / threadsPerBlock.y;
    dim3 numBlocks(BlockX, BlockY);
    hipDeviceSynchronize(); //FIXME: Need to check where all this synchronize call is needed
    ContextPtr context = CreateCudaDevice(argc, argv, true);
    hipError_t err = hipGetLastError();

    int *flag4; //Same as F4. New flag for creating vertex list. Assigning the new ids.
    hipMallocManaged(&flag4, numEdges * sizeof(int));

    bool *change;
    hipMallocManaged(&change, sizeof(bool));
    //FIXME: Make this initialization run in parallel?
    //TODO: Figure out if this initialization is required??
    for (int i = 0; i < numEdges; i++)
    {
        EdgeList[i].Weight = 0;
    }

    dev_output.download(output);

    int32_t tmp_V, tmp_Wt;

    int numthreads = 32;
    int numBlock = numVertices/numthreads;

    int *UV, *W;
    hipMallocManaged(&UV,numEdges*sizeof(int64_t));
    hipMallocManaged(&W,numEdges*sizeof(int64_t));
    int32_t *flag3;
    hipMallocManaged(&flag3,numEdges*sizeof(int32_t));
    int32_t *Flag4;
    hipMallocManaged(&Flag4,numEdges*sizeof(int32_t));

    numEdges = ImagetoGraphSerial(image, EdgeList, VertexList, BitEdgeList);

    /**** MST Starts ****/
    bool DidReduce; //Boolean to check if more segments got created or not
    DidReduce = 1;

    printf("Vertex\n");
    for (int i = 0; i < numVertices; i++)
    {
        tmp_V = VertexList[i];
        printf("%d, ", tmp_V);
        OnlyVertex[i]=tmp_V;
    }

    printf("\nEdge\n");
    for (int i = 0; i < numEdges; i++)
    {
        tmp_V = BitEdgeList[i] % (2 << 15);
        printf("%d, ", tmp_V);
        OnlyEdge[i] = tmp_V;
        if (tmp_V != EdgeList[i].Vertex)
        {    printf("ERROR!!!");
            exit(-1);
        }
    }

    printf("\nWeight\n");
    for (int32_t i = 0; i < numEdges; i++)
    {
        tmp_Wt = BitEdgeList[i]>>16;
        printf("%d, ", tmp_Wt);
        OnlyWeight[i]=(tmp_Wt * (2<<15)) + i;
    }

    while(numVertices>1)
    {
        printf("\n*****************\n");
        printf("\n*****************\n");

        printf("\nStarting new iteration\n");
        if(numVertices>1024)
        numthreads = min(1024,numVertices);
        else if(numVertices>512)
            numthreads = min(512,numVertices);
        else if(numVertices>256)
            numthreads = min(256,numVertices);
        else if(numVertices>128)
            numthreads = min(128,numVertices);
        else if(numVertices>64)
            numthreads = min(64,numVertices);
        else
            numthreads = min(32,numVertices);

        numBlock = numVertices/numthreads;

        //1. The graph creation step above takes care of this

        //2. Mark the segments in the flag array. Being used for the uid array below
        ClearFlagArray<<<numBlock, numthreads>>>(flag, numEdges);

        hipError_t err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error: Flag Array%s\n", hipGetErrorString(err));
            exit(-1);
        }

        MarkSegments<<<numBlock, numthreads>>>(flag, VertexList, numEdges);

        err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error: Mark Segments%s\n", hipGetErrorString(err));
            exit(-1);
        }
        printf("\n Printing Only Weight Array Indices before minsegment: \n");
        for(int i =0; i< numVertices; i++)
        {
            printf("%d ,", OnlyWeight[i]%(2<<15));
        }
        printf("\n Printing BitEdgeList Array before minsegment: \n");
        for(int i =0; i< numEdges; i++)
        {
            printf("%d ,", BitEdgeList[i]);
        }
        //3. Segmented min scan
        SegmentedReduction(*context, VertexList, BitEdgeList, MinSegmentedList, numEdges, numVertices);
        SegmentedReduction(*context, VertexList, OnlyWeight, tempArray, numEdges, numVertices);

        err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error: Segment Reduction%s\n", hipGetErrorString(err));
            exit(-1);
        }
        hipDeviceSynchronize();
        printf("\n Printing MinSegment Array Values: \n");
        for(int i =0; i< numVertices; i++)
        {
            printf("%d ,", MinSegmentedList[i]);
        }
        printf("\n Printing MinSegment Array Weights: \n");
        for(int i =0; i< numVertices; i++)
        {
            printf("%d ,", MinSegmentedList[i]>>16);
        }
        printf("\n Printing MinSegment Array Indices: \n");
        for(int i =0; i< numVertices; i++)
        {
            printf("%d ,", MinSegmentedList[i]%(2<<15));
        }
        printf("\n Printing Only Weight Array Indices: \n");
        for(int i =0; i< numVertices; i++)
        {
            printf("%d ,", tempArray[i]%(2<<15));
        }
        // Create NWE array
        CreateNWEArray<<<numBlock, numthreads>>>(NWE, tempArray, numVertices);
        err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error: CreateNWEArray %s\n", hipGetErrorString(err));
            exit(-1);
        }
        hipDeviceSynchronize();
        printf("\n Printing NWE Array: \n");
        for(int i =0; i< numVertices; i++)
        {
            printf("%d ,", NWE[i]);
        }
        //4. Find Successor array of each vertex
        FindSuccessorArray<<<numBlock, numthreads>>>(Successor, BitEdgeList, NWE, numVertices);
        err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error: FindSuccessorArray %s\n", hipGetErrorString(err));
            exit(-1);
        }
        hipDeviceSynchronize();

        printf("\nPrinting Successor Array: \n");
        for(int i =0; i< numVertices; i++)
        {
            printf("%d ,", Successor[i]);
        }

        RemoveCycles<<<numBlock, numthreads>>>(Successor, numVertices);
        err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error RemoveCycles: %s\n", hipGetErrorString(err));
            exit(-1);
        }
        hipDeviceSynchronize();

        printf("\n After removing cycles printing Successor Array: \n");
        for(int i =0; i< numVertices; i++)
        {
            printf("%d ,", Successor[i]);
        }

        //C. Merging vertices and assigning IDs to supervertices
        //7. Propagate representative vertex IDs using pointer doubling
        hipDeviceSynchronize(); //because PropagateRepresentative is on host

        PropagateRepresentativeVertices(Successor, numVertices);

        hipDeviceSynchronize();
        printf("\n After propagating representative vertices printing Successor Array: \n");
        for(int i =0; i< numVertices; i++)
        {
            if(Successor[i]!=18145)
          printf("%d ,", Successor[i]);
        }

        //8, 9 Append appendSuccessorArray
        appendSuccessorArray<<<numBlock, numthreads>>>(Representative, VertexIds, Successor, numVertices);
        err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error: AppendSuccessorArray %s\n", hipGetErrorString(err));
            exit(-1);
            exit(-1);
        }
        hipDeviceSynchronize();

        printf("\n Representative array \n");
        for(int i =0; i< numVertices; i++)
        {
            printf("%d ,", Representative[i]);
        }

        printf("\n Vertex \n");
        for(int i =0; i< numVertices; i++)
        {
            printf("%d ,", VertexIds[i]);
        }
        //hipDeviceSynchronize();
        //9. Create F2, Assign new IDs based on Flag2 array
        hipDeviceSynchronize();
        //SortedSplit(Representative, VertexIds, Successor, Flag2, numVertices);
        thrust::sort_by_key(thrust::device, Representative, Representative + numVertices, VertexIds);
        CreateFlag2Array<<<numBlock, numthreads>>>(Representative, Flag2, numVertices);
        thrust::inclusive_scan(Flag2, Flag2 + numVertices, Flag2, thrust::plus<int>());

        hipDeviceSynchronize();

        printf("\n Sorted representative array \n");
        for(int i =0; i< numVertices; i++)
        {
            printf("%d ,", Representative[i]);
        }

        printf("\n Sorted Vertex Labels \n");
        for(int i =0; i< numVertices; i++)
        {
            printf("%d ,", VertexIds[i]);
        }

        printf("Flag\n");
        for(int i =0; i< numVertices; i++)
        {
            printf("%d ,", Flag2[i]);
        }

        //D. Finding the Supervertex ids and storing it in an array
        CreateSuperVertexArray<<<numBlock,numthreads>>>(SuperVertexId, VertexIds, Flag2, numVertices);
        err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error CreateSuperVertexArray: %s\n", hipGetErrorString(err));
            exit(-1);
        }
        hipDeviceSynchronize();
       printf("\n SuperVertexIds\n");
        for(int i =0; i< numVertices; i++)
        {
            printf("%d ,", SuperVertexId[i]);
        }

        //Create UID array. 10.2
        CreateUid(uid, flag, numEdges); //Isnt this same as the vertex list??
        hipDeviceSynchronize();

        printf("\n Uid\n");
        for(int i =0; i< numEdges; i++)
        {
            printf("%d ,", uid[i]);
        }
        printf("\nPrinting Only Edge Array before self edges: \n");
        for (int i = 0; i < numEdges; i++)
        {
            printf("%d, ", OnlyEdge[i]);
        }
        //11. Removing self edges
        RemoveSelfEdges<<<numBlock,numthreads>>>(OnlyEdge, numEdges, uid, SuperVertexId);
        err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error RemoveSelfEdges: %s\n", hipGetErrorString(err));
            exit(-1);
        }
        hipDeviceSynchronize();

        printf("\n SuperVertex\n");
        for(int i =0; i< numVertices; i++)
        {
            printf("%d ,", SuperVertexId[i]);
        }
        printf("\nPrinting only Edge Array after marked for removal: \n");
        for (int i = 0; i < numEdges; i++)
        {
            if(OnlyEdge[i]!=INT_MAX)
            printf("%d, ", OnlyEdge[i]);
        }
        //E 12.
        CreateUVWArray<<<numBlock,numthreads>>>(BitEdgeList, OnlyEdge, numEdges, uid, SuperVertexId, UV, W);
        err = hipGetLastError();        // Get error code
        if ( err != hipSuccess )
        {
            printf("CUDA Error CreateUVWArray: %s\n", hipGetErrorString(err));
            exit(-1);
        }
        hipDeviceSynchronize();
        printf("\n Printing UVW array: before calling SortUVW");
        for(int i = 0; i< numEdges;i++)
        {
            printf("%d %d %d , ", UV[i]>>16, UV[i]%(2<<15), W[i]);
        }
        printf("\n");
        //12.2 Sort the UVW Array
        thrust::sort_by_key(thrust::device, UV, UV + numEdges, W);
        hipDeviceSynchronize();
        printf("\n Printing UVW array after SortUVW: ");
        for(int i = 0; i< numEdges;i++)
        {
            printf("%d %d %d , ", UV[i]>>16, UV[i]%(2<<15), W[i]);
        }
        printf("\n");
        flag3[0]=1;
        CreateFlag3Array<<<numBlock,numthreads>>>(UV, W, numEdges, flag3, MinMaxScanArray);
        int *new_edge_size = thrust::max_element(thrust::device, MinMaxScanArray, MinMaxScanArray + numEdges);
        hipDeviceSynchronize();
        //*new_edge_size = *new_edge_size+1;
        printf("\nnew_edge_size %d", *new_edge_size);

        thrust::inclusive_scan(flag3, flag3 + *new_edge_size, compactLocations, thrust::plus<int>());
        hipDeviceSynchronize();
        printf("\n Printing compact locations array before subtract\n");
        for(int i = 0; i< *new_edge_size;i++)
        {
            printf("%d, ", compactLocations[i]);
        }
        printf("\n");
        ResetCompactLocationsArray<<<numBlock,numthreads>>>(compactLocations, *new_edge_size);
        hipDeviceSynchronize();
        printf("\nPrinting inputs for CreatNewEdgeList\n");

        printf("\n Printing flag3 array\n");
        for(int i = 0; i< *new_edge_size;i++)
        {
            printf("%d, ", flag3[i]);
        }

        printf("\n Printing compact locations array\n");
        for(int i = 0; i< *new_edge_size;i++)
        {
            printf("%d, ", compactLocations[i]);
        }
        printf("\n");
        printf("\n Printing UVW array\n");

        for(int i=0; i< *new_edge_size; i++)
        {
            printf("%d %d %d, ", UV[i]>>16, UV[i]%(2<<15), W[i]);
        }
        printf("\n");
        CreateNewEdgeList<<<numBlock,numthreads>>>( BitEdgeList, compactLocations, OnlyEdge, OnlyWeight, UV, W, flag3, *new_edge_size, new_E_size, new_V_size, expanded_u);
        int *new_E_sizeptr = thrust::max_element(thrust::device, new_E_size, new_E_size + *new_edge_size);
        int *new_V_sizeptr = thrust::max_element(thrust::device, new_V_size, new_V_size + *new_edge_size);
        hipDeviceSynchronize();
        numVertices = *new_V_sizeptr;
        numEdges = *new_E_sizeptr;
        hipDeviceSynchronize();
        printf("\nAfter CreateNewEdgeList\n");

        printf("\nPrinting E\n");
        for(int i=0; i< numEdges;i++)
            printf("%d, ", OnlyEdge[i]);

        printf("\nPrinting W\n");
        for(int i=0; i< numEdges;i++)
            printf("%d, ", OnlyWeight[i]);

        printf("\nPrinting expanded_u\n");
        for(int i=0; i< numEdges;i++)
            printf("%d, ", expanded_u[i]);

        Flag4[0]=1;
        CreateFlag4Array<<<numBlock,numthreads>>>(expanded_u, Flag4, numEdges);

        hipDeviceSynchronize();

        printf("\nPrinting expanded_u\n");
        for(int i=0; i<numEdges; i++)
                printf("%d, ", expanded_u[i]);

        printf("\nPrinting Flag4\n");
        for(int i=0; i<numEdges; i++)
            printf("%d, ", Flag4[i]);

        CreateNewVertexList<<<numBlock,numthreads>>>(OnlyVertex, Flag4, numEdges, expanded_u);

        hipDeviceSynchronize();

        printf("\n numVertices: %d numEdges %d", numVertices, numEdges);

        printf("\nPrinting new Vertex List:\n");
        for(int i=0;i<numVertices;i++)
            printf("%d, ", OnlyVertex[i]);

        printf("\nPrinting new Edge List:\n");
        for(int i=0;i<numEdges;i++)
            printf("%d, ", OnlyEdge[i]);

        printf("\nPrinting new Weight List:\n");
        for(int i=0;i<numEdges;i++)
            printf("%d, ", OnlyWeight[i]);

        numVertices=1;
    }

    return 0;
}
