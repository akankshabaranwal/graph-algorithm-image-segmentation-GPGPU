#include <iostream>
#include <hip/hip_runtime_api.h>
#include <opencv2/cudafilters.hpp>
#include <opencv2/highgui.hpp>
#include "CreateGraph.h"
#include "moderngpu.cuh"		// Include all MGPU kernels.
#include "FastMST.h"

using namespace cv;
using namespace cv::cuda;
using namespace mgpu;

int main(int argc, char **argv)
{
    Mat image, output;
    GpuMat dev_image, dev_output;

    image = imread("data/beach.png", IMREAD_COLOR);
    printf("Size of image obtained is: Rows: %d, Columns: %d, Pixels: %d\n", image.rows, image.cols, image.rows * image.cols);
    //TODO: Add checker for image size depending on the bits decided for representing edge weight and vertex index
    dev_image.upload(image);

    Ptr<Filter> filter = createGaussianFilter(CV_8UC3, CV_8UC3, Size(5, 5), 1.0);
    filter->apply(dev_image, dev_output);

    //Graph parameters
    int numVertices = image.rows*image.cols;
    int numEdges= (image.rows)*(image.cols)*8;

    //Convert image to graph
    int32_t *VertexList, *BitEdgeList, *FlagList, *OutList, *NWE, *Successor, *Representative, *Vertex;
    edge *EdgeList;

    hipMallocManaged(&VertexList,numVertices*sizeof(int32_t));
    hipMallocManaged(&FlagList,numVertices*sizeof(int32_t));
    hipMallocManaged(&OutList,numVertices*sizeof(int32_t));
    hipMallocManaged(&EdgeList,numEdges*sizeof(edge));
    hipMallocManaged(&BitEdgeList,numEdges*sizeof(int32_t));
    hipMallocManaged(&NWE,numVertices*sizeof(int32_t));
    hipMallocManaged(&Successor,numVertices*sizeof(int32_t));

    hipMallocManaged(&Representative,numVertices*sizeof(int32_t));
    hipMallocManaged(&Vertex,numVertices*sizeof(int32_t));

    dim3 threadsPerBlock(32,32);
    int BlockX = image.rows/threadsPerBlock.x;
    int BlockY = image.cols/threadsPerBlock.y;
    dim3 numBlocks(BlockX, BlockY);
    hipDeviceSynchronize();
    for(int i =0;i<numEdges;i++)
    {
    EdgeList[i].Weight=0;
    }
    dev_output.download(output);
    //ImagetoGraph<<<numBlocks,threadsPerBlock>>>(dev_image, VertexList, EdgeList, BitEdgeList, FlagList, dev_image.step, 3);
    numEdges = ImagetoGraphSerial(image, EdgeList, VertexList, BitEdgeList);
    hipError_t err = hipGetLastError();
    if ( err != hipSuccess )
    {
        printf("CUDA Error in ImagetoGraph function call: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

   // printf("INFO: Checking the edge list and bit edge list\n");
/*
   for(int i =0;i<4000;i++)
   {
       //if (EdgeList[i].Vertex != 0)
       //{
       //    if ((EdgeList[i].Vertex > 60000) || (BitEdgeList[i] % (2 << 15) > 60000))
       //        printf("ERROR: Something went wrong: ");
           //printf("%d %d %d %d; ##", EdgeList[i].Vertex, EdgeList[i].Weight, BitEdgeList[i] % (2 << 15), BitEdgeList[i] >> 16);
       //}
   }
*/
    ContextPtr context = CreateCudaDevice(argc, argv, true);
    //For the first iteration VertexList and FlagList are exactly same
    //Maybe we don't need separate OutList and NWE arrays
    SegmentedReduction(*context, VertexList, BitEdgeList, OutList, NWE, numEdges, numVertices);
    int numthreads = 1024;
    int numBlock = numVertices/numthreads;
    FindSuccessorArray<<<numBlock,numthreads>>>(Successor, NWE, numVertices);
    err = hipGetLastError();
    if ( err != hipSuccess )
    {
        printf("CUDA Error in FindSuccessorArray function call: %s\n", hipGetErrorString(err));
    }
    RemoveCycles<<<numBlock,numthreads>>>(Successor, numVertices);
    err = hipGetLastError();
    if ( err != hipSuccess )
    {
        printf("CUDA Error in RemoveCycles function call: %s\n", hipGetErrorString(err));
    }
    //numVertices = image.rows * image.cols;
    PropagateRepresentativeVertices(Successor, numVertices);
    SortedSplit(Representative, Vertex, Successor, numVertices);

    return 0;
}
