#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <opencv2/cudafilters.hpp>
#include <opencv2/highgui.hpp>
#include "CreateGraph.h"
#include "moderngpu.cuh"		// Include all MGPU kernels.
#include "FastMST.h"

using namespace cv;
using namespace cv::cuda;
using namespace mgpu;

int main(int argc, char **argv)
{
    Mat image, output;
    GpuMat dev_image, dev_output;

    image = imread("data/beach.png", IMREAD_COLOR);
    printf("Size of image obtained is: Rows: %d, Columns: %d, Pixels: %d\n", image.rows, image.cols, image.rows * image.cols);
    //TODO: Add checker for image size depending on the bits decided for representing edge weight and vertex index
    dev_image.upload(image);

    Ptr<Filter> filter = createGaussianFilter(CV_8UC3, CV_8UC3, Size(5, 5), 1.0);
    filter->apply(dev_image, dev_output);

    //Graph parameters
    int numVertices = image.rows*image.cols;
    int numEdges= (image.rows)*(image.cols)*4;

    //Convert image to graph
    int32_t *VertexList, *BitEdgeList, *FlagList, *OutList, *NWE, *Successor, *newSuccessor, *Representative, *Vertex;
    edge *EdgeList;

    int *flag;
    hipMallocManaged(&flag,numEdges*sizeof(int32_t));
    //Allocating memory
    hipMallocManaged(&VertexList,numVertices*sizeof(int32_t));
    hipMallocManaged(&FlagList,numVertices*sizeof(int32_t));
    hipMallocManaged(&OutList,numVertices*sizeof(int32_t));
    hipMallocManaged(&EdgeList,numEdges*sizeof(edge));
    hipMallocManaged(&BitEdgeList,numEdges*sizeof(int32_t));
    hipMallocManaged(&NWE,numVertices*sizeof(int32_t));
    hipMallocManaged(&Successor,numVertices*sizeof(int32_t));
    hipMallocManaged(&newSuccessor,numVertices*sizeof(int32_t));

    hipMallocManaged(&Representative,numVertices*sizeof(int32_t));
    hipMallocManaged(&Vertex,numVertices*sizeof(int32_t));

    int *Flag2;
    hipMallocManaged(&Flag2,numEdges*sizeof(int32_t));
    int *SuperVertexId;
    hipMallocManaged(&SuperVertexId,numVertices*sizeof(int32_t));

    int *uid;
    hipMallocManaged(&uid,numVertices*sizeof(int32_t));
    dim3 threadsPerBlock(32,32);
    int BlockX = image.rows/threadsPerBlock.x;
    int BlockY = image.cols/threadsPerBlock.y;
    dim3 numBlocks(BlockX, BlockY);
    hipDeviceSynchronize();//FIXME: Need to check where all this synchronize call is needed
    ContextPtr context = CreateCudaDevice(argc, argv, true);
    hipError_t err = hipGetLastError();

    int *flag4; //Same as F4. New flag for creating vertex list. Assigning the new ids.
    hipMallocManaged(&flag4, numEdges * sizeof(int));

    //FIXME: Make this initialization run in parallel?
    //TODO: Figure out if this initialization is required??
    for(int i =0;i<numEdges;i++)
    {
    EdgeList[i].Weight=0;
    }

    dev_output.download(output);

    int32_t tmp_V, tmp_Wt;
    /*
    for(int i =0; i<numEdges; i++)
    {
        tmp_V = BitEdgeList[i]% (2 << 15);
        tmp_Wt = BitEdgeList[i]>>16;
        printf("EdgeListV:%d, EdgeListWt:%d, BitVertex:%d, BitWt:%d\n", EdgeList[i].Vertex, EdgeList[i].Weight, tmp_V, tmp_Wt);
    }*/

    int numthreads = 1024;
    int numBlock = numVertices/numthreads;

    int *UV, *W;
    hipMallocManaged(&UV,numEdges*sizeof(int64_t));
    hipMallocManaged(&W,numEdges*sizeof(int64_t));
    int32_t *flag3;
    hipMallocManaged(&flag3,numEdges*sizeof(int32_t));

    numEdges = ImagetoGraphSerial(image, EdgeList, VertexList, BitEdgeList);

    /**** MST Starts ****/
    bool DidReduce; //Boolean to check if more segments got created or not
    DidReduce = 1;

    while(numVertices>1)
    {
        printf("it\n");

        //1. The graph creation step above takes care of this

        //2. Mark the segments in the flag array. Being used for the uid array below
        ClearFlagArray<<<numBlock, numthreads>>>(flag, numEdges);
        MarkSegments<<<numBlock, numthreads>>>(flag, VertexList, numEdges);

        //3. Segmented min scan
        SegmentedReduction(*context, VertexList, BitEdgeList, OutList, numEdges, numVertices);

        //4. Find Successor array of each vertex
        FindSuccessorArray<<<numBlock, numthreads>>>(Successor, VertexList, OutList, numVertices);
        hipDeviceSynchronize();

        /*printf("\n Printing Segmented Array: \n");
        for(int i =0; i< 1000; i++)
        {
            printf("%d ,", OutList[i]);
        }
        printf("Printing Successor Array: \n");
        for(int i =0; i< 1000; i++)
        {
            printf("%d ,", Successor[i]);
        }*/

        //5. Remove cycle making edges from NWE. But NWE is not used anywhere here??
        RemoveCycles<<<numBlock, numthreads>>>(Successor, numVertices);
        hipDeviceSynchronize();
        /*printf("\n After removing cycles printing Successor Array: \n");
        for(int i =0; i< 1000; i++)
        {
            printf("%d ,", Successor[i]);
        }*/

        //C. Merging vertices and assigning IDs to supervertices
        //7. Propagate representative vertex IDs using pointer doubling

        hipDeviceSynchronize(); //because PropagateRepresentative is on host
        bool change;

        //Code copied
        do{
            CopySuccessorToNewSuccessor<<<numBlock, numthreads>>>(Successor, newSuccessor, numVertices);
            PropagateRepresentativeVertices<<<numBlock, numthreads>>>(Successor, newSuccessor, numVertices, &change);
            CopyNewSuccessorToSuccessor<<<numBlock, numthreads>>>(Successor, newSuccessor, numVertices);
        }while(change);

        hipDeviceSynchronize();
        printf("\n After propagating representative vertices printing Successor Array: \n");
        for(int i =0; i< 1000; i++)
        {
            printf("%d ,", Successor[i]);
        }
        //8, 9 Append appendSuccessorArray
        appendSuccessorArray<<<numBlock,numthreads>>>(Representative, Vertex, Successor, numVertices);
        hipDeviceSynchronize();
        printf("\n Representative array \n");
        for(int i =0; i< 1000; i++)
        {
            printf("%d ,", Representative[i]);
        }
        printf("\n Vertex \n");
        for(int i =0; i< 1000; i++)
        {
            printf("%d ,", Vertex[i]);
        }
        //hipDeviceSynchronize();
        //9. Create F2, Assign new IDs based on Flag2 array
        //SortedSplit(Representative, Vertex, Successor, Flag2, numVertices);
        //hipDeviceSynchronize();
        thrust::sort_by_key(thrust::host, Representative, Representative + numVertices, Vertex);
        printf("\n Sorted representative array \n");
        for(int i =0; i< 1000; i++)
        {
            printf("%d ,", Representative[i]);
        }
        printf("Flag\n");
        for(int i =0; i< 6000; i++)
        {
            printf("%d ,", Flag2[i]);
        }
        //D. Finding the Supervertex ids and storing it in an array
        //CreateSuperVertexArray<<<numBlock,numthreads>>>(SuperVertexId, Vertex, Flag2, numVertices);
        //hipDeviceSynchronize();

        //Create UID array. 10.2
        //CreateUid(uid, flag, numVertices);
        //hipDeviceSynchronize();

        //11. Removing self edges
        //RemoveSelfEdges<<<numBlock,numthreads>>>(BitEdgeList,numEdges, uid, SuperVertexId);
        //hipDeviceSynchronize();

        //E 12.
        //CreateUVWArray<<<numBlock,numthreads>>>(BitEdgeList, numEdges, uid, SuperVertexId, UV, W);
        //hipDeviceSynchronize();
        //printf("Printing UVW array: ");
        //for(int i = 0; i< 1000;i++)
        //{
        //    printf("%d, ", UV[i]);
        //}
        //printf("\n");
        //int new_edge_size = SortUVW(UV, W, numEdges, flag3);

        //flag3 could be renamed to compact location
        //numVertices = CreateNewEdgeVertexList(BitEdgeList, VertexList, UV, W, flag3, new_edge_size, flag4);
        //numEdges = new_edge_size; //This is incorrect. Need to return new_E_size as well
        //printf("\n numVertices: %d numEdges %d", numVertices, numEdges);
        numVertices = 1;

    }

    return 0;
}
