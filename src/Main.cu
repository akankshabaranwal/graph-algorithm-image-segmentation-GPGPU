#include <iostream>
#include <hip/hip_runtime_api.h>
#include <opencv2/cudafilters.hpp>
#include <opencv2/highgui.hpp>
#include "CreateGraph.h"
#include "moderngpu.cuh"		// Include all MGPU kernels.
#include "FastMST.h"

using namespace cv;
using namespace cv::cuda;
using namespace mgpu;

int main(int argc, char **argv)
{
    Mat image, output;
    GpuMat dev_image, dev_output;

    image = imread("data/beach.png", IMREAD_COLOR);

    //TODO: Add checker for image size depending on the bits decided for representing edge weight and vertex index
    dev_image.upload(image);

    Ptr<Filter> filter = createGaussianFilter(CV_8UC3, CV_8UC3, Size(5, 5), 1.0);
    filter->apply(dev_image, dev_output);

    //Graph parameters
    int numVertices = image.rows*image.cols;
    int numEdges= (image.rows)*(image.cols)*8;

    //Convert image to graph
    int32_t *VertexList, *BitEdgeList, *FlagList, *OutList;
    edge *EdgeList;

    hipMallocManaged(&VertexList,numVertices*sizeof(int32_t));
    hipMallocManaged(&FlagList,numVertices*sizeof(int32_t));
    hipMallocManaged(&OutList,numVertices*sizeof(int32_t));
    hipMallocManaged(&EdgeList,numEdges*sizeof(edge));
    hipMallocManaged(&BitEdgeList,numEdges*sizeof(int32_t));

    dim3 threadsPerBlock(32,32);
    int BlockX = image.rows/threadsPerBlock.x;
    int BlockY = image.cols/threadsPerBlock.y;
    dim3 numBlocks(BlockX, BlockY);
    hipDeviceSynchronize();
    for(int i =0;i<numEdges;i++)
    {
    EdgeList[i].Weight=0;
    }
    dev_output.download(output);
    ImagetoGraph<<<numBlocks,threadsPerBlock>>>(dev_image, VertexList, EdgeList, BitEdgeList, FlagList, dev_image.step, 3);
    hipError_t err = hipGetLastError();
    if ( err != hipSuccess )
    {
        printf("CUDA Error in ImagetoGraph function call: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

   // printf("INFO: Checking the edge list and bit edge list\n");

   for(int i =0;i<numEdges;i++)
    {
        if(EdgeList[i].Vertex != 0)
            printf("%d %d %d %d\n", EdgeList[i].Vertex, EdgeList[i].Weight, BitEdgeList[i]%(2<<16), BitEdgeList[i]>>16);
    }

    ContextPtr context = CreateCudaDevice(argc, argv, true);
    SegmentedReduction(*context, VertexList, BitEdgeList, OutList, numEdges,numVertices);

    return 0;
}
