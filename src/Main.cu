#include <iostream>

#include "CreateGraph.h"
#include <hip/hip_runtime_api.h>
#include <opencv2/cudafilters.hpp>
#include <opencv2/highgui.hpp>

using namespace cv;
using namespace cv::cuda;

int main(int argc, char **argv)
{
    Mat image, output;
    GpuMat dev_image, dev_output;

    image = imread("data/beach.png", IMREAD_COLOR);
    dev_image.upload(image);

    Ptr<Filter> filter = createGaussianFilter(CV_8UC3, CV_8UC3, Size(5, 5), 1.0);
    filter->apply(dev_image, dev_output);

    //Graph parameters
    int numVertices = image.rows*image.cols;
    int numEdges= image.rows*image.cols*8;

    //Convert image to graph
    int *VertexList;
    edge *EdgeList;

    hipMallocManaged(&VertexList,numVertices*sizeof(int));
    hipMallocManaged(&EdgeList,numEdges*sizeof(edge));

    dim3 threadsPerBlock(32,32);
    int BlockX = image.rows/threadsPerBlock.x;
    int BlockY = image.cols/threadsPerBlock.y;
    dim3 numBlocks(BlockX,BlockY);

    ImagetoGraph<<<numBlocks,threadsPerBlock>>>(dev_output, VertexList, EdgeList, dev_output.step, dev_output.channels());
    hipError_t err = hipGetLastError();
    if ( err != hipSuccess )
    {
        printf("CUDA Error in ImagetoGraph function call: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    printf("\n");
    dev_output.download(output);
    imshow("Source Image", image);
    imshow("After Blur (CUDA)", output);

    waitKey();
    //Print the pitch information
 //   hipDeviceProp_t devProp;
 //   hipGetDeviceProperties(&devProp, 0);
 //   printf("Maximum memory pitch:%lu\n",  devProp.memPitch);

    return 0;
}
