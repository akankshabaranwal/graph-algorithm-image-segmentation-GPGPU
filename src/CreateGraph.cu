#include "hip/hip_runtime.h"
//
// Created by akanksha on 20.11.20.
//
#include "CreateGraph.h"

__global__ void ImagetoGraph(cv::cuda::GpuMat Image, int32_t *VertexList, edge *EdgeList, int32_t *BitEdgeList, int32_t *FlagList, int32_t pitch, int32_t Channels){

    int32_t i = blockIdx.x*blockDim.x + threadIdx.x ;
    int32_t j = blockIdx.y*blockDim.y + threadIdx.y ;

    int32_t rows = Image.rows;
    int32_t cols = Image.cols;
    //TODO: Check if this needs to be fixed. Right now I am removing all border pixels
    if(i>rows)
        return;
    if(j>cols)
        return;

    //Add 8 neighbors of each pixel to the list of edges
    int32_t PixIdx = i*cols + j;

    //TODO: Check if we really need 8 neighbors?
    int32_t SrcPixX, SrcPixY, SrcPixZ;
    int32_t DestPixX, DestPixY, DestPixZ;
    int32_t DiffX, DiffY, DiffZ;

    //Using 16 bits for Weight and 16 for vertex id

    VertexList[PixIdx] = 8*PixIdx; //VertexList stores the start of each index
    SrcPixX = Image.data[ (i*Image.step) + j*Channels + 0];
    SrcPixY = Image.data[ (i*Image.step) + j*Channels + 1];
    SrcPixZ = Image.data[ (i*Image.step) + j*Channels + 2];

    //TODO: Remove the weight parameter from edgelist array
    EdgeList[8*PixIdx].Vertex = i*cols + j-1; //Left
    DestPixX = Image.data[ (i*Image.step) + (j-1)*Channels + 0];
    DestPixY = Image.data[ (i*Image.step) + (j-1)*Channels + 1];
    DestPixZ = Image.data[ (i*Image.step) + (j-1)*Channels + 2];
    DiffX = DestPixX - SrcPixX;
    DiffY = DestPixY - SrcPixY;
    DiffZ = DestPixZ - SrcPixZ;
    EdgeList[8*PixIdx].Weight = int32_t(sqrtf(DiffX*DiffX + DiffY*DiffY + DiffZ*DiffZ));
    BitEdgeList[8*PixIdx] = (EdgeList[8*PixIdx].Weight * (2<<15)) + EdgeList[8*PixIdx].Vertex;

    EdgeList[8*PixIdx+1].Vertex = (i-1)*cols + j-1; //LeftTop
    DestPixX = Image.data[ ((i-1)*Image.step) + (j-1)*Channels + 0];
    DestPixY = Image.data[ ((i-1)*Image.step) + (j-1)*Channels + 1];
    DestPixZ = Image.data[ ((i-1)*Image.step) + (j-1)*Channels + 2];
    DiffX = DestPixX - SrcPixX;
    DiffY = DestPixY - SrcPixY;
    DiffZ = DestPixZ - SrcPixZ;
    EdgeList[8*PixIdx+1].Weight = int(sqrtf(DiffX*DiffX + DiffY*DiffY + DiffZ*DiffZ));
    BitEdgeList[8*PixIdx+1] = (EdgeList[8*PixIdx+1].Weight*(2<<15)) + EdgeList[8*PixIdx+1].Vertex;

    EdgeList[8*PixIdx+2].Vertex = (i-1)*cols + j; //Top
    DestPixX = Image.data[ ((i-1)*Image.step) + j*Channels + 0];
    DestPixY = Image.data[ ((i-1)*Image.step) + j*Channels + 1];
    DestPixZ = Image.data[ ((i-1)*Image.step) + j*Channels + 2];
    DiffX = DestPixX - SrcPixX;
    DiffY = DestPixY - SrcPixY;
    DiffZ = DestPixZ - SrcPixZ;
    EdgeList[8*PixIdx+2].Weight = int(sqrtf(DiffX*DiffX + DiffY*DiffY + DiffZ*DiffZ));
    BitEdgeList[8*PixIdx+2] = (EdgeList[8*PixIdx+2].Weight*(2<<15)) + EdgeList[8*PixIdx+2].Vertex;

    EdgeList[8*PixIdx+3].Vertex = (i-1)*cols + j+1; //TopRight
    DestPixX = Image.data[ ((i-1)*Image.step) + (j+1)*Channels + 0];
    DestPixY = Image.data[ ((i-1)*Image.step) + (j+1)*Channels + 1];
    DestPixZ = Image.data[ ((i-1)*Image.step) + (j+1)*Channels + 2];
    DiffX = DestPixX - SrcPixX;
    DiffY = DestPixY - SrcPixY;
    DiffZ = DestPixZ - SrcPixZ;
    EdgeList[8*PixIdx+3].Weight = int(sqrtf(DiffX*DiffX + DiffY*DiffY + DiffZ*DiffZ));
    BitEdgeList[8*PixIdx+3] = (EdgeList[8*PixIdx+3].Weight*(2<<15)) + EdgeList[8*PixIdx+3].Vertex;

    EdgeList[8*PixIdx+4].Vertex = i*cols + j+1; //Right
    DestPixX = Image.data[ (i*Image.step) + (j+1)*Channels + 0];
    DestPixY = Image.data[ (i*Image.step) + (j+1)*Channels + 1];
    DestPixZ = Image.data[ (i*Image.step) + (j+1)*Channels + 2];
    DiffX = DestPixX - SrcPixX;
    DiffY = DestPixY - SrcPixY;
    DiffZ = DestPixZ - SrcPixZ;
    EdgeList[8*PixIdx+4].Weight = int(sqrtf(DiffX*DiffX + DiffY*DiffY + DiffZ*DiffZ));
    BitEdgeList[8*PixIdx+4] = (EdgeList[8*PixIdx+4].Weight*(2<<15)) + EdgeList[8*PixIdx+4].Vertex;

    EdgeList[8*PixIdx+5].Vertex = (i+1)*cols + j +1; //BottomRight
    DestPixX = Image.data[ ((i+1)*Image.step) + (j+1)*Channels + 0];
    DestPixY = Image.data[ ((i+1)*Image.step) + (j+1)*Channels + 1];
    DestPixZ = Image.data[ ((i+1)*Image.step) + (j+1)*Channels + 2];
    DiffX = DestPixX - SrcPixX;
    DiffY = DestPixY - SrcPixY;
    DiffZ = DestPixZ - SrcPixZ;
    EdgeList[8*PixIdx+5].Weight = int(sqrtf(DiffX*DiffX + DiffY*DiffY + DiffZ*DiffZ));
    BitEdgeList[8*PixIdx+5] = (EdgeList[8*PixIdx+5].Weight*(2<<15)) + EdgeList[8*PixIdx+5].Vertex;

    EdgeList[8*PixIdx+6].Vertex = (i+1)*cols + j; //Bottom
    DestPixX = Image.data[ ((i-1)*Image.step) + j*Channels + 0];
    DestPixY = Image.data[ ((i-1)*Image.step) + j*Channels + 1];
    DestPixZ = Image.data[ ((i-1)*Image.step) + j*Channels + 2];
    DiffX = DestPixX - SrcPixX;
    DiffY = DestPixY - SrcPixY;
    DiffZ = DestPixZ - SrcPixZ;
    EdgeList[8*PixIdx+6].Weight = int(sqrtf(DiffX*DiffX + DiffY*DiffY + DiffZ*DiffZ));
    BitEdgeList[8*PixIdx+6] =(EdgeList[8*PixIdx+6].Weight*(2<<15))+ EdgeList[8*PixIdx+6].Vertex;

    EdgeList[8*PixIdx+7].Vertex = (i+1)*cols + j-1; //BottomLeft
    DestPixX = Image.data[ ((i+1)*Image.step) + (j-1)*Channels + 0];
    DestPixY = Image.data[ ((i+1)*Image.step) + (j-1)*Channels + 1];
    DestPixZ = Image.data[ ((i+1)*Image.step) + (j-1)*Channels + 2];
    DiffX = DestPixX - SrcPixX;
    DiffY = DestPixY - SrcPixY;
    DiffZ = DestPixZ - SrcPixZ;
    EdgeList[8*PixIdx+7].Weight = int(sqrtf(DiffX*DiffX + DiffY*DiffY + DiffZ*DiffZ));
    BitEdgeList[8*PixIdx+7] =(EdgeList[8*PixIdx+7].Weight *(2<<15)) + EdgeList[8*PixIdx+7].Vertex;
}