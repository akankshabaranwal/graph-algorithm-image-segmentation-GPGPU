#include "hip/hip_runtime.h"
//
// Created by akanksha on 20.11.20.
//
#include "CreateGraph.h"

double dissimilarity(Mat image, uint32_t row1, uint32_t col1, uint32_t row2, uint32_t col2) {
    double dis = 0;
    Point3_<uchar>* u = image.ptr<Point3_<uchar> >(row1,col1);
    Point3_<uchar>* v = image.ptr<Point3_<uchar> >(row2,col2);
    dis = pow((u->x - v->x), 2) + pow((u->y - v->y), 2) + pow((u->z - v->z), 2);
    return sqrt(dis);
}

int ImagetoGraphSerial(Mat image, edge *EdgeList, uint32_t *VertexList, uint64_t *BitEdgeList)
{
    uint32_t cur_edge_idx, cur_vertex_idx, left_node, right_node, bottom_node, top_node;
    cur_edge_idx = 0;
    for(int i=0;i<image.rows;i++)
    {
        for(int j=0;j<image.cols;j++)
        {
            left_node = i * image.cols + j - 1;
            right_node = i * image.cols + j + 1;
            bottom_node = (i+1) * image.cols + j;
            top_node = (i - 1) * image.cols + j;
            //Add the index for VertexList
            cur_vertex_idx = i * image.cols + j;
            VertexList[cur_vertex_idx] = cur_edge_idx;
            if (j > 0){
                EdgeList[cur_edge_idx].Vertex = left_node;
                EdgeList[cur_edge_idx].Weight = dissimilarity(image, i, j, i, j - 1);
                //BitEdgeList[cur_edge_idx] = (EdgeList[cur_edge_idx].Weight * (2<<15)) + left_node;
                BitEdgeList[cur_edge_idx] = (EdgeList[cur_edge_idx].Weight <<32) | left_node;
                cur_edge_idx++;
            }
            if (j < image.cols - 1){
                EdgeList[cur_edge_idx].Vertex = right_node;
                EdgeList[cur_edge_idx].Weight = dissimilarity(image, i, j, i, j + 1);
                //BitEdgeList[cur_edge_idx] = (EdgeList[cur_edge_idx].Weight * (2<<15)) + right_node;
                BitEdgeList[cur_edge_idx] = (EdgeList[cur_edge_idx].Weight <<32) | right_node;
                cur_edge_idx++;
            }
            if (i < image.rows - 1){
                EdgeList[cur_edge_idx].Vertex = bottom_node;
                EdgeList[cur_edge_idx].Weight = dissimilarity(image, i, j, i+1, j);
                //BitEdgeList[cur_edge_idx] = (EdgeList[cur_edge_idx].Weight * (2<<15)) + bottom_node;
                BitEdgeList[cur_edge_idx] = (EdgeList[cur_edge_idx].Weight <<32) | bottom_node;
                cur_edge_idx++;
            }
            if (i > 0){
                EdgeList[cur_edge_idx].Vertex = top_node;
                EdgeList[cur_edge_idx].Weight = dissimilarity(image, i, j, i-1, j);
                //BitEdgeList[cur_edge_idx] = (EdgeList[cur_edge_idx].Weight * (2<<15)) + top_node;
                BitEdgeList[cur_edge_idx] = (EdgeList[cur_edge_idx].Weight <<32) | top_node;
                cur_edge_idx++;
            }
        }
    }
    return cur_edge_idx;
}

__global__ void ImagetoGraph(cv::cuda::GpuMat Image, int32_t *VertexList, edge *EdgeList, int32_t *BitEdgeList, int32_t *FlagList, int32_t pitch, int32_t Channels){

    int32_t i = blockIdx.x*blockDim.x + threadIdx.x +1;
    int32_t j = blockIdx.y*blockDim.y + threadIdx.y +1;

    int32_t rows = Image.rows;
    int32_t cols = Image.cols;

    //TODO: Check if this needs to be fixed. Right now I am removing all border pixels
    if(i>rows-1)
        return;
    if(j>cols-1)
        return;

    //Add 8 neighbors of each pixel to the list of edges
    int32_t PixIdx = i*cols + j;
    if(PixIdx >= 60000)
    {
        printf("ERROR: Something went wrong: %d i, %d j\n", i, j);
    }
    //TODO: Check if we really need 8 neighbors?
    int32_t SrcPixX, SrcPixY, SrcPixZ;
    int32_t DestPixX, DestPixY, DestPixZ;
    int32_t DiffX, DiffY, DiffZ;

    //Using 16 bits for Weight and 16 for vertex id

    VertexList[PixIdx] = 8*PixIdx; //VertexList stores the start of each index
    SrcPixX = Image.data[ (i*Image.step) + j*Channels + 0];
    SrcPixY = Image.data[ (i*Image.step) + j*Channels + 1];
    SrcPixZ = Image.data[ (i*Image.step) + j*Channels + 2];

    //TODO: Remove the weight parameter from edgelist array
    EdgeList[8*PixIdx].Vertex = i*cols + j-1; //Left
    DestPixX = Image.data[ (i*Image.step) + (j-1)*Channels + 0];
    DestPixY = Image.data[ (i*Image.step) + (j-1)*Channels + 1];
    DestPixZ = Image.data[ (i*Image.step) + (j-1)*Channels + 2];
    DiffX = DestPixX - SrcPixX;
    DiffY = DestPixY - SrcPixY;
    DiffZ = DestPixZ - SrcPixZ;
    EdgeList[8*PixIdx].Weight = int32_t(sqrtf(DiffX*DiffX + DiffY*DiffY + DiffZ*DiffZ));
    BitEdgeList[8*PixIdx] = (EdgeList[8*PixIdx].Weight * (2<<15)) + EdgeList[8*PixIdx].Vertex;

    EdgeList[8*PixIdx+1].Vertex = (i-1)*cols + j-1; //LeftTop
    DestPixX = Image.data[ ((i-1)*Image.step) + (j-1)*Channels + 0];
    DestPixY = Image.data[ ((i-1)*Image.step) + (j-1)*Channels + 1];
    DestPixZ = Image.data[ ((i-1)*Image.step) + (j-1)*Channels + 2];
    DiffX = DestPixX - SrcPixX;
    DiffY = DestPixY - SrcPixY;
    DiffZ = DestPixZ - SrcPixZ;
    EdgeList[8*PixIdx+1].Weight = int(sqrtf(DiffX*DiffX + DiffY*DiffY + DiffZ*DiffZ));
    BitEdgeList[8*PixIdx+1] = (EdgeList[8*PixIdx+1].Weight*(2<<15)) + EdgeList[8*PixIdx+1].Vertex;

    EdgeList[8*PixIdx+2].Vertex = (i-1)*cols + j; //Top
    DestPixX = Image.data[ ((i-1)*Image.step) + j*Channels + 0];
    DestPixY = Image.data[ ((i-1)*Image.step) + j*Channels + 1];
    DestPixZ = Image.data[ ((i-1)*Image.step) + j*Channels + 2];
    DiffX = DestPixX - SrcPixX;
    DiffY = DestPixY - SrcPixY;
    DiffZ = DestPixZ - SrcPixZ;
    EdgeList[8*PixIdx+2].Weight = int(sqrtf(DiffX*DiffX + DiffY*DiffY + DiffZ*DiffZ));
    BitEdgeList[8*PixIdx+2] = (EdgeList[8*PixIdx+2].Weight*(2<<15)) + EdgeList[8*PixIdx+2].Vertex;

    EdgeList[8*PixIdx+3].Vertex = (i-1)*cols + j+1; //TopRight
    DestPixX = Image.data[ ((i-1)*Image.step) + (j+1)*Channels + 0];
    DestPixY = Image.data[ ((i-1)*Image.step) + (j+1)*Channels + 1];
    DestPixZ = Image.data[ ((i-1)*Image.step) + (j+1)*Channels + 2];
    DiffX = DestPixX - SrcPixX;
    DiffY = DestPixY - SrcPixY;
    DiffZ = DestPixZ - SrcPixZ;
    EdgeList[8*PixIdx+3].Weight = int(sqrtf(DiffX*DiffX + DiffY*DiffY + DiffZ*DiffZ));
    BitEdgeList[8*PixIdx+3] = (EdgeList[8*PixIdx+3].Weight*(2<<15)) + EdgeList[8*PixIdx+3].Vertex;

    EdgeList[8*PixIdx+4].Vertex = i*cols + j+1; //Right
    DestPixX = Image.data[ (i*Image.step) + (j+1)*Channels + 0];
    DestPixY = Image.data[ (i*Image.step) + (j+1)*Channels + 1];
    DestPixZ = Image.data[ (i*Image.step) + (j+1)*Channels + 2];
    DiffX = DestPixX - SrcPixX;
    DiffY = DestPixY - SrcPixY;
    DiffZ = DestPixZ - SrcPixZ;
    EdgeList[8*PixIdx+4].Weight = int(sqrtf(DiffX*DiffX + DiffY*DiffY + DiffZ*DiffZ));
    BitEdgeList[8*PixIdx+4] = (EdgeList[8*PixIdx+4].Weight*(2<<15)) + EdgeList[8*PixIdx+4].Vertex;

    EdgeList[8*PixIdx+5].Vertex = (i+1)*cols + j +1; //BottomRight
    DestPixX = Image.data[ ((i+1)*Image.step) + (j+1)*Channels + 0];
    DestPixY = Image.data[ ((i+1)*Image.step) + (j+1)*Channels + 1];
    DestPixZ = Image.data[ ((i+1)*Image.step) + (j+1)*Channels + 2];
    DiffX = DestPixX - SrcPixX;
    DiffY = DestPixY - SrcPixY;
    DiffZ = DestPixZ - SrcPixZ;
    EdgeList[8*PixIdx+5].Weight = int(sqrtf(DiffX*DiffX + DiffY*DiffY + DiffZ*DiffZ));
    BitEdgeList[8*PixIdx+5] = (EdgeList[8*PixIdx+5].Weight*(2<<15)) + EdgeList[8*PixIdx+5].Vertex;

    EdgeList[8*PixIdx+6].Vertex = (i+1)*cols + j; //Bottom
    DestPixX = Image.data[ ((i-1)*Image.step) + j*Channels + 0];
    DestPixY = Image.data[ ((i-1)*Image.step) + j*Channels + 1];
    DestPixZ = Image.data[ ((i-1)*Image.step) + j*Channels + 2];
    DiffX = DestPixX - SrcPixX;
    DiffY = DestPixY - SrcPixY;
    DiffZ = DestPixZ - SrcPixZ;
    EdgeList[8*PixIdx+6].Weight = int(sqrtf(DiffX*DiffX + DiffY*DiffY + DiffZ*DiffZ));
    BitEdgeList[8*PixIdx+6] =(EdgeList[8*PixIdx+6].Weight*(2<<15))+ EdgeList[8*PixIdx+6].Vertex;

    EdgeList[8*PixIdx+7].Vertex = (i+1)*cols + j-1; //BottomLeft
    DestPixX = Image.data[ ((i+1)*Image.step) + (j-1)*Channels + 0];
    DestPixY = Image.data[ ((i+1)*Image.step) + (j-1)*Channels + 1];
    DestPixZ = Image.data[ ((i+1)*Image.step) + (j-1)*Channels + 2];
    DiffX = DestPixX - SrcPixX;
    DiffY = DestPixY - SrcPixY;
    DiffZ = DestPixZ - SrcPixZ;
    EdgeList[8*PixIdx+7].Weight = int(sqrtf(DiffX*DiffX + DiffY*DiffY + DiffZ*DiffZ));
    BitEdgeList[8*PixIdx+7] =(EdgeList[8*PixIdx+7].Weight *(2<<15)) + EdgeList[8*PixIdx+7].Vertex;
}