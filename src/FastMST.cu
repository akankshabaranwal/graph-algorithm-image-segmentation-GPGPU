#include "hip/hip_runtime.h"
//
// Created by akanksha on 28.11.20.
//
#include "FastMST.h"
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

using namespace mgpu;

////////////////////////////////////////////////////////////////////////////////
// Scan
//https://moderngpu.github.io/faq.html

__global__ void CreateNWEArray(int32_t *NWE, int32_t *Out, int numSegments)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id < numSegments)
    {          NWE[id] = Out[id] % (2 << 15);
    }
}

void SegmentedReduction(CudaContext& context, int32_t *flag, int32_t *a, int32_t *Out, int32_t *NWE, int numElements, int numSegs)
{
    SegReduceCsr(a, flag, numElements, numSegs, false, Out,(int32_t)INT_MAX, mgpu::minimum<int32_t>(),context);
    hipDeviceSynchronize();

    //Create NWE array with the index of each minimum edge
    int numthreads = 1024;
    int numBlock = numSegs/numthreads;
    CreateNWEArray<<<numBlock,numthreads>>>(NWE, Out, numSegs);
    hipError_t err = hipGetLastError();
    err = hipGetLastError();
    if ( err != hipSuccess )
    {
        printf("CUDA Error in CreateNWEArray function call: %s\n", hipGetErrorString(err));
    }
    //hipDeviceSynchronize();

}

__global__ void FindSuccessorArray(int32_t *Successor, int32_t *NWE, int numSegments)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    int32_t min_edge_index;

    if (id < numSegments)
    {   min_edge_index = NWE[id]; //TODO: Check if this is correct. This will eliminate passing the Out array to this kernel
        Successor[id] = NWE[min_edge_index];
    }
}

__global__ void RemoveCycles(int32_t *Successor, int numSegments)
{
    int vertex = blockIdx.x*blockDim.x+threadIdx.x;
    int32_t successor_2;
    if(vertex<numSegments)
    {
        successor_2 = Successor[Successor[vertex]];
        if(vertex == successor_2) //Cycle detected
        {
            if(vertex < successor_2)
                Successor[vertex] = vertex;
            else
            {
                Successor[Successor[vertex]] = Successor[vertex];
            }
        }
    }
}

__global__ void PropagateParallel(int32_t *Successor, int numSegments, bool *change)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    int32_t successor_2, successor;
    successor = Successor[id];
    successor_2 = Successor[successor];
    if (id < numSegments)
    {   if(successor!=successor_2)
        {
            *change = true;
            Successor[id] = successor_2;
        }
    }
    //TODO: How to return boolean change??
}

void PropagateRepresentativeVertices(int *Successor, int numSegments)
{
    bool change =true;
    while(change)
    {
        change = false;
        int numthreads = 1024;
        int numBlock = numSegments/numthreads;
        //TODO: Is it worth to make this parallel? Repeat copy of change between host and device??
        //PropagateParallel<<<numBlock,numthreads>>>(Successor, numSegments, change);
        int32_t successor, successor_2;
        for(int i=0; i<numSegments;i++)
        {
            successor = Successor[i];
            successor_2 = Successor[successor];
            if(successor!=successor_2)
            {
                change=true;
                Successor[i] = successor_2;
            }
        }
    }
}

__global__ void appendSuccessorArray(int *Representative, int *Vertex, int *Successor, int numSegments)
{
    int vertex = blockIdx.x*blockDim.x+threadIdx.x;
    if(vertex<numSegments)
    {
        Representative[vertex] = Successor[vertex];
        Vertex[vertex] = vertex;
    }
}

__global__ void CreateFlagArray(int *Representative, int *Vertex, int *Flag2, int numSegments)
{
    int vertex = blockIdx.x*blockDim.x+threadIdx.x;
    if((vertex<numSegments)&&(vertex>0))
    {
        if(Representative[vertex] != Representative[vertex-1])
            Flag2[vertex]=1;
        else
            Flag2[vertex]=0;
    }
}

//https://thrust.github.io/doc/group__sorting_gabe038d6107f7c824cf74120500ef45ea.html#gabe038d6107f7c824cf74120500ef45ea
void SortedSplit(int *Representative, int *Vertex, int *Successor, int numSegments)
{
    int numthreads = 1024;
    int numBlock = numSegments/numthreads;
    appendSuccessorArray<<<numBlock,numthreads>>>(Representative, Vertex, Successor, numSegments);
    hipError_t err = hipGetLastError();
    err = hipGetLastError();
    if ( err != hipSuccess )
    {
        printf("CUDA Error in appendSuccessorArray function call: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    thrust::sort_by_key(thrust::host, Representative, Representative + numSegments, Vertex);
    int *Flag2;
    hipMallocManaged(&Flag2,numSegments*sizeof(int32_t));
    CreateFlagArray<<<numBlock,numthreads>>>(Representative, Vertex, Flag2, numSegments);
    //Scan to assign new vertex ids. Use exclusive scan. Run exclusive scan on the flag array
    thrust::inclusive_scan(Flag2, Flag2 + numSegments, Flag2, thrust::plus<int>());
}

void RemoveSelfEdges(){

}