#include "hip/hip_runtime.h"
//
// Created by akanksha on 28.11.20.
//
#include "FastMST.h"
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

using namespace mgpu;

////////////////////////////////////////////////////////////////////////////////
// Scan
//https://moderngpu.github.io/faq.html

__global__ void MarkSegments(int *flag, int *VertexList,int numElements)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id<numElements)
    {
        flag[VertexList[id]] = 1;
    }
}

__global__ void CreateNWEArray(int32_t *NWE, int32_t *Out, int numSegments)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id < numSegments)
    {          NWE[id] = Out[id] % (2 << 15);
    }
}

void SegmentedReduction(CudaContext& context, int32_t *flag, int32_t *a, int32_t *Out, int32_t *NWE, int numElements, int numSegs)
{
    SegReduceCsr(a, flag, numElements, numSegs, false, Out,(int32_t)INT_MAX, mgpu::minimum<int32_t>(),context);
    hipDeviceSynchronize();

    //Create NWE array with the index of each minimum edge
    int numthreads = 1024;
    int numBlock = numSegs/numthreads;
    CreateNWEArray<<<numBlock,numthreads>>>(NWE, Out, numSegs);
    hipError_t err = hipGetLastError();
    err = hipGetLastError();
    if ( err != hipSuccess )
    {
        printf("CUDA Error in CreateNWEArray function call: %s\n", hipGetErrorString(err));
    }
    //hipDeviceSynchronize();

}

__global__ void FindSuccessorArray(int32_t *Successor, int32_t *NWE, int numSegments)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    int32_t min_edge_index;

    if (id < numSegments)
    {   min_edge_index = NWE[id]; //TODO: Check if this is correct. This will eliminate passing the Out array to this kernel
        Successor[id] = NWE[min_edge_index];
    }
}

__global__ void RemoveCycles(int32_t *Successor, int numSegments)
{
    int vertex = blockIdx.x*blockDim.x+threadIdx.x;
    int32_t successor_2;
    if(vertex<numSegments)
    {
        successor_2 = Successor[Successor[vertex]];
        if(vertex == successor_2) //Cycle detected
        {
            if(vertex < successor_2)
                Successor[vertex] = vertex;
            else
            {
                Successor[Successor[vertex]] = Successor[vertex];
            }
        }
    }
}

__global__ void PropagateParallel(int32_t *Successor, int numSegments, bool *change)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    int32_t successor_2, successor;
    successor = Successor[id];
    successor_2 = Successor[successor];
    if (id < numSegments)
    {   if(successor!=successor_2)
        {
            *change = true;
            Successor[id] = successor_2;
        }
    }
    //TODO: How to return boolean change??
}

void PropagateRepresentativeVertices(int *Successor, int numSegments)
{
    bool change =true;
    while(change)
    {
        change = false;
        int numthreads = 1024;
        int numBlock = numSegments/numthreads;
        //TODO: Is it worth to make this parallel? Repeat copy of 'change' between host and device??
        //PropagateParallel<<<numBlock,numthreads>>>(Successor, numSegments, change);
        int32_t successor, successor_2;
        for(int i=0; i<numSegments;i++)
        {
            successor = Successor[i];
            successor_2 = Successor[successor];
            if(successor!=successor_2)
            {
                change=true;
                Successor[i] = successor_2;
            }
        }
    }
}

__global__ void appendSuccessorArray(int *Representative, int *Vertex, int *Successor, int numSegments)
{
    int vertex = blockIdx.x*blockDim.x+threadIdx.x;
    if(vertex<numSegments)
    {
        Representative[vertex] = Successor[vertex];
        Vertex[vertex] = vertex;
    }
}

__global__ void CreateFlagArray(int *Representative, int *Vertex, int *Flag2, int numSegments)
{
    int vertex = blockIdx.x*blockDim.x+threadIdx.x;
    if((vertex<numSegments)&&(vertex>0))
    {
        if(Representative[vertex] != Representative[vertex-1])
            Flag2[vertex]=1;
        else
            Flag2[vertex]=0;
    }
}

//https://thrust.github.io/doc/group__sorting_gabe038d6107f7c824cf74120500ef45ea.html#gabe038d6107f7c824cf74120500ef45ea
void SortedSplit(int *Representative, int *Vertex, int *Successor, int *Flag2, int numSegments)
{
    int numthreads = 1024;
    int numBlock = numSegments/numthreads;
    appendSuccessorArray<<<numBlock,numthreads>>>(Representative, Vertex, Successor, numSegments);
    hipError_t err = hipGetLastError();
    err = hipGetLastError();
    if ( err != hipSuccess )
    {
        printf("CUDA Error in appendSuccessorArray function call: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    thrust::sort_by_key(thrust::host, Representative, Representative + numSegments, Vertex);

    CreateFlagArray<<<numBlock,numthreads>>>(Representative, Vertex, Flag2, numSegments);
    //Scan to assign new vertex ids. Use exclusive scan. Run exclusive scan on the flag array
    thrust::inclusive_scan(Flag2, Flag2 + numSegments, Flag2, thrust::plus<int>());
}

//TODO: The array names need to be verified
__global__ void RemoveSelfEdges(int *SuperVertexId, int *Vertex, int *Flag2, int numSegments){
    // Find supervertex id. Create a supervertex array for the original vertex ids
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int32_t vertex, supervertex_id;
    if(idx < numSegments)
    {
        vertex = Vertex[idx];
        supervertex_id = Flag2[idx];
        SuperVertexId[vertex] = Vertex[idx];
    }
}

//10.2
void CreateUid(int *uid, int *flag, int numElements)
{
    thrust::inclusive_scan(flag, flag + numElements, uid, thrust::plus<int>());
}

//11 Removing self edges
__global__ void RemoveSelfEdges(int *BitEdgeList, int numEdges, int *uid, int *SuperVertexId)
{   int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int32_t supervertexid_u, supervertexid_v, id_u, id_v;
    if(idx<numEdges)
    {
        id_u = uid[idx];
        supervertexid_u = SuperVertexId[id_u];

        id_v = BitEdgeList[idx]% (2 << 15);
        supervertexid_v = SuperVertexId[id_v];

        if(supervertexid_u == supervertexid_v)
        {
            BitEdgeList[idx] = -1; //Marking edge to remove it
        }
    }
}


//12 Removing duplicate edges
//Instead of UVW array create separate U, V, W array.
__global__ void CreateUVWArray(int *BitEdgeList, int numEdges, int *uid, int *SuperVertexId, int *UV, int *W)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x; //Index for accessing Edge
    int32_t id_u, id_v, edge_weight;
    int32_t supervertexid_u, supervertexid_v;
    if(idx < numEdges)
    {
        id_u = uid[idx];
        id_v = BitEdgeList[idx]>>15; //TODO: Check if this is correct
        edge_weight = BitEdgeList[idx]% (2 << 15);//TODO: Check if we can use the NWE array?
        if(id_v != -1) //Check if the edge is marked using the criteria from before
        {
            supervertexid_u = SuperVertexId[id_u];
            supervertexid_v = SuperVertexId[id_v];
            UV[idx] = supervertexid_u*(2<<15) + supervertexid_v; //TODO: Check if the UV here needs to be 64bit??
            W[idx] = edge_weight;
        }
        else
        {
            UV[idx] = -1;
            W[idx] = -1; //TODO: Need to replace the -1 with INT_MAX
        }
    }
}

//12.2
void SortUVW(int *UV, int *W, int numEdges)
{
    thrust::sort_by_key(thrust::host, UV, UV + numEdges, W);
    //12.3
    //Initialize F3 array

}

//Create new edge list and vertex list
void CreateNewEdgeVertexList(int *newBitEdgeList, int *newVertexList, int *U, int *V, int *W, int numnewEdges, int numnewVertices)
{
    //Check if this can be parallelized? can we move the min (new_edge_size) part to somewhere before?
    int32_t supervertex_id_u, supervertex_id_v;
    for(int i=0;i<numnewEdges;i++)
    {

    }
}
