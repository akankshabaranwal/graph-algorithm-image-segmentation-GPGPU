#include "hip/hip_runtime.h"
//
// Created by akanksha on 28.11.20.
//
#include "FastMST.h"

using namespace mgpu;

////////////////////////////////////////////////////////////////////////////////
// Scan
//https://moderngpu.github.io/faq.html

__global__ void CreateNWEArray(int32_t *NWE, int32_t *Out, int numSegments)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id < numSegments)
    {          NWE[id] = Out[id] % (2 << 15);
    }
}

void SegmentedReduction(CudaContext& context, int32_t *flag, int32_t *a, int32_t *Out, int32_t *NWE, int numElements, int numSegs)
{
    SegReduceCsr(a, flag, numElements, numSegs, false, Out,(int32_t)INT_MAX, mgpu::minimum<int32_t>(),context);
    hipDeviceSynchronize();

    //Create NWE array with the index of each minimum edge
    int numthreads = 1024;
    int numBlock = numSegs/numthreads;
    CreateNWEArray<<<numBlock,numthreads>>>(NWE, Out, numSegs);
    hipError_t err = hipGetLastError();
    err = hipGetLastError();
    if ( err != hipSuccess )
    {
        printf("CUDA Error in CreateNWEArray function call: %s\n", hipGetErrorString(err));
    }
    //hipDeviceSynchronize();

    //for (int i = 0; i < numSegs; i++)
    //{
        //NWE[i] = Out[i] % (2 << 15);
     //   printf("%d, ", NWE[i]);
        //if((NWE[i]<0)||(NWE[i]>59999))
        //{
        //    printf("Indexing Error for Out %d Flag %d!!\n", Out[i], flag[i]);
        //}
    //}
}

__global__ void FindSuccessorArray(int32_t *Successor, int32_t *NWE, int numSegments)
{
 /* Iterate through the NWE array from SegmentedReduction
  * Create a Successor array in parallel based on the vertex id of the minimum edge weight that was selected
  */
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    int32_t min_edge_index;

    if (id < numSegments)
    {   min_edge_index = NWE[id]; //TODO: Check if this is correct. This will eliminate passing the Out array to this kernel
        Successor[id] = NWE[min_edge_index];
    }
}

__global__ void RemoveCycles(int32_t *Successor, int numSegments)
{
    int vertex = blockIdx.x*blockDim.x+threadIdx.x;
    int32_t successor_2;
    if(vertex<numSegments)
    {
        successor_2 = Successor[Successor[vertex]];
        if(vertex == successor_2) //Cycle detected
        {
            if(vertex < successor_2)
                Successor[vertex] = vertex;
            else
            {
                Successor[Successor[vertex]] = Successor[vertex];
            }
        }
    }
}

__global__ void PropagateParallel(int32_t *Successor, int numSegments)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    bool change;
    int32_t successor_2, successor;
    successor = Successor[id];
    successor_2 = Successor[successor];
    if (id < numSegments)
    {   if(successor!=successor_2)
        {
            change = true;
            Successor[id] = successor_2;
        }
    }
    //TODO: How to return boolean change??
}
void PropagateRepresentativeVertices(int *Successor, int numSegments)
{
    bool change =true;
    while(change)
    {
        change = false;
        int numthreads = 1024;
        int numBlock = numSegments/numthreads;
        PropagateParallel<<<numBlock,numthreads>>>(NWE, Out, numSegs);
    }
}
