/***********************************************************************************
  Implementing Minimum Spanning Tree on CUDA using primitive operations for the 
  algorithm given in "Fast Minimum Spanning Tree Computation", by Pawan Harish, 
  P.J. Narayanan, Vibhav Vineet, and Suryakant Patidar.

  Chapter 7 of Nvidia GPU Computing Gems, Jade Edition, 2011.
  
  Copyright (c) 2011 International Institute of Information Technology - Hyderabad. 
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for 
  educational purpose is hereby granted without fee, provided that the above copyright 
  notice and this permission notice appear in all copies of this software and that you do 
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND, EXPRESSED, IMPLIED OR 
  OTHERWISE.

  Kernels for MST implementation, by Pawan Harish.
 ************************************************************************************/

#ifndef _KERNELS_H_
#define _KERNELS_H_

#define MOVEBITS 26 						// Amount of bits in X for vertex ID
#define NO_OF_BITS_TO_SPLIT_ON 32			// Amount of bits for L split (32 bits one vertex, 32 other)
#define NO_OF_BITS_MOVED_FOR_VERTEX_IDS 26
#define MAX_THREADS_PER_BLOCK 1024 			// IMPORTANT TO SET CORRECTLY
#define INF 10000000						// Make sure larger than amount of edges, maybe best equal to max possible vertex ID // TODO: set to 2^MOVEBITS - 1
#define CHANNEL_SIZE 3						// Amount of color channels, 3 for RGB
#define SCALE 8 							// Make sure to set scale so weight less than assigned amount of bits

////////////////////////////////////////////////////////////////////////////////////////////
// Graph creation kernels
////////////////////////////////////////////////////////////////////////////////////////////
__global__ void createCornerGraphKernel(unsigned char *image, unsigned int *d_vertex, unsigned int *d_edge, unsigned int *d_weight, unsigned int no_of_rows, unsigned int no_of_cols, size_t pitch) 
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if (tid < 4) {
		unsigned int row = 0;
		unsigned int col = 0;
		unsigned int write_offset = 0;

		if (tid == 1) {
			col = no_of_cols - 1;
			write_offset = 3 * (no_of_cols - 1) - 1;
		}
		if (tid == 2) {
			row = no_of_rows - 1;
			write_offset = 4 + 6 * (no_of_rows-2) + 3 * (no_of_cols-2) + 4 * (no_of_rows-2) * (no_of_cols-2);
		}
		if (tid == 3) {
			col = no_of_cols - 1;
			row = no_of_rows - 1;
			write_offset = 6 + 6 * (no_of_rows-2) + 6 * (no_of_cols-2) + 4 * (no_of_rows-2) * (no_of_cols-2);
		}

		unsigned int left_node = row * no_of_cols + col - 1;
        unsigned int right_node = row * no_of_cols + col + 1;
        unsigned int top_node = (row - 1) * no_of_cols + col;
        unsigned int bottom_node = (row+1) * no_of_cols + col;

      	unsigned int this_img_idx = row * pitch + col * CHANNEL_SIZE;
    	unsigned char this_r = image[this_img_idx];
    	unsigned char this_g = image[this_img_idx + 1];
    	unsigned char this_b = image[this_img_idx + 2];

    	unsigned char other_r;
    	unsigned char other_g;
    	unsigned char other_b;
    	unsigned int other_img_idx;
    	double distance;

    	unsigned long cur_vertex_idx = row * no_of_cols + col;
        d_vertex[cur_vertex_idx] = write_offset;

    	// Left node
    	if (tid == 1 || tid == 3) {
    		d_edge[write_offset] = left_node;

	    	other_img_idx = row * pitch + (col - 1) * CHANNEL_SIZE;
	        other_r = image[other_img_idx];
	        other_g = image[other_img_idx + 1];
	        other_b = image[other_img_idx + 2];
	    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
	    	d_weight[write_offset] = (unsigned int) round(distance);
    	}
    	
    	// Right node
    	if (tid == 0 || tid == 2) {
    		d_edge[write_offset+1] = right_node;

	        other_img_idx = row * pitch + (col + 1) * CHANNEL_SIZE;
	        other_r = image[other_img_idx];
	        other_g = image[other_img_idx + 1];
	        other_b = image[other_img_idx + 2];
	    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
	    	d_weight[write_offset+1] = (unsigned int) round(distance);
    	}
       

    	// Top node
    	if (tid == 2 || tid == 3) {
    		d_edge[write_offset+2] = top_node;

	        other_img_idx = (row-1) * pitch + col * CHANNEL_SIZE;
	        other_r = image[other_img_idx];
	        other_g = image[other_img_idx + 1];
	        other_b = image[other_img_idx + 2];
	    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
	    	d_weight[write_offset+2] = (unsigned int) round(distance);
    	}
        

    	// Bottom node
    	if (tid == 0 || tid == 1) {
    		d_edge[write_offset+3] = bottom_node;

	        other_img_idx = (row+1) * pitch + col * CHANNEL_SIZE;
	        other_r = image[other_img_idx];
	        other_g = image[other_img_idx + 1];
	        other_b = image[other_img_idx + 2];
	    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
	    	d_weight[write_offset+3] = (unsigned int) round(distance);
    	}
	}
}


__global__ void createFirstRowGraphKernel(unsigned char *image, unsigned int *d_vertex, unsigned int *d_edge, unsigned int *d_weight, unsigned int no_of_rows, unsigned int no_of_cols, size_t pitch) 
{
	unsigned int row = 0;
	unsigned int col = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;

    if (col > 0 && col < no_of_cols) {
    	unsigned int left_node = row * no_of_cols + col - 1;
        unsigned int right_node = row * no_of_cols + col + 1;
        unsigned int bottom_node = (row+1) * no_of_cols + col;

      	unsigned int this_img_idx = row * pitch + col * CHANNEL_SIZE;
    	unsigned char this_r = image[this_img_idx];
    	unsigned char this_g = image[this_img_idx + 1];
    	unsigned char this_b = image[this_img_idx + 2];

    	unsigned int write_offset = 2 + (col-1) * 3;

    	unsigned char other_r;
    	unsigned char other_g;
    	unsigned char other_b;
    	unsigned int other_img_idx;
    	double distance;

    	unsigned long cur_vertex_idx = row * no_of_cols + col;
        d_vertex[cur_vertex_idx] = write_offset;

    	// Left node
    	d_edge[write_offset] = left_node;

    	other_img_idx = row * pitch + (col - 1) * CHANNEL_SIZE;
        other_r = image[other_img_idx];
        other_g = image[other_img_idx + 1];
        other_b = image[other_img_idx + 2];
    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
    	d_weight[write_offset] = (unsigned int) round(distance);

    	// Right node
        d_edge[write_offset+1] = right_node;

        other_img_idx = row * pitch + (col + 1) * CHANNEL_SIZE;
        other_r = image[other_img_idx];
        other_g = image[other_img_idx + 1];
        other_b = image[other_img_idx + 2];
    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
    	d_weight[write_offset+1] = (unsigned int) round(distance);

    	// Bottom node
        d_edge[write_offset+2] = bottom_node;

        other_img_idx = (row+1) * pitch + col * CHANNEL_SIZE;
        other_r = image[other_img_idx];
        other_g = image[other_img_idx + 1];
        other_b = image[other_img_idx + 2];
    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
    	d_weight[write_offset+2] = (unsigned int) round(distance);
    }
}

__global__ void createLastRowGraphKernel(unsigned char *image, unsigned int *d_vertex, unsigned int *d_edge, unsigned int *d_weight, unsigned int no_of_rows, unsigned int no_of_cols, size_t pitch) 
{
	unsigned int row = no_of_rows-1;;
    unsigned int col = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;

    if (col > 0 && col < no_of_cols) {
    	unsigned int left_node = row * no_of_cols + col - 1;
        unsigned int right_node = row * no_of_cols + col + 1;
        unsigned int top_node = (row - 1) * no_of_cols + col;

      	unsigned int this_img_idx = row * pitch + col * CHANNEL_SIZE;
    	unsigned char this_r = image[this_img_idx];
    	unsigned char this_g = image[this_img_idx + 1];
    	unsigned char this_b = image[this_img_idx + 2];

    	unsigned int first_row_offset = 4 + 3 * (no_of_cols-2);
    	unsigned int extra_cur_row_offset = 3 + (row-1) * (6 + 4 * (no_of_cols-2));
    	unsigned int extra_cur_col_offset = 3 * (col-1);
    	unsigned int write_offset = first_row_offset + extra_cur_row_offset + extra_cur_col_offset;

    	unsigned char other_r;
    	unsigned char other_g;
    	unsigned char other_b;
    	unsigned int other_img_idx;
    	double distance;

    	unsigned long cur_vertex_idx = row * no_of_cols + col;
        d_vertex[cur_vertex_idx] = write_offset;

    	// Left node
    	d_edge[write_offset] = left_node;

    	other_img_idx = row * pitch + (col - 1) * CHANNEL_SIZE;
        other_r = image[other_img_idx];
        other_g = image[other_img_idx + 1];
        other_b = image[other_img_idx + 2];
    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
    	d_weight[write_offset] = (unsigned int) round(distance);

    	// Right node
        d_edge[write_offset+1] = right_node;

        other_img_idx = row * pitch + (col + 1) * CHANNEL_SIZE;
        other_r = image[other_img_idx];
        other_g = image[other_img_idx + 1];
        other_b = image[other_img_idx + 2];
    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
    	d_weight[write_offset+1] = (unsigned int) round(distance);

    	// Top node
        d_edge[write_offset+2] = top_node;

        other_img_idx = (row-1) * pitch + col * CHANNEL_SIZE;
        other_r = image[other_img_idx];
        other_g = image[other_img_idx + 1];
        other_b = image[other_img_idx + 2];
    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
    	d_weight[write_offset+2] = (unsigned int) round(distance);
    }
}

__global__ void createFirstColumnGraphKernel(unsigned char *image, unsigned int *d_vertex, unsigned int *d_edge, unsigned int *d_weight, unsigned int no_of_rows, unsigned int no_of_cols, size_t pitch) 
{
	unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int col = 0;

    if (row > 0 && row < no_of_rows) {
        unsigned int right_node = row * no_of_cols + col + 1;
        unsigned int top_node = (row - 1) * no_of_cols + col;
        unsigned int bottom_node = (row+1) * no_of_cols + col;

      	unsigned int this_img_idx = row * pitch + col * CHANNEL_SIZE;
    	unsigned char this_r = image[this_img_idx];
    	unsigned char this_g = image[this_img_idx + 1];
    	unsigned char this_b = image[this_img_idx + 2];

    	unsigned int first_row_offset = 4 + 3 * (no_of_cols-2);
    	unsigned int extra_cur_row_offset = (row-1) * (6 + 4 * (no_of_cols-2));
    	unsigned int write_offset = first_row_offset + extra_cur_row_offset;

    	unsigned char other_r;
    	unsigned char other_g;
    	unsigned char other_b;
    	unsigned int other_img_idx;
    	double distance;

    	unsigned long cur_vertex_idx = row * no_of_cols + col;
        d_vertex[cur_vertex_idx] = write_offset;

    	// Right node
        d_edge[write_offset] = right_node;

        other_img_idx = row * pitch + (col + 1) * CHANNEL_SIZE;
        other_r = image[other_img_idx];
        other_g = image[other_img_idx + 1];
        other_b = image[other_img_idx + 2];
    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
    	d_weight[write_offset] = (unsigned int) round(distance);

    	// Top node
        d_edge[write_offset+1] = top_node;

        other_img_idx = (row-1) * pitch + col * CHANNEL_SIZE;
        other_r = image[other_img_idx];
        other_g = image[other_img_idx + 1];
        other_b = image[other_img_idx + 2];
    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
    	d_weight[write_offset+1] = (unsigned int) round(distance);

    	// Bottom node
        d_edge[write_offset+2] = bottom_node;

        other_img_idx = (row+1) * pitch + col * CHANNEL_SIZE;
        other_r = image[other_img_idx];
        other_g = image[other_img_idx + 1];
        other_b = image[other_img_idx + 2];
    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
    	d_weight[write_offset+2] = (unsigned int) round(distance);
    }
}

__global__ void createLastColumnGraphKernel(unsigned char *image, unsigned int *d_vertex, unsigned int *d_edge, unsigned int *d_weight, unsigned int no_of_rows, unsigned int no_of_cols, size_t pitch) 
{
	unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int col = no_of_cols - 1;

    if (row > 0 && col > 0 && row < no_of_rows && col < no_of_cols) {
    	unsigned int left_node = row * no_of_cols + col - 1;
        unsigned int top_node = (row - 1) * no_of_cols + col;
        unsigned int bottom_node = (row+1) * no_of_cols + col;

      	unsigned int this_img_idx = row * pitch + col * CHANNEL_SIZE;
    	unsigned char this_r = image[this_img_idx];
    	unsigned char this_g = image[this_img_idx + 1];
    	unsigned char this_b = image[this_img_idx + 2];

    	unsigned int first_row_offset = 4 + 3 * (no_of_cols-2);
    	unsigned int extra_cur_row_offset = 3 + (row-1) * (6 + 4 * (no_of_cols-2));
    	unsigned int extra_cur_col_offset = 4 * (col-1);
    	unsigned int write_offset = first_row_offset + extra_cur_row_offset + extra_cur_col_offset;

    	unsigned char other_r;
    	unsigned char other_g;
    	unsigned char other_b;
    	unsigned int other_img_idx;
    	double distance;

    	unsigned long cur_vertex_idx = row * no_of_cols + col;
        d_vertex[cur_vertex_idx] = write_offset;

    	// Left node
    	d_edge[write_offset] = left_node;

    	other_img_idx = row * pitch + (col - 1) * CHANNEL_SIZE;
        other_r = image[other_img_idx];
        other_g = image[other_img_idx + 1];
        other_b = image[other_img_idx + 2];
    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
    	d_weight[write_offset] = (unsigned int) round(distance);

    	// Top node
        d_edge[write_offset+1] = top_node;

        other_img_idx = (row-1) * pitch + col * CHANNEL_SIZE;
        other_r = image[other_img_idx];
        other_g = image[other_img_idx + 1];
        other_b = image[other_img_idx + 2];
    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
    	d_weight[write_offset+1] = (unsigned int) round(distance);

    	// Bottom node
        d_edge[write_offset+2] = bottom_node;

        other_img_idx = (row+1) * pitch + col * CHANNEL_SIZE;
        other_r = image[other_img_idx];
        other_g = image[other_img_idx + 1];
        other_b = image[other_img_idx + 2];
    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
    	d_weight[write_offset+2] = (unsigned int) round(distance);
    }
}

__global__ void createInnerGraphKernel(unsigned char *image, unsigned int *d_vertex, unsigned int *d_edge, unsigned int *d_weight, unsigned int no_of_rows, unsigned int no_of_cols, size_t pitch) 
{
	unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int col = blockDim.y * blockIdx.y + threadIdx.y;

    if (row > 0 && col > 0 && row < no_of_rows && col < no_of_cols) {
    	unsigned int left_node = row * no_of_cols + col - 1;
        unsigned int right_node = row * no_of_cols + col + 1;
        unsigned int top_node = (row - 1) * no_of_cols + col;
        unsigned int bottom_node = (row+1) * no_of_cols + col;

      	unsigned int this_img_idx = row * pitch + col * CHANNEL_SIZE;
    	unsigned char this_r = image[this_img_idx];
    	unsigned char this_g = image[this_img_idx + 1];
    	unsigned char this_b = image[this_img_idx + 2];

    	unsigned int first_row_offset = 4 + 3 * (no_of_cols-2);
    	unsigned int extra_cur_row_offset = 3 + (row-1) * (6 + 4 * (no_of_cols-2));
    	unsigned int extra_cur_col_offset = 4 * (col-1);
    	unsigned int write_offset = first_row_offset + extra_cur_row_offset + extra_cur_col_offset;

    	unsigned char other_r;
    	unsigned char other_g;
    	unsigned char other_b;
    	unsigned int other_img_idx;
    	double distance;

    	unsigned long cur_vertex_idx = row * no_of_cols + col;
        d_vertex[cur_vertex_idx] = write_offset;

    	// Left node
    	d_edge[write_offset] = left_node;

    	other_img_idx = row * pitch + (col - 1) * CHANNEL_SIZE;
        other_r = image[other_img_idx];
        other_g = image[other_img_idx + 1];
        other_b = image[other_img_idx + 2];
    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
    	d_weight[write_offset] = (unsigned int) round(distance);

    	// Right node
        d_edge[write_offset+1] = right_node;

        other_img_idx = row * pitch + (col + 1) * CHANNEL_SIZE;
        other_r = image[other_img_idx];
        other_g = image[other_img_idx + 1];
        other_b = image[other_img_idx + 2];
    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
    	d_weight[write_offset+1] = (unsigned int) round(distance);

    	// Top node
        d_edge[write_offset+2] = top_node;

        other_img_idx = (row-1) * pitch + col * CHANNEL_SIZE;
        other_r = image[other_img_idx];
        other_g = image[other_img_idx + 1];
        other_b = image[other_img_idx + 2];
    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
    	d_weight[write_offset+2] = (unsigned int) round(distance);

    	// Bottom node
        d_edge[write_offset+3] = bottom_node;

        other_img_idx = (row+1) * pitch + col * CHANNEL_SIZE;
        other_r = image[other_img_idx];
        other_g = image[other_img_idx + 1];
        other_b = image[other_img_idx + 2];
    	distance = SCALE * sqrt(pow((this_r - other_r), 2) + pow((this_g - other_g), 2) + pow((this_b - other_b), 2));
    	d_weight[write_offset+3] = (unsigned int) round(distance);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////
// Segment extraction kernels
////////////////////////////////////////////////////////////////////////////////////////////
__global__ void RandFloatToRandRGB(char* d_component_colours, float *d_component_colours_float, unsigned int n_numbers) 
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if (tid < n_numbers) {
		float color = 255 *d_component_colours_float[tid];
		d_component_colours[tid] = (char) color;
	}
}

__global__ void InitPrevLevelComponents(unsigned int* d_prev_level_component, unsigned int no_of_rows, unsigned int no_of_cols) 
{
	unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int col = blockDim.y * blockIdx.y + threadIdx.y;

    if (row < no_of_rows && col < no_of_cols) {
		d_prev_level_component[row * no_of_cols + col] = row * no_of_cols + col;
	}
}

__global__ void CreateLevelOutput(char *d_output_image, char *d_component_colours, unsigned int* d_level, unsigned int* d_prev_level_component, unsigned int no_of_rows, unsigned int no_of_cols) 
{
	unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int col = blockDim.y * blockIdx.y + threadIdx.y;

    if (row < no_of_rows && col < no_of_cols) {

    	unsigned int prev_component = d_prev_level_component[row * no_of_cols + col];
		unsigned int new_component = d_level[prev_component];

		int img_pos = CHANNEL_SIZE * (row * no_of_cols + col);
		int colour_pos = CHANNEL_SIZE * new_component;

		d_output_image[img_pos] = d_component_colours[colour_pos];
		d_output_image[img_pos + 1] = d_component_colours[colour_pos+1];
		d_output_image[img_pos + 2] = d_component_colours[colour_pos+2];

        d_prev_level_component[row * no_of_cols + col] = new_component;
	}
}

////////////////////////////////////////////////////////////////////////////////////////////
// Append the Weight And Vertex ID into segmented min scan input array, Runs for Edge Length
////////////////////////////////////////////////////////////////////////////////////////////
__global__ void AppendKernel_1(unsigned long long int *d_segmented_min_scan_input, unsigned int *d_weight, unsigned int *d_edges, unsigned int no_of_edges) 
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<no_of_edges) {
		unsigned long long int val=d_weight[tid];

        val = val<<MOVEBITS; // TODO
        val = val|tid; // TODO

		val=val<<MOVEBITS;
		val=val|d_edges[tid];
     
		d_segmented_min_scan_input[tid]=val;
	}
}

////////////////////////////////////////////////////////////////////////////////
// Make the flag for Input to the segmented min scan, Runs for Edge Length
////////////////////////////////////////////////////////////////////////////////
__global__ void ClearArray(unsigned int *d_array, unsigned int size) 
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<size) {
		d_array[tid]=0;
	}
}

__global__ void ClearEdgeStuff(unsigned int *d_edge, unsigned int *d_weight, unsigned int *d_edge_mapping_copy, unsigned int *d_pick_array, unsigned int size) 
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<size) {
		d_edge[tid]=0;
		d_weight[tid]=0;
		d_edge_mapping_copy[tid]=0;
		d_pick_array[tid]=0;
	}
}

////////////////////////////////////////////////////////////////////////////////
// Make the flag for Input to the segmented min scan, Runs for Vertex Length
////////////////////////////////////////////////////////////////////////////////
__global__ void MakeFlag_3(unsigned int *d_edge_flag, unsigned int *d_vertex, unsigned int no_of_vertices) 
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<no_of_vertices) {
		unsigned int pointingvertex = d_vertex[tid];
		d_edge_flag[pointingvertex]=1;
	}
}


////////////////////////////////////////////////////////////////////////////////
// Make the Successor array, Runs for Vertex Length
////////////////////////////////////////////////////////////////////////////////
__global__ void MakeSucessorArray(unsigned int *d_successor, unsigned int *d_vertex, unsigned long long int *d_segmented_min_scan_output, unsigned int no_of_vertices, unsigned int no_of_edges) 
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<no_of_vertices) {
		unsigned int end; // Result values always stored at end of each segment
		if(tid<no_of_vertices-1) {
			end = d_vertex[tid+1]-1; // Get end of my segment
		} else {
			end = no_of_edges-1; // Last segment: end = last edge
		}
		unsigned long long int mask = pow(2.0,MOVEBITS)-1; // Mask to extract vertex ID MWOE // TODO
		d_successor[tid] = d_segmented_min_scan_output[end]&mask; // Get vertex part of each (weight|to_vertex_id) element
	}
}

////////////////////////////////////////////////////////////////////////////////
// Remove Cycles Using Successor array, Runs for Vertex Length
////////////////////////////////////////////////////////////////////////////////
__global__ void RemoveCycles(unsigned int *d_successor, unsigned int no_of_vertices) 
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x; // TID: vertex ID
	if(tid<no_of_vertices) {
		unsigned int succ = d_successor[tid];
		unsigned int nextsucc = d_successor[succ];
		if(tid == nextsucc) { //Found a Cycle
			//Give the minimum one its own value, breaking the cycle and setting the Representative Vertices
			if(tid < succ) {
				d_successor[tid]=tid;
			} else {
				d_successor[succ]=succ;
			}
		}
	}
}


__global__ void SuccToCopy(unsigned int *d_successor, unsigned int *d_successor_copy, unsigned int no_of_vertices)
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<no_of_vertices) {
		d_successor_copy[tid] = d_successor[tid];
	}
}

////////////////////////////////////////////////////////////////////////////////
// Propagate Representative IDs by setting S(u)=S(S(u)), Runs for Vertex Length
////////////////////////////////////////////////////////////////////////////////
__global__ void PropagateRepresentativeID(unsigned int *d_successor, unsigned int *d_successor_copy, bool *d_succchange, unsigned int no_of_vertices)
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<no_of_vertices) {
		unsigned int succ = d_successor[tid];
		unsigned int newsucc = d_successor[succ];
		if(succ!=newsucc) { //Execution goes on
            printf("d_successor_copy[%d]=%d \n", tid, newsucc);
			d_successor_copy[tid] = newsucc; //cannot have input and output in the same array!!!!!
			*d_succchange=true;
		}
	}
}

__global__ void CopyToSucc(unsigned int *d_successor, unsigned int *d_successor_copy, unsigned int no_of_vertices)
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<no_of_vertices)
		d_successor[tid] = d_successor_copy[tid];
}


////////////////////////////////////////////////////////////////////////////////
// Append Vertex IDs with SuperVertex IDs, Runs for Vertex Length
////////////////////////////////////////////////////////////////////////////////
__global__ void AppendVertexIDsForSplit(unsigned long long int *d_vertex_split, unsigned int *d_successor, unsigned int no_of_vertices)
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<no_of_vertices) {
		unsigned long long int val;
		val = d_successor[tid]; // representative
		val = val<<NO_OF_BITS_TO_SPLIT_ON;
		val |= tid; // u
		d_vertex_split[tid]=val;
	}
}


////////////////////////////////////////////////////////////////////////////////
// Mark New SupervertexID per vertex, Runs for Vertex Length
////////////////////////////////////////////////////////////////////////////////
__global__ void MakeSuperVertexIDPerVertex(unsigned int *d_new_supervertexIDs, unsigned long long int *d_vertex_split, unsigned int *d_vertex_flag,unsigned int no_of_vertices)
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<no_of_vertices)
	{
		unsigned long long int mask = pow(2.0, NO_OF_BITS_TO_SPLIT_ON)-1;
		unsigned long long int vertexid = d_vertex_split[tid]&mask;
		d_vertex_flag[vertexid] = d_new_supervertexIDs[tid];
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Copy New SupervertexID per vertex, resolving read after write inconsistancies, Runs for Vertex Length  // IMPORTANT! RAW
////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void CopySuperVertexIDPerVertex(unsigned int *d_new_supervertexIDs, unsigned int *d_vertex_flag, unsigned int no_of_vertices)
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<no_of_vertices) {
		d_new_supervertexIDs[tid] = d_vertex_flag[tid];
	}
}


////////////////////////////////////////////////////////////////////////////////
// Make flag for Scan, assigning new ids to supervertices, Runs for Vertex Length
////////////////////////////////////////////////////////////////////////////////
__global__ void MakeFlagForScan(unsigned int *d_vertex_flag, unsigned long long int *d_split_input,unsigned int no_of_vertices)
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<no_of_vertices)
	{
		if(tid>0)
		{
			//unsigned long long int mask = pow(2.0,NO_OF_BITS_TO_SPLIT_ON)-1;
			unsigned long long int val = d_split_input[tid-1];
			unsigned long long int supervertexid_prev  = val>>NO_OF_BITS_TO_SPLIT_ON;
			val = d_split_input[tid];
			unsigned long long int supervertexid  = val>>NO_OF_BITS_TO_SPLIT_ON;
			if(supervertexid_prev!=supervertexid)
				d_vertex_flag[tid]=1;
		}
	}
}


////////////////////////////////////////////////////////////////////////////////
// Make flag to assign old vertex ids, Runs for Vertex Length
////////////////////////////////////////////////////////////////////////////////
__global__ void MakeFlagForUIds(unsigned int *d_edge_flag, unsigned int *d_vertex, unsigned int no_of_vertices)
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<no_of_vertices) {
		if(tid>0) {
			unsigned int pointingvertex = d_vertex[tid];
			d_edge_flag[pointingvertex]=1;
		}
	}
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Copy Edge Array to somewhere to resolve read after write inconsistancies, Runs for Edge Length
////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void CopyEdgeArray(unsigned int *d_edge, unsigned int *d_edge_mapping_copy, unsigned int no_of_edges)
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<no_of_edges)
		d_edge_mapping_copy[tid] = d_edge[tid];
}

////////////////////////////////////////////////////////////////////////////////
// Remove self edges based on new supervertex ids, Runs for Edge Length
////////////////////////////////////////////////////////////////////////////////
__global__ void RemoveSelfEdges(unsigned int *d_edge, unsigned int *d_old_uIDs, unsigned int *d_new_supervertexIDs, unsigned int *d_edge_mapping_copy, unsigned int no_of_edges)
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<no_of_edges) {
		unsigned int uid = d_old_uIDs[tid];
		unsigned int vid = d_edge[tid];
		unsigned int usuperid = d_new_supervertexIDs[uid];
		unsigned int vsuperid = d_new_supervertexIDs[vid];
		if(usuperid == vsuperid){
			d_edge_mapping_copy[tid]=INF; //Nullify the edge if both vertices have same supervertex id, do not use the same array for output
		}
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Copy Edge Array Back, resolving read after write inconsistancies, Runs for Edge Length
////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void CopyEdgeArrayBack(unsigned int *d_edge, unsigned int *d_edge_mapping_copy, unsigned int no_of_edges)
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<no_of_edges)
		d_edge[tid]=d_edge_mapping_copy[tid];
}


////////////////////////////////////////////////////////////////////////////////
// Append U,V,W for duplicate edge removal, Runs for Edge Length
////////////////////////////////////////////////////////////////////////////////
__global__ void AppendForDuplicateEdgeRemoval(unsigned long long int *d_appended_uvw, unsigned int *d_edge, unsigned int *d_old_uIDs, unsigned int *d_weight, unsigned int *d_new_supervertexIDs, unsigned int no_of_edges)
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<no_of_edges) {
		unsigned long long int val;
		unsigned int u,v,superuid=INF,supervid=INF;
		u = d_old_uIDs[tid];
		v = d_edge[tid];

		if (v == INF) { // TODO: maybe useful. else (u, INF, w)
			u = INF;
		}

		if(u!=INF && v!=INF) {
			superuid = d_new_supervertexIDs[u];
			supervid = d_new_supervertexIDs[v];
		}
		val = superuid;
		val = val<<NO_OF_BITS_MOVED_FOR_VERTEX_IDS;
		val |= supervid;
		val = val<<(64-(NO_OF_BITS_MOVED_FOR_VERTEX_IDS+NO_OF_BITS_MOVED_FOR_VERTEX_IDS));
		val |= d_weight[tid];
		d_appended_uvw[tid]=val;
	}
}


////////////////////////////////////////////////////////////////////////////////
// Mark the starting edge for each uv combination, Runs for Edge Length
////////////////////////////////////////////////////////////////////////////////
__global__ void MarkEdgesUV(unsigned int *d_edge_flag, unsigned long long int *d_appended_uvw, unsigned int *d_size, unsigned int no_of_edges)
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<no_of_edges) {
		if(tid>0) {
			unsigned long long int test = INF;
			test = test << NO_OF_BITS_MOVED_FOR_VERTEX_IDS;
			test |=INF;
			unsigned long long int test1 = d_appended_uvw[tid]>>(64-(NO_OF_BITS_MOVED_FOR_VERTEX_IDS+NO_OF_BITS_MOVED_FOR_VERTEX_IDS)); // uv[i]
			unsigned long long int test2 = d_appended_uvw[tid-1]>>(64-(NO_OF_BITS_MOVED_FOR_VERTEX_IDS+NO_OF_BITS_MOVED_FOR_VERTEX_IDS)); // uv[i-1]

			if(test1>test2) {
				d_edge_flag[tid]=1;
			}

			if(test1 == test) { // TODO: might be different if change line 334. Not sure if correct now either
				atomicMin(d_size,tid); //also to know the last element in the array, i.e. the size of new edge list
			}
		} else {
			d_edge_flag[tid]=1;
		}
	}
}


///////////////////////////////////////////////////////////////////////////////////////////////
// Compact the edgelist and weight list, keep a mapping for each edge, Runs for d_size Length
///////////////////////////////////////////////////////////////////////////////////////////////
__global__ void CompactEdgeList(unsigned int *d_edge, unsigned int *d_weight, 
								unsigned int *d_old_uIDs, unsigned int *d_edge_flag, unsigned long long int *d_appended_uvw,
								unsigned int *d_pick_array, unsigned int *d_size, 
								unsigned int *d_edge_list_size, unsigned int *d_vertex_list_size)
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<*d_size) {
		if(	d_edge_flag[tid]==1) {
			unsigned long long int UVW = d_appended_uvw[tid];
			unsigned int writepos = d_old_uIDs[tid];
			unsigned long long int mask = pow(2.0,64-(NO_OF_BITS_MOVED_FOR_VERTEX_IDS+NO_OF_BITS_MOVED_FOR_VERTEX_IDS))-1;
			unsigned long long int w  = UVW&mask;
			unsigned long long int test = UVW>>(64-(NO_OF_BITS_MOVED_FOR_VERTEX_IDS+NO_OF_BITS_MOVED_FOR_VERTEX_IDS));
			unsigned long long int mask2 = pow(2.0,NO_OF_BITS_MOVED_FOR_VERTEX_IDS)-1;
			unsigned long long int v = test&mask2;
			unsigned long long int u = test>>NO_OF_BITS_MOVED_FOR_VERTEX_IDS;
			if(u!=INF && v!=INF) {
				//Copy the edge_mapping into a temporary array, used to resolve read after write inconsistancies
				d_pick_array[writepos]=u; // reusing this to store u's
				d_edge[writepos] = v;
				d_weight[writepos] = w;
				//max writepos will give the new edge list size
				atomicMax(d_edge_list_size,(writepos+1));
				atomicMax(d_vertex_list_size,(v+1));
				// Orig: atomicMax(d_vertex_list_size,(u+1)); //how can max(v) be > max(u), error!!!!! TODO check this whole thing
			}
		}		
	}
}

////////////////////////////////////////////////////////////////////////////////
//Copy the temporary array to the actual mapping array, Runs for Edge length
////////////////////////////////////////////////////////////////////////////////
__global__ void CopyEdgeMap(unsigned int *d_edge_mapping, unsigned int *d_edge_mapping_copy, unsigned int no_of_edges)
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<no_of_edges)
		d_edge_mapping[tid] = d_edge_mapping_copy[tid]; 
}

////////////////////////////////////////////////////////////////////////////////
//Make Flag for Vertex List Compaction, Runs for Edge length
////////////////////////////////////////////////////////////////////////////////
__global__ void MakeFlagForVertexList(unsigned int *d_pick_array, unsigned int *d_edge_flag, unsigned int no_of_edges)
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<no_of_edges) {
		if(tid>0) {
			if(d_pick_array[tid] != d_pick_array[tid-1]) { //This line may be causing problems TODO: maybe != such as in python code but should be fine. Change back to > for orig
				d_edge_flag[tid]=1;
			}
		} else {
			d_edge_flag[tid]=1;
			//atomicMax(d_edge_list_size,(tid));
		}
	}
}

////////////////////////////////////////////////////////////////////////////////
//Vertex List Compaction, Runs for Edge length
////////////////////////////////////////////////////////////////////////////////
__global__ void MakeVertexList(unsigned int *d_vertex, unsigned int *d_pick_array, unsigned int *d_edge_flag, unsigned int no_of_edges)
{
	unsigned int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if(tid<no_of_edges) {
		if(d_edge_flag[tid]==1) {
			unsigned int writepos=d_pick_array[tid]; //get the u value
			d_vertex[writepos]=tid; //write the index to the u'th value in the array to create the vertex list
			//atomicMax(d_vertex_list_size,(writepos+1));
		}
	}
}


#endif // #ifndef _KERNELS_H_