#include "hip/hip_runtime.h"
/***********************************************************************************
  Implementing Minimum Spanning Tree on CUDA using primitive operations for the 
  algorithm given in "Fast Minimum Spanning Tree Computation", by Pawan Harish, 
  P.J. Narayanan, Vibhav Vineet, and Suryakant Patidar.

  Chapter 7 of Nvidia GPU Computing Gems, Jade Edition, 2011.
  
  Copyright (c) 2011 International Institute of Information Technology - Hyderabad. 
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for 
  educational purpose is hereby granted without fee, provided that the above copyright 
  notice and this permission notice appear in all copies of this software and that you do 
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND, EXPRESSED, IMPLIED OR 
  OTHERWISE.

  Created by: Pawan Harish.
  Split Implementation by: Suryakant Patidar and Parikshit Sakurikar.
 ************************************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
// #include <cutil.h> // Removed, should just have been for CUDA_SAFE_CALL and CUDA_CUT_CALL which has been deprecated

// includes, kernels
#include "Kernels.cu"
// #include <cudpp.h> 
#include "splitFuncs.h"
splitSort sp;

// Thrust stuff
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/transform.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/functional.h>

////////////////////////////////////////////////
// Variables
////////////////////////////////////////////////
int no_of_vertices,no_of_vertices_orig;				//Actual input graph sizes
int no_of_edges, no_of_edges_orig;					//Current graph sizes
int *h_edge, *h_vertex, *h_weight;					//Graph held in these variables at the host end, 3 arrays for compressed adjacency list format
int *d_edge, *d_vertex, *d_weight;					//Graph held in these variables at the device end
int *d_segmented_min_scan_input;					//Input to the Segmented Min Scan, appended array of weights and edge IDs (X in paper)
int *d_segmented_min_scan_output;					//Output of the Segmented Min Scan, minimum weight outgoing edge as (weight|to_vertex_id elements) for each verte
unsigned int *d_edge_flag;							//Flag for the segmented min scan
unsigned int *d_edge_flag_thrust;					//NEW! Flag for the segmented min scan in thrust Needs to be 000111222 instead of 100100100
unsigned int *d_vertex_flag;						//Flag for the scan input for supervertex ID generation
unsigned int *d_output_MST;							//Final output, marks 1 for selected edges in MST, 0 otherwise
int *d_pick_array;									//PickArray for each edge. For each edge from u, segmented scan location min edge going out of u if not removed. Else -1 if removed (representative doesn't add edges)
int *d_successor;									//Successor Array, S
int *d_successor_copy;
bool *d_succchange;									//Variable to check for execution while propagating representative vertex IDs
unsigned long long int *d_vertex_split;				//Input to the split function
unsigned long long int *d_vertex_split_scratchmem;	//Scratch memory to the split function
unsigned long long int *d_vertex_split_rank;		//Ranking arrary to the split function
unsigned long long int *d_vertex_rank_scratchmem;	//Scratch memory to the split function
unsigned int *d_new_supervertexIDs;					//new supervertex ids after scanning older IDs
unsigned int *d_old_uIDs;							//old ids, stored per edge, needed to remove self edges (orig ID of source vertex u for each edge(weight|dest_vertex_id_v))
unsigned long long int *d_appended_uvw;				//Appended u,v,w array for duplicate edge removal
unsigned long long int *d_edge_split_scratchmem;	//Scratch memory to the split function
unsigned long long int *d_edge_rank;				//Rank array for duplicate edge removal
unsigned long long int *d_edge_rank_scratchmem;		//Scratch memory to the split function
unsigned int *d_size;								//Stores amount of edges
unsigned int *d_edge_mapping;
unsigned int *d_edge_mapping_copy;
int	*d_edge_list_size;
int	*d_vertex_list_size;

unsigned int *h_output_MST_test;					//Final output on host, marks 1 for selected edges in MST, 0 otherwise
unsigned long long int *h_vertex_split_rank_test;	//Used to copy split rank to device, initially 1 2 3 4 5 ...
unsigned long long int *h_edge_rank_test;			//Used to copy edge rank to device, initially 1 2 3 4 5 ...

//CUDPP Scan and Segmented Scan Variables
// CUDPPHandle			segmentedScanPlan_min, scanPlan_add ;   // DONE: remove
// CUDPPConfiguration	config_segmented_min, config_scan_add ; // DONE: remove

// Debug helper function
void printIntArr(int* d_data, int n_elements) {
	int* h_data = (int *)malloc(sizeof(int)*n_elements);
	hipMemcpy(h_data, d_data, sizeof(int) * n_elements, hipMemcpyDeviceToHost);
	for (int i = 0; i < n_elements; i++) {
		printf("%d ",h_data[i]);
	}
	printf("\n");
	free(h_data);
}

void printXArr(int* d_data, int n_elements) {
	int* h_data = (int *)malloc(sizeof(int)*n_elements);
	hipMemcpy(h_data, d_data, sizeof(int) * n_elements, hipMemcpyDeviceToHost);
	for (int i = 0; i < n_elements; i++) {
		int mask = pow(2.0,MOVEBITS)-1;
		int vertex = h_data[i]&mask;
		int weight = h_data[i]>>MOVEBITS;
		printf("%d|%d ",weight, vertex);
	}
	printf("\n");
	free(h_data);
}

void printUVWArr(unsigned long long int *d_data, int n_elements) {
	unsigned long long int* h_data = (unsigned long long int *)malloc(sizeof(unsigned long long int)*n_elements);
	hipMemcpy(h_data, d_data, sizeof(unsigned long long int) * n_elements, hipMemcpyDeviceToHost);
	for (int i = 0; i < n_elements; i++) {
		unsigned long long int UVW = h_data[i];
		unsigned long long int mask = pow(2.0,64-(NO_OF_BITS_MOVED_FOR_VERTEX_IDS+NO_OF_BITS_MOVED_FOR_VERTEX_IDS))-1;
		unsigned long long int w  = (int) UVW&mask;
		unsigned long long int test = UVW>>(64-(NO_OF_BITS_MOVED_FOR_VERTEX_IDS+NO_OF_BITS_MOVED_FOR_VERTEX_IDS));
		unsigned long long int mask2 = pow(2.0,NO_OF_BITS_MOVED_FOR_VERTEX_IDS)-1;
		unsigned long long int v = test&mask2;
		unsigned long long int u = test>>NO_OF_BITS_MOVED_FOR_VERTEX_IDS;
		printf("%llu|%llu|%llu ",u, v, w);
	}
	printf("\n");
	free(h_data);
}

void printUIntArr(unsigned int* d_data, int n_elements) {
	unsigned int* h_data = (unsigned int *)malloc(sizeof(unsigned int)*n_elements);
	hipMemcpy(h_data, d_data, sizeof(unsigned int) * n_elements, hipMemcpyDeviceToHost);
	for (int i = 0; i < n_elements; i++) {
		printf("%u ",h_data[i]);
	}
	printf("\n");
	free(h_data);
}

void printULongArr(long* d_data, int n_elements) {
	unsigned long* h_data = (unsigned long *)malloc(sizeof(unsigned long)*n_elements);
	hipMemcpy(h_data, d_data, sizeof(unsigned long) * n_elements, hipMemcpyDeviceToHost);
	for (int i = 0; i < n_elements; i++) {
		printf("%lu ",h_data[i]);
	}
	printf("\n");
	free(h_data);
}

void printLongArr(long* d_data, int n_elements) {
	long* h_data = (long *)malloc(sizeof(long)*n_elements);
	hipMemcpy(h_data, d_data, sizeof(long) * n_elements, hipMemcpyDeviceToHost);
	for (int i = 0; i < n_elements; i++) {
		printf("%ld ",h_data[i]);
	}
	printf("\n");
	free(h_data);
}

void printInt(int *d_val) {
	int h_val;
	hipMemcpy( &h_val, d_val, sizeof(int), hipMemcpyDeviceToHost);
	printf("%d", h_val);
}

void printUInt(unsigned int *d_val) {
	unsigned int h_val;
	hipMemcpy( &h_val, d_val, sizeof(unsigned int), hipMemcpyDeviceToHost);
	printf("%u", h_val);
}

////////////////////////////////////////////////
// Read the Graph in our format (Compressed adjacency list)
////////////////////////////////////////////////
void ReadGraph(char *filename)
{
	FILE *fp;
	fp = fopen(filename,"r");

	// Read number of vertices
	fscanf(fp,"%d",&no_of_vertices); 
	h_vertex = (int*)malloc(sizeof(int)*no_of_vertices);
	no_of_vertices_orig = no_of_vertices ;

	// Read V (start index edges for each vertex)
	int start, index ;
	for ( int i = 0 ; i < no_of_vertices ; i++ )
	{
		fscanf(fp,"%d %d",&start, &index) ; // Format: start edges, ignored
		h_vertex[i] = start ;
	}

	// Read "root" of graph (unused)
	int source = 0 ;
	fscanf(fp,"%d",&source);

	// Read number of edges
	fscanf(fp,"%d",&no_of_edges);
	no_of_edges_orig = no_of_edges ;

	// Read edges
	h_edge = (int*) malloc (sizeof(int)*no_of_edges);
	h_weight = (int*) malloc (sizeof(int)*no_of_edges);

	int edgeindex, edgeweight ;
	for( int i = 0 ; i < no_of_edges ; i++ )
	{
		fscanf(fp,"%d %d",&edgeindex, &edgeweight); // Format: to, weight
		h_edge[i] = edgeindex ;
		h_weight[i] =  edgeweight ;
	}
	fclose(fp);

	printf("File read successfully %d %d\n",no_of_vertices, no_of_edges);
}


////////////////////////////////////////////////
// Allocate and Initialize Arrays
////////////////////////////////////////////////
void Init()
{

	/*
	//Setting the CUDPP configurations for SCAN and SEGMENTED MIN SCAN // DONE: remove
	// Min inclusive segmented scan on ints from start to end.
	config_segmented_min.algorithm = CUDPP_SEGMENTED_SCAN;
	config_segmented_min.op = CUDPP_MIN;
	config_segmented_min.datatype = CUDPP_INT;
	config_segmented_min.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;

	// Summation scan on ints from start to end. Each summation sums elements up to the current element i
	config_scan_add.algorithm = CUDPP_SCAN;
	config_scan_add.op = CUDPP_ADD;
	config_scan_add.datatype = CUDPP_INT;
	config_scan_add.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;
	*/

	//Copy the Graph to Device
	hipMalloc( (void**) &d_edge, sizeof(int)*no_of_edges);
	hipMalloc( (void**) &d_vertex, sizeof(int)*no_of_vertices);
	hipMalloc( (void**) &d_weight, sizeof(int)*no_of_edges);
	hipMemcpy( d_edge, h_edge, sizeof(int)*no_of_edges, hipMemcpyHostToDevice);
	hipMemcpy( d_vertex, h_vertex, sizeof(int)*no_of_vertices, hipMemcpyHostToDevice);
	hipMemcpy( d_weight, h_weight, sizeof(int)*no_of_edges, hipMemcpyHostToDevice);
	printf("Graph Copied to Device\n");

	//Allocate memory for other arrays
	hipMalloc( (void**) &d_segmented_min_scan_input, sizeof(int)*no_of_edges);
	hipMalloc( (void**) &d_segmented_min_scan_output, sizeof(int)*no_of_edges);
	hipMalloc( (void**) &d_edge_flag, sizeof(unsigned int)*no_of_edges);
	hipMalloc( (void**) &d_edge_flag_thrust, sizeof(unsigned int)*no_of_edges);
	hipMalloc( (void**) &d_pick_array, sizeof(unsigned int)*no_of_edges);
	hipMalloc( (void**) &d_successor,sizeof(int)*no_of_vertices);
	hipMalloc( (void**) &d_successor_copy,sizeof(int)*no_of_vertices);
	hipMalloc( (void**) &d_output_MST, sizeof(unsigned int)*no_of_edges);
	
	//Clear Output MST array
	unsigned int *h_test=(unsigned int*)malloc(sizeof(unsigned int)*no_of_edges);
	for(int i=0;i<no_of_edges;i++)h_test[i]=0;
	hipMemcpy( d_output_MST, h_test, sizeof(unsigned int)*no_of_edges, hipMemcpyHostToDevice);

	hipMalloc( (void**) &d_succchange, sizeof(bool));
	hipMalloc( (void**) &d_vertex_split, sizeof(unsigned long long int)*no_of_vertices);
	hipMalloc( (void**) &d_vertex_split_scratchmem, sizeof(unsigned long long int)*no_of_vertices);
	hipMalloc( (void**) &d_vertex_flag, sizeof(unsigned int)*no_of_vertices);
	hipMalloc( (void**) &d_new_supervertexIDs, sizeof(unsigned int)*no_of_vertices);
	hipMalloc( (void**) &d_old_uIDs, sizeof(unsigned int)*no_of_edges);
	hipMalloc( (void**) &d_appended_uvw, sizeof(unsigned long long int)*no_of_edges);
	hipMalloc( (void**) &d_edge_split_scratchmem, sizeof(unsigned long long int)*no_of_edges);
	hipMalloc( (void**) &d_size, sizeof(unsigned int));
	hipMalloc( (void**) &d_edge_mapping, sizeof(unsigned int)*no_of_edges); 
	hipMalloc( (void**) &d_edge_mapping_copy, sizeof(unsigned int)*no_of_edges); 
	//Initiaize the d_edge_mapping array
	for(int i=0;i<no_of_edges;i++)h_test[i]=i;
	hipMemcpy( d_edge_mapping, h_test, sizeof(unsigned int)*no_of_edges, hipMemcpyHostToDevice);

	hipMalloc( (void**) &d_edge_list_size, sizeof(int));
	hipMalloc( (void**) &d_vertex_list_size, sizeof(int));
	

	h_output_MST_test = (unsigned int*)malloc(sizeof(unsigned int)*no_of_edges);

	hipMalloc( (void**) &d_vertex_split_rank, sizeof(unsigned long long int)*no_of_vertices);
	hipMalloc( (void**) &d_vertex_rank_scratchmem, sizeof(unsigned long long int)*no_of_vertices);
	h_vertex_split_rank_test=(unsigned long long int*)malloc(sizeof(unsigned long long int)*no_of_vertices);
	for(int i=0;i<no_of_vertices;i++)h_vertex_split_rank_test[i]=i;
	hipMemcpy( d_vertex_split_rank, h_vertex_split_rank_test, sizeof(unsigned long long int)*no_of_vertices, hipMemcpyHostToDevice);

	hipMalloc( (void**) &d_edge_rank, sizeof(unsigned long long int)*no_of_edges);
	hipMalloc( (void**) &d_edge_rank_scratchmem, sizeof(unsigned long long int)*no_of_edges);
	//Initialize the edge rank list
	h_edge_rank_test=(unsigned long long int*)malloc(sizeof(unsigned long long int)*no_of_edges);
	for(int i=0;i<no_of_edges;i++)h_edge_rank_test[i]=i;
	hipMemcpy( d_edge_rank, h_edge_rank_test, sizeof(unsigned long long int)*no_of_edges, hipMemcpyHostToDevice);

	free(h_test);
}


////////////////////////////////////////////////
// Helper function to set the grid sizes
////////////////////////////////////////////////
void SetGridThreadLen(int number, int *num_of_blocks, int *num_of_threads_per_block)
{
	*num_of_blocks = 1;
	*num_of_threads_per_block = number;

	//Make execution Parameters according to the number of nodes
	//Distribute threads across multiple Blocks if necessary
	if(number>MAX_THREADS_PER_BLOCK)
	{
		*num_of_blocks = (int)ceil(number/(double)MAX_THREADS_PER_BLOCK); 
		*num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
	}
}


////////////////////////////////////////////////
// Perform Our Recursive MST Algorithm
////////////////////////////////////////////////
void HPGMST()
{
	
	//Reinitialize the ranking arrays
	hipMemcpy( d_vertex_split_rank, h_vertex_split_rank_test, sizeof(unsigned long long int)*no_of_vertices, hipMemcpyHostToDevice);
	hipMemcpy( d_edge_rank, h_edge_rank_test, sizeof(unsigned long long int)*no_of_edges, hipMemcpyHostToDevice);
	
	//Make both CUDA grids needed for execution, no_of_vertices and no_of_edges length sizes
	int num_of_blocks, num_of_threads_per_block;

	//Grid and block sizes so each edge has one thread (fit as much threads as possible in one block)
	SetGridThreadLen(no_of_edges, &num_of_blocks, &num_of_threads_per_block);
	dim3 grid_edgelen(num_of_blocks, 1, 1);
	dim3 threads_edgelen(num_of_threads_per_block, 1, 1);

	// Grid and block sizes so each vertex has one thread (fit as much threads as possible in one block)
	SetGridThreadLen(no_of_vertices, &num_of_blocks, &num_of_threads_per_block);
	dim3 grid_vertexlen(num_of_blocks, 1, 1);
	dim3 threads_vertexlen(num_of_threads_per_block, 1, 1);


	/*
	 * A. Find minimum weighted edge
	 */

	// 1. Append weight w and outgoing vertex v per edge into a single array, X.
    // Normally 8-10 bit for weight, 20-22 bits for ID. Because of 32 bit limitation CUDPP scan primitive, TODO: probably not relevant anymore
	//Append in Parallel on the Device itself, call the append kernel
	AppendKernel_1<<< grid_edgelen, threads_edgelen, 0>>>(d_segmented_min_scan_input, d_weight, d_edge, no_of_edges);

	// d_edge_flag = F
	//Create the Flag needed for segmented min scan operation, similar operation will also be used at other places
	ClearArray<<< grid_edgelen, threads_edgelen, 0>>>( d_edge_flag, no_of_edges );


	// 2. Divide the edge-list, E, into segments with 1 indicating the start of each segment and 0 otherwise, store this in flag array F.
	// Mark the segments for the segmented min scan
	MakeFlag_3<<< grid_vertexlen, threads_vertexlen, 0>>>( d_edge_flag, d_vertex, no_of_vertices);


	// 3. Perform segmented min scan on X with F indicating segments to find minimum outgoing edge-index per vertex. Min can be found at end of each segment after scan // DONE: change to thrust
	// Prepare key vector for thrust
	thrust::inclusive_scan(thrust::device, d_edge_flag, d_edge_flag + no_of_edges, d_edge_flag_thrust);

	//printf("X:\n");
	//printUIntArr(d_edge_flag, no_of_edges);
	//printXArr(d_segmented_min_scan_input, no_of_edges);

	// Min inclusive segmented scan on ints from start to end.
	thrust::equal_to<unsigned int> binaryPred;
	thrust::minimum<int> binaryOp;
	thrust::inclusive_scan_by_key(thrust::device, d_edge_flag_thrust, d_edge_flag_thrust + no_of_edges, d_segmented_min_scan_input, d_segmented_min_scan_output, binaryPred, binaryOp);

	//printXArr(d_segmented_min_scan_output, no_of_edges);
	//printf("\n");

	/*
	cudppPlan(&segmentedScanPlan_min, config_segmented_min, no_of_edges, 1, 0 ); //Make the segmented min scan plan
	cudppSegmentedScan(segmentedScanPlan_min, d_segmented_min_scan_output, d_segmented_min_scan_input, (const unsigned int*)d_edge_flag, no_of_edges);
	cudppDestroyPlan(segmentedScanPlan_min);
	*/

	/*
	 * B. Finding and removing cycles
	 */

	// 4. Find the successor of each vertex and add to successor array, S.
	MakeSucessorArray<<< grid_vertexlen, threads_vertexlen, 0>>>(d_successor, d_vertex, d_segmented_min_scan_output, no_of_vertices, no_of_edges);


	// 5. Remove cycle making edges using S, and identify representatives vertices.
	RemoveCycles<<< grid_vertexlen, threads_vertexlen, 0>>>(d_successor,no_of_vertices);


	/*
	 * Can possibly be moved in future once remove pick array stuff
	 */
	//Scan the flag to get u at every edge, use the u to index d_vertex to get the last entry in each segment
	//U at every edge will also be useful later in the algorithm.

	// Set F[0] = 0. F is the same as previous F but first element is 0 instead of 1
	ClearArray<<< grid_edgelen, threads_edgelen, 0>>>( d_edge_flag, no_of_edges );
	MakeFlagForUIds<<< grid_vertexlen, threads_vertexlen, 0>>>(d_edge_flag, d_vertex,no_of_vertices); 

	// 10.2 Create vector indicating source vertex u for each edge // DONE: change to thrust
	thrust::inclusive_scan(thrust::device, d_edge_flag, d_edge_flag + no_of_edges, d_old_uIDs);

	//printf("Expanded U:\n");
	//printUIntArr(d_old_uIDs, no_of_edges);

	/*
	cudppPlan(&scanPlan_add, config_scan_add, no_of_edges , 1, 0);// Create scanplan 
	cudppScan(scanPlan_add, d_old_uIDs, d_edge_flag, no_of_edges);
	cudppDestroyPlan(scanPlan_add);
	*/

	//Fill the pick array using the above and the d_successor array TODO REMOVE: not needed for image segmentation
	ClearArray<<< grid_edgelen, threads_edgelen, 0>>>((unsigned int*)d_pick_array, no_of_edges);
 	MakePickArray<<< grid_edgelen, threads_edgelen, 0>>>(d_pick_array,d_successor,d_vertex,d_old_uIDs,no_of_vertices,no_of_edges);

	//Mark the Remaining Edges in the Output MST array. This not so elegant. TODO REMOVE: not needed for image segmentation
	//Because we do not know which edge index was selected by the segmented min scan,
	//we check each edge with the selected edges and write to output if same
	MarkOutputEdges<<< grid_edgelen, threads_edgelen, 0>>>(d_pick_array, d_segmented_min_scan_input, d_segmented_min_scan_output, d_output_MST,d_edge_mapping,no_of_edges);


	/*
	 * C. Merging vertices and assigning IDs to supervertices
	 */

	// 7. Propagate Representative Vertex IDs to all vertices iteratively using pointer Doubling until no change occures in Successor Array
	bool succchange;
	do
	{
		succchange=false; //if no thread changes this value, the loop stops
		hipMemcpy( d_succchange, &succchange, sizeof(bool), hipMemcpyHostToDevice);
		//Reusing Vertex Flag
		SuccToCopy<<< grid_vertexlen, threads_vertexlen, 0>>>(d_successor, d_successor_copy, no_of_vertices); // for conflicts
		PropagateRepresentativeID<<< grid_vertexlen, threads_vertexlen, 0>>>(d_successor, d_successor_copy, d_succchange,no_of_vertices);
		CopyToSucc<<< grid_vertexlen, threads_vertexlen, 0>>>(d_successor, d_successor_copy, no_of_vertices); // for conflicts

		hipMemcpy( &succchange, d_succchange, sizeof(bool), hipMemcpyDeviceToHost);
	}
	while(succchange);


	// 8. Append successor array’s entries with its index to form a list, L. Representative left, vertex id right, 64 bit. TODO look into needed sizes
	//    Important! L different order than in python code!
	//    Append Vertex Ids with SuperVertexIDs
	AppendVertexIDsForSplit<<< grid_vertexlen, threads_vertexlen, 0>>>(d_vertex_split, d_successor,no_of_vertices);


	//9. Split L, create flag over split output and scan the flag to find new ids per vertex, store new ids in C.
    // 9.1 Split L using representative as key. In parallel using a split of O(V) with log(V) bit key size.
    //     split based on supervertex IDs using 64 bit version of split
	sp.split(d_vertex_split, d_vertex_split_rank, d_vertex_split_scratchmem, d_vertex_rank_scratchmem, no_of_vertices, NO_OF_BITS_TO_SPLIT_ON, 0); 	// TODO: maybe can just use sort.

	// 9.2 Create flag for assigning new vertex IDs based on difference in supervertex IDs
	//     first element not flagged so that can use simple sum for scan
	ClearArray<<< grid_vertexlen, threads_vertexlen, 0>>>( d_vertex_flag, no_of_vertices);
	MakeFlagForScan<<< grid_vertexlen, threads_vertexlen, 0>>>(d_vertex_flag, d_vertex_split, no_of_vertices);
 
	// 9.3 Scan flag to assign new IDs to supervertices, Using a scan on O(V) elements // DONE: change to thrust
	//printf("New supervertex ids:\n");
	thrust::inclusive_scan(thrust::device, d_vertex_flag, d_vertex_flag + no_of_vertices, d_new_supervertexIDs);
	//printUIntArr(d_new_supervertexIDs, no_of_vertices);
	/*
	cudppPlan(&scanPlan_add, config_scan_add, no_of_vertices , 1, 0);
	cudppScan(scanPlan_add, d_new_supervertexIDs, d_vertex_flag, no_of_vertices);
	cudppDestroyPlan(scanPlan_add);
	*/


	/*
	 * D. Removing self edges
	 */

	// 10.1 Create mapping from each original vertex ID to its new supervertex ID so we can lookup supervertex IDs directly
	MakeSuperVertexIDPerVertex<<< grid_vertexlen, threads_vertexlen, 0>>>(d_new_supervertexIDs, d_vertex_split, d_vertex_flag, no_of_vertices);
	CopySuperVertexIDPerVertex<<< grid_vertexlen, threads_vertexlen, 0>>>(d_new_supervertexIDs, d_vertex_flag, no_of_vertices); // for concurrent access problems
	
	//Remove Self Edges from the edge-list
	// 11. Remove edge from edge-list if u, v have same supervertex id (remove self edges)
	CopyEdgeArray<<< grid_edgelen, threads_edgelen, 0>>>(d_edge,d_edge_mapping_copy, no_of_edges); // for conflicts
	RemoveSelfEdges<<< grid_edgelen, threads_edgelen, 0>>>(d_edge, d_old_uIDs, d_new_supervertexIDs, d_vertex_split_rank, d_edge_mapping_copy, no_of_edges);
	CopyEdgeArrayBack<<< grid_edgelen, threads_edgelen, 0>>>(d_edge,d_edge_mapping_copy, no_of_edges); // for conflicts


	/*
	 * D. Removing duplicate edges. This is not mandatory, however, reduces the edge-list size significantly. You may choose to use it once in the initial 
	 *    iterations of the algorithm, later edge-list size is small anyways so not much is gained by doing this in later iterations
	 */


	// 12. Remove the largest duplicate edges using split over new u,v and w.
	// 12.1 Append supervertex ids of u and v along with weight w into single 64 bit array (u 24 bit, v 24 bit, w 16 bit)
	AppendForDuplicateEdgeRemoval<<< grid_edgelen, threads_edgelen, 0>>>(d_appended_uvw, d_edge, d_old_uIDs, d_weight,d_new_supervertexIDs, no_of_edges);

	//12.2 Split the array using {u,v) as the key. Pick First distinct (u,v) entry as the edge, nullify others
	//     You may also replace the split with sort, but we could not find a 64-bit sort.
	// sp.split(d_appended_uvw, d_edge_rank, d_edge_split_scratchmem, d_edge_rank_scratchmem, no_of_edges, NO_OF_BITS_TO_SPLIT_ON_UVW, 0);
	// thrust::sort(thrust::device, d_appended_uvw, d_appended_uvw + no_of_edges); // TODO: check

	thrust::sort_by_key(thrust::device, d_appended_uvw, d_appended_uvw + no_of_edges, d_edge_rank); // TODO: can just use sort for segmentation

	
	//Pick the first distinct (u,v) combination, mark these edges and compact
	// 12.3 Create flag indicating smallest edges, 0 for larger duplicates
	ClearArray<<< grid_edgelen, threads_edgelen, 0>>>( d_edge_flag, no_of_edges ); // d_edge_flag = F3
	unsigned int dsize=no_of_edges; //just make sure
	hipMemcpy( d_size, &dsize, sizeof(unsigned int), hipMemcpyHostToDevice);
	MarkEdgesUV<<< grid_edgelen, threads_edgelen, 0>>>(d_edge_flag, d_appended_uvw, d_size, no_of_edges);

	//printf("UVW:");
	//printUVWArr(d_appended_uvw, no_of_edges);
	//printUIntArr(d_edge_flag, no_of_edges);

	//printf("New edge size: ");
	//printUInt(d_size);

	// 13. Compact and create new edge and weight list
	// 13.1 Scan the flag array to know where to write the value in new edge and weight lists // DONE: change to thrust
	thrust::inclusive_scan(thrust::device, d_edge_flag, d_edge_flag + no_of_edges, d_old_uIDs);
	/*
	cudppPlan(&scanPlan_add, config_scan_add, no_of_edges, 1, 0);
	cudppScan(scanPlan_add, d_old_uIDs, d_edge_flag, no_of_edges); //Just reusing the d_old_uIDs array for compating
	cudppDestroyPlan(scanPlan_add);
	*/

	// NEW! Maybe not needed. Make sure new locations start from 0 instead of 1. TODO: can be done more efficient in case works
	thrust::transform(thrust::device,
				  d_old_uIDs,
                  d_old_uIDs + no_of_edges,
                  thrust::make_constant_iterator(1),
                  d_old_uIDs,
                  thrust::minus<unsigned int>());

	//printf("Write positions:");

	//******************************************************************************************
	//Do all clearing in a single kernel, no need to call multiple times, OK for testing only TODO
	//******************************************************************************************
	ClearArray<<< grid_edgelen, threads_edgelen, 0>>>((unsigned int*)d_edge, no_of_edges );
	ClearArray<<< grid_edgelen, threads_edgelen, 0>>>((unsigned int*)d_weight, no_of_edges );
	ClearArray<<< grid_edgelen, threads_edgelen, 0>>>( d_edge_mapping_copy, no_of_edges);
	ClearArray<<< grid_edgelen, threads_edgelen, 0>>>( (unsigned int*)d_pick_array, no_of_edges); //Reusing the Pick Array
	int negative=0;
	hipMemcpy( d_edge_list_size, &negative, sizeof( int), hipMemcpyHostToDevice);
	hipMemcpy( d_vertex_list_size, &negative, sizeof( int), hipMemcpyHostToDevice);
	
	//Compact the edge and weight lists
	unsigned int validsize=0;
	hipMemcpy( &validsize, d_size, sizeof(unsigned int), hipMemcpyDeviceToHost);

	//Make a new grid for valid entries in the d_edge_flag array
	SetGridThreadLen(validsize, &num_of_blocks, &num_of_threads_per_block);
	dim3 grid_validsizelen(num_of_blocks, 1, 1);
	dim3 threads_validsizelen(num_of_threads_per_block, 1, 1);

	// 13.2 Compact and create new edge and weight list
	//      Reusing d_pick_array for storing the u ids
	CompactEdgeList<<< grid_validsizelen, threads_validsizelen, 0>>>(d_edge, d_weight, d_edge_mapping, d_edge_mapping_copy, d_old_uIDs, d_edge_flag, d_appended_uvw, d_pick_array, d_edge_rank, d_size, d_edge_list_size, d_vertex_list_size);
	CopyEdgeMap<<< grid_edgelen, threads_edgelen, 0>>>(d_edge_mapping, d_edge_mapping_copy,no_of_edges);

	// 14. Build the vertex list from the newly formed edge list
	ClearArray<<< grid_edgelen, threads_edgelen, 0>>>( d_edge_flag, no_of_edges);
	ClearArray<<< grid_vertexlen, threads_vertexlen, 0>>>((unsigned int*)d_vertex, no_of_vertices);

	//14.1 Create flag based on difference in u on the new edge list (based on diffference of u ids)
	MakeFlagForVertexList<<< grid_edgelen, threads_edgelen, 0>>>(d_pick_array, d_edge_flag, no_of_edges); // d_edge_flag = F4

	// 14.2 Build the vertex list from the newly formed edge list
	MakeVertexList<<< grid_edgelen, threads_edgelen, 0>>>(d_vertex, d_pick_array, d_edge_flag, no_of_edges);
	
	hipMemcpy( &no_of_edges, d_edge_list_size, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy( &no_of_vertices, d_vertex_list_size, sizeof(int), hipMemcpyDeviceToHost);

}



////////////////////////////////////////////////
//Free All memory from Host and Device
////////////////////////////////////////////////
void FreeMem()
{
	free(h_edge);
	free(h_vertex);
	free(h_weight);
	free(h_output_MST_test);
	free(h_vertex_split_rank_test);
	free(h_edge_rank_test);
	hipFree(d_edge);
	hipFree(d_vertex);
	hipFree(d_weight);
	hipFree(d_segmented_min_scan_input);
	hipFree(d_segmented_min_scan_output);
	hipFree(d_edge_flag);
	hipFree(d_pick_array);
	hipFree(d_successor);
	hipFree(d_successor_copy);
	hipFree(d_output_MST);
	hipFree(d_succchange);
	hipFree(d_vertex_split);
	hipFree(d_vertex_split_scratchmem);
	hipFree(d_vertex_flag);
	hipFree(d_new_supervertexIDs);
	hipFree(d_old_uIDs);
	hipFree(d_edge_split_scratchmem);
	hipFree(d_size);
	hipFree(d_edge_mapping);
	hipFree(d_edge_mapping_copy);
	hipFree(d_edge_list_size);
	hipFree(d_vertex_list_size);
	hipFree(d_vertex_split_rank);
	hipFree(d_vertex_rank_scratchmem);
	hipFree(d_edge_rank);
	hipFree(d_edge_rank_scratchmem);
	hipFree(d_appended_uvw);
}



////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
	if(argc<2) {
		printf("Specify an Input Graph\n");
		exit(1);
	}

	ReadGraph(argv[1]);
	Init();
	//printf("\n\n");

	/*unsigned int	timer;
	cutCreateTimer( &timer);	
	cutStartTimer( timer);*/
	//Perform Our MST algorhtm
	do
	{
	    HPGMST();
	    //printf("\n");
	}
	while(no_of_vertices>1);
	/*cutStopTimer( timer);
	printf("\n=================== Time taken To perform MST :: %3.3f ms===================\n",cutGetTimerValue(timer));*/
	//printf("\n\nOutputs:\n");

	//Copy the Final MST array to the CPU memory, a 1 at the index means that edge was selected in the MST, 0 otherwise.
	//It should be noted that each edge has an opposite edge also, out of whcih only one is selected in this output.
	//So total number of 1s in this array must be equal to no_of_vertices_orig-1.
	hipMemcpy( h_output_MST_test, d_output_MST, sizeof(unsigned int)*no_of_edges_orig, hipMemcpyDeviceToHost);
	int k=0;
	int weight=0;
	//printf("\n\nSelected Edges in MST...\n\n");
	for(int i=0;i<no_of_edges_orig;i++)
		if(h_output_MST_test[i]==1)
			{
				printf("%d %d\n",h_edge[i],h_weight[i]);
				k++;
				weight+=h_weight[i];
			}
		//else {
		//	printf("not %d %d\n",h_edge[i],h_weight[i]);
		//}
	printf("\nNumber of edges in MST, must be=(no_of_vertices-1)): %d,(%d)\nTotal MST weight: %d\n",k, no_of_vertices_orig,weight);
	
	FreeMem();
}

