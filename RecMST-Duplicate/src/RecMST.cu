#include "hip/hip_runtime.h"
/***********************************************************************************
  Implementing Minimum Spanning Tree on CUDA using primitive operations for the 
  algorithm given in "Fast Minimum Spanning Tree Computation", by Pawan Harish, 
  P.J. Narayanan, Vibhav Vineet, and Suryakant Patidar.

  Chapter 7 of Nvidia GPU Computing Gems, Jade Edition, 2011.
  
  Copyright (c) 2011 International Institute of Information Technology - Hyderabad. 
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for 
  educational purpose is hereby granted without fee, provided that the above copyright 
  notice and this permission notice appear in all copies of this software and that you do 
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND, EXPRESSED, IMPLIED OR 
  OTHERWISE.

  Created by: Pawan Harish.
  Split Implementation by: Suryakant Patidar and Parikshit Sakurikar.
 ************************************************************************************/

/***********************************************************************************
  General bit size info
  ---------------------
  Vertex ID 25 bit -> 33.554.432 
  - 8K image: 7680 × 4320 = 33.177.600 pixels -> supports 1 8K images

  Weight 13 bit -> Max weight = 8192
  - Could reduce weight precision to support higher resolution images

  1. Segmented min scan: 10 bit weight, 22 bit ID
  -> Changed to long long; 13 bit weight, 25 bit ID
  8. List L: 32 bit vertex ID left, 32 bit vertex ID right
  12. UVW: u.id 24 bit, v.id 24 bit, weight 16 bit
  -> Change to u.id 25 bit, v.id 25 bit, weight 13 bit
************************************************************************************/

////////////////////////////////////////////////
// Variables
////////////////////////////////////////////////

// Standard C stuff
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>

// C++ stuff
#include <iostream>
#include <vector>

// Command line options
#include <getopt.h>
#include "Options.h"

// Timings
#include <chrono>
#include <sys/time.h>

// Kernels
#include "Kernels.cu"

// Thrust stuff
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/transform.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/functional.h>

// Opencv stuff
#include "opencv2/imgproc.hpp"
#include "opencv2/core/cuda.hpp"
#include "opencv2/imgcodecs.hpp"
#include "opencv2/cudaimgproc.hpp"
#include "opencv2/cudafilters.hpp" // cv::cuda::Filter
#include "opencv2/cudaarithm.hpp" // cv::cuda::abs or cv::cuda::addWeighted

using namespace cv;
using namespace cv::cuda;

// Curand stuff
#include <hip/hip_runtime.h>
#include <hiprand.h>


////////////////////////////////////////////////
// Variables
////////////////////////////////////////////////
unsigned int no_of_rows;									// Number of rows in image
unsigned int no_of_cols;									// Number of columns in image

unsigned int no_of_vertices;								//Actual input graph sizes
unsigned int no_of_vertices_orig;							//Original number of vertices graph (constant)

unsigned int no_of_edges;									//Current graph sizes
unsigned int no_of_edges_orig;								//Original number of edges graph (constant)

unsigned int *d_edge;										// Starts as h_edge
unsigned int *d_vertex;										// starts as h_vertex
unsigned int *d_weight;										// starts as h_weight
unsigned int *d_edge_strength;
unsigned int *d_edge_strength_copy;
unsigned char *d_avg_color;
unsigned char *d_avg_color_copy;
float* d_avg_color_r;
float* d_avg_color_g;
float* d_avg_color_b;
float* d_avg_color_r_copy;
float* d_avg_color_g_copy;
float* d_avg_color_b_copy;
unsigned int *d_component_size;	
unsigned int *d_old_component_size;	
unsigned int *d_component_size_copy;	

unsigned long long int *d_segmented_min_scan_input;			//X, Input to the Segmented Min Scan, appended array of weights and edge IDs
unsigned long long int *d_segmented_min_scan_output;		//Output of the Segmented Min Scan, minimum weight outgoing edge as (weight|to_vertex_id elements) can be found at end of each segment
unsigned int *d_vertex_flag_thrust;
unsigned int *d_edge_flag;									//Flag for the segmented min scan
unsigned int *d_edge_flag_thrust;							//NEW! Flag for the segmented min scan in thrust Needs to be 000111222 instead of 100100100
unsigned int *d_vertex_flag;								//F2, Flag for the scan input for supervertex ID generation
unsigned int *d_pick_array;									//PickArray for each edge. index min weight outgoing edge of u in sorted array if not removed. Else -1 if removed (representative doesn't add edges)
unsigned int *d_successor;									//S, Successor Array
unsigned int *d_successor_copy;								//Helper array for pointer doubling
bool *d_succchange;											//Variable to check if can stop pointer doubling

unsigned int *d_new_supervertexIDs;							//mapping from each original vertex ID to its new supervertex ID so we can lookup supervertex IDs directly
unsigned int *d_old_uIDs;									//expanded old u ids, stored per edge, needed to remove self edges (orig ID of source vertex u for each edge(weight|dest_vertex_id_v))
unsigned long long int *d_appended_uvw;						//Appended u,v,w array for duplicate edge removal

unsigned int *d_size;										//Stores amount of edges
unsigned int *d_edge_mapping_copy;
unsigned int *d_edge_list_size;
unsigned int *d_vertex_list_size;

unsigned long long int *d_vertex_split;						//L, Input to the split function

// Hierarchy output
int cur_hierarchy_size; 									// Size current hierarchy

enum timing_mode {NO_TIME, TIME_COMPLETE, TIME_PARTS};
enum timing_mode TIMING_MODE;
std::vector<int> timings;

bool NO_WRITE = false;

////////////////////////////////////////////////
// Debugging helper functions
////////////////////////////////////////////////
void printIntArr(int* d_data, int n_elements) {
	int* h_data = (int *)malloc(sizeof(int)*n_elements);
	hipMemcpy(h_data, d_data, sizeof(int) * n_elements, hipMemcpyDeviceToHost);
	for (int i = 0; i < n_elements; i++) {
		printf("%d ",h_data[i]);
	}
	printf("\n");
	free(h_data);
}

void printXArr(int* d_data, int n_elements) {
	int* h_data = (int *)malloc(sizeof(int)*n_elements);
	hipMemcpy(h_data, d_data, sizeof(int) * n_elements, hipMemcpyDeviceToHost);
	for (int i = 0; i < n_elements; i++) {
		int mask = pow(2.0,MOVEBITS)-1;
		int vertex = h_data[i]&mask;
		int weight = h_data[i]>>MOVEBITS;
		printf("%d|%d ",weight, vertex);
	}
	printf("\n");
	free(h_data);
}

void printUVWArr(unsigned long long int *d_data, int n_elements) {
	unsigned long long int* h_data = (unsigned long long int *)malloc(sizeof(unsigned long long int)*n_elements);
	hipMemcpy(h_data, d_data, sizeof(unsigned long long int) * n_elements, hipMemcpyDeviceToHost);
	for (int i = 0; i < n_elements; i++) {
		unsigned long long int UVW = h_data[i];
		unsigned long long int mask = pow(2.0,64-(NO_OF_BITS_MOVED_FOR_VERTEX_IDS+NO_OF_BITS_MOVED_FOR_VERTEX_IDS))-1;
		unsigned long long int w  = (int) UVW&mask;
		unsigned long long int test = UVW>>(64-(NO_OF_BITS_MOVED_FOR_VERTEX_IDS+NO_OF_BITS_MOVED_FOR_VERTEX_IDS));
		unsigned long long int mask2 = pow(2.0,NO_OF_BITS_MOVED_FOR_VERTEX_IDS)-1;
		unsigned long long int v = test&mask2;
		unsigned long long int u = test>>NO_OF_BITS_MOVED_FOR_VERTEX_IDS;
		printf("%llu|%llu|%llu ",u, v, w);
	}
	printf("\n");
	free(h_data);
}

void printUIntArr(unsigned int* d_data, int n_elements) {
	unsigned int* h_data = (unsigned int *)malloc(sizeof(unsigned int)*n_elements);
	hipMemcpy(h_data, d_data, sizeof(unsigned int) * n_elements, hipMemcpyDeviceToHost);
	for (int i = 0; i < n_elements; i++) {
		printf("%u ",h_data[i]);
	}
	printf("\n");
	free(h_data);
}

void printULongArr(long* d_data, int n_elements) {
	unsigned long* h_data = (unsigned long *)malloc(sizeof(unsigned long)*n_elements);
	hipMemcpy(h_data, d_data, sizeof(unsigned long) * n_elements, hipMemcpyDeviceToHost);
	for (int i = 0; i < n_elements; i++) {
		printf("%lu ",h_data[i]);
	}
	printf("\n");
	free(h_data);
}

void printLongArr(long* d_data, int n_elements) {
	long* h_data = (long *)malloc(sizeof(long)*n_elements);
	hipMemcpy(h_data, d_data, sizeof(long) * n_elements, hipMemcpyDeviceToHost);
	for (int i = 0; i < n_elements; i++) {
		printf("%ld ",h_data[i]);
	}
	printf("\n");
	free(h_data);
}

void printInt(int *d_val) {
	int h_val;
	hipMemcpy( &h_val, d_val, sizeof(int), hipMemcpyDeviceToHost);
	printf("%d", h_val);
}

void printUInt(unsigned int *d_val) {
	unsigned int h_val;
	hipMemcpy( &h_val, d_val, sizeof(unsigned int), hipMemcpyDeviceToHost);
	printf("%u", h_val);
}

////////////////////////////////////////////////
// Helper functions to set the grid sizes
////////////////////////////////////////////////
void SetGridThreadLen(int number, int *num_of_blocks, int *num_of_threads_per_block)
{
	*num_of_blocks = 1;
	*num_of_threads_per_block = number;

	//Make execution Parameters according to the number of nodes
	//Distribute threads across multiple Blocks if necessary
	if(number>MAX_THREADS_PER_BLOCK)
	{
		*num_of_blocks = (int)ceil(number/(double)MAX_THREADS_PER_BLOCK); 
		*num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
	}
}

void SetImageGridThreadLen(int no_of_rows, int no_of_cols, int no_of_vertices, dim3* encode_threads, dim3* encode_blocks)
{    
    if (no_of_vertices < 1024) {
        encode_threads->x = no_of_rows;
        encode_threads->y = no_of_cols;
        encode_blocks->x = 1;
        encode_blocks->y = 1;
    } else {
        encode_threads->x = 32;
        encode_threads->y = 32;
        encode_blocks->x = no_of_rows / 32 + 1;
        encode_blocks->y = no_of_cols / 32 + 1;
    }
}

////////////////////////////////////////////////
// Allocate and Free segmentation Arrays
////////////////////////////////////////////////
void Init()
{

	//Allocate graph device memory
	hipMalloc( (void**) &d_edge, sizeof(unsigned int)*no_of_edges_orig);
	hipMalloc( (void**) &d_vertex, sizeof(unsigned int)*no_of_vertices_orig);
	hipMalloc( (void**) &d_weight, sizeof(unsigned int)*no_of_edges_orig);
	hipMalloc( (void**) &d_edge_strength, sizeof(unsigned int)*no_of_edges_orig);
	hipMalloc( (void**) &d_edge_strength_copy, sizeof(unsigned int)*no_of_edges_orig);
	hipMalloc( (void**) &d_component_size, sizeof(unsigned int)*no_of_vertices_orig);
	hipMalloc( (void**) &d_old_component_size, sizeof(unsigned int)*no_of_vertices_orig);
	hipMalloc( (void**) &d_component_size_copy, sizeof(unsigned int)*no_of_vertices_orig);
	hipMalloc( (void**) &d_avg_color, sizeof(unsigned char)*3*no_of_vertices_orig);
	hipMalloc( (void**) &d_avg_color_r, sizeof(float)*no_of_vertices_orig);
	hipMalloc( (void**) &d_avg_color_g, sizeof(float)*no_of_vertices_orig);
	hipMalloc( (void**) &d_avg_color_b, sizeof(float)*no_of_vertices_orig);
	hipMalloc( (void**) &d_avg_color_r_copy, sizeof(float)*no_of_vertices_orig);
	hipMalloc( (void**) &d_avg_color_g_copy, sizeof(float)*no_of_vertices_orig);
	hipMalloc( (void**) &d_avg_color_b_copy, sizeof(float)*no_of_vertices_orig);
	hipMalloc( (void**) &d_avg_color_copy, sizeof(unsigned char)*3*no_of_vertices_orig);

	//Allocate memory for other arrays
	hipMalloc( (void**) &d_segmented_min_scan_input, sizeof(unsigned long long int)*no_of_edges_orig);
	hipMalloc( (void**) &d_segmented_min_scan_output, sizeof(unsigned long long int)*no_of_edges_orig);
	hipMalloc( (void**) &d_edge_flag, sizeof(unsigned int)*no_of_edges_orig);
	hipMalloc( (void**) &d_edge_flag_thrust, sizeof(unsigned int)*no_of_edges_orig);
	hipMalloc( (void**) &d_pick_array, sizeof(unsigned int)*no_of_edges_orig);
	hipMalloc( (void**) &d_successor,sizeof(unsigned int)*no_of_vertices_orig);
	hipMalloc( (void**) &d_successor_copy,sizeof(unsigned int)*no_of_vertices_orig);
	
	//Clear Output MST array
	hipMalloc( (void**) &d_succchange, sizeof(bool));
	hipMalloc( (void**) &d_vertex_split, sizeof(unsigned long long int)*no_of_vertices_orig);
	hipMalloc( (void**) &d_vertex_flag, sizeof(unsigned int)*no_of_vertices_orig);
	hipMalloc( (void**) &d_vertex_flag_thrust, sizeof(unsigned int)*no_of_vertices_orig);
	hipMalloc( (void**) &d_new_supervertexIDs, sizeof(unsigned int)*no_of_vertices_orig);
	hipMalloc( (void**) &d_old_uIDs, sizeof(unsigned int)*no_of_edges_orig);
	hipMalloc( (void**) &d_appended_uvw, sizeof(unsigned long long int)*no_of_edges_orig);
	hipMalloc( (void**) &d_size, sizeof(unsigned int));
	hipMalloc( (void**) &d_edge_mapping_copy, sizeof(unsigned int)*no_of_edges_orig); 

	hipMalloc( (void**) &d_edge_list_size, sizeof(unsigned int));
	hipMalloc( (void**) &d_vertex_list_size, sizeof(unsigned int));
	
}

void FreeMem()
{
	hipFree(d_edge);
	hipFree(d_vertex);
	hipFree(d_weight);
	hipFree(d_edge_strength);
	hipFree(d_edge_strength_copy);
	hipFree(d_component_size);
	hipFree(d_old_component_size);
	hipFree(d_component_size_copy);
	hipFree(d_avg_color);
	hipFree(d_avg_color_r);
	hipFree(d_avg_color_g);
	hipFree(d_avg_color_b);
	hipFree(d_avg_color_r_copy);
	hipFree(d_avg_color_g_copy);
	hipFree(d_avg_color_b_copy);
	hipFree(d_avg_color_copy);
	hipFree(d_segmented_min_scan_input);
	hipFree(d_segmented_min_scan_output);
	hipFree(d_edge_flag);
	hipFree(d_edge_flag_thrust);
	hipFree(d_pick_array);
	hipFree(d_successor);
	hipFree(d_successor_copy);
	hipFree(d_succchange);
	hipFree(d_vertex_split);
	hipFree(d_vertex_flag);
	hipFree(d_vertex_flag_thrust);
	hipFree(d_new_supervertexIDs);
	hipFree(d_old_uIDs);
	hipFree(d_size);
	hipFree(d_edge_mapping_copy);
	hipFree(d_edge_list_size);
	hipFree(d_vertex_list_size);
	hipFree(d_appended_uvw);
}

////////////////////////////////////////////////
// Create graph in compressed adjacency list
////////////////////////////////////////////////
void createGraph(Mat image) {
	std::chrono::high_resolution_clock::time_point start, end;

	// Gaussian init
   	GpuMat dev_image, d_blurred; 	 // Released automatically in destructor
   	cv::Ptr<cv::cuda::Filter> filter;

   	// Sobel init
   	GpuMat d_blurred_gray, d_resultx, d_abs_resultx, d_resulty, d_abs_resulty, d_sobel;
   	cv::Ptr<cv::cuda::Filter> filtersobelx;
   	cv::Ptr<cv::cuda::Filter> filtersobely;
   	int ddepth = CV_16S; // use 16 bits unsigned to avoid overflow

	if (TIMING_MODE == TIME_PARTS) { // Start gaussian filter timer
		start = std::chrono::high_resolution_clock::now();
	}

	// 1. Apply gaussian filter
    dev_image.upload(image);
    filter = cv::cuda::createGaussianFilter(CV_8UC3, CV_8UC3, cv::Size(5, 5), 1.0);
    filter->apply(dev_image, d_blurred);

    // 2. Convert the blurred image to grayscale
    cv::cuda::cvtColor(d_blurred, d_blurred_gray, COLOR_RGB2GRAY);

    // 3.1 Apply sobel in x direction
   	filtersobelx = cv::cuda::createSobelFilter(d_blurred_gray.type(),ddepth,1,0);
	filtersobelx->apply(d_blurred_gray, d_resultx);
	cv::cuda::abs(d_resultx, d_resultx);
	d_resultx.convertTo(d_abs_resultx, CV_8UC1);

	// 3.2 Apply sobel in y direction
	filtersobely = cv::cuda::createSobelFilter(d_blurred_gray.type(),ddepth,0,1);
	filtersobely->apply(d_blurred_gray, d_resulty);
	cv::cuda::abs(d_resulty, d_resulty);
	d_resulty.convertTo(d_abs_resulty, CV_8UC1);

	// 4. Combine sobel results
	cv::cuda::addWeighted(d_abs_resultx, 0.5, d_abs_resulty, 0.5, 0, d_sobel);

	if (TIMING_MODE == TIME_PARTS) { // End gaussian filter timer
		hipDeviceSynchronize();
		end = std::chrono::high_resolution_clock::now();
		int time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
		timings.push_back(time);
	}

	dev_image.release();
	d_blurred_gray.release();
	d_resultx.release();
	d_abs_resultx.release();
	d_resulty.release();
	d_abs_resulty.release();

	if (TIMING_MODE == TIME_PARTS) { // Start graph creation timer
		start = std::chrono::high_resolution_clock::now();
	}

	// Allocate GPU segmentation memory
	Init();
	size_t pitch = d_blurred.step;
	size_t edge_pitch = d_sobel.step;

	// Create graphs. Kernels executed in different streams for concurrency
	dim3 encode_threads;
	dim3 encode_blocks;
	SetImageGridThreadLen(no_of_rows, no_of_cols, no_of_vertices_orig, &encode_threads, &encode_blocks);

    int num_of_blocks, num_of_threads_per_block;

	SetGridThreadLen(no_of_cols, &num_of_blocks, &num_of_threads_per_block);
	dim3 grid_row(num_of_blocks, 1, 1);
	dim3 threads_row(num_of_threads_per_block, 1, 1);

	SetGridThreadLen(no_of_rows, &num_of_blocks, &num_of_threads_per_block);
	dim3 grid_col(num_of_blocks, 1, 1);
	dim3 threads_col(num_of_threads_per_block, 1, 1);

    dim3 grid_corner(1, 1, 1);
	dim3 threads_corner(4, 1, 1);

	SetGridThreadLen(no_of_rows * no_of_cols, &num_of_blocks, &num_of_threads_per_block);
	dim3 grid_cmp(num_of_blocks, 1, 1);
	dim3 threads_cmp(num_of_threads_per_block, 1, 1);

    // Create inner graph
    createInnerGraphKernel<<< encode_blocks, encode_threads, 0>>>((unsigned char*) d_sobel.cudaPtr(), d_vertex, d_edge, d_edge_strength, no_of_rows, no_of_cols, edge_pitch);

    // Create outer graph
   	createFirstRowGraphKernel<<< grid_row, threads_row, 1>>>((unsigned char*) d_sobel.cudaPtr(), d_vertex, d_edge, d_edge_strength, no_of_rows, no_of_cols, edge_pitch);
   	createLastRowGraphKernel<<< grid_row, threads_row, 2>>>((unsigned char*) d_sobel.cudaPtr(), d_vertex, d_edge, d_edge_strength, no_of_rows, no_of_cols, edge_pitch);

   	createFirstColumnGraphKernel<<< grid_col, threads_col, 3>>>((unsigned char*) d_sobel.cudaPtr(), d_vertex, d_edge, d_edge_strength, no_of_rows, no_of_cols, edge_pitch);
   	createLastColumnGraphKernel<<< grid_col, threads_col, 4>>>((unsigned char*) d_sobel.cudaPtr(), d_vertex, d_edge, d_edge_strength, no_of_rows, no_of_cols, edge_pitch);

    // Create corners
	createCornerGraphKernel<<< grid_corner, threads_corner, 5>>>((unsigned char*) d_sobel.cudaPtr(), d_vertex, d_edge, d_edge_strength, no_of_rows, no_of_cols, edge_pitch);

	createAvgColorArray<<< encode_blocks, encode_threads, 6>>>((unsigned char*) d_blurred.cudaPtr(), d_avg_color, no_of_rows, no_of_cols, pitch);

	InitComponentSizes<<<grid_cmp, threads_cmp ,7>>>(d_component_size, no_of_rows * no_of_cols);
	
	hipDeviceSynchronize(); // Needed to synchronise streams!

	if (TIMING_MODE == TIME_PARTS) {
		end = std::chrono::high_resolution_clock::now();
		int time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
		timings.push_back(time);
	}

	fprintf(stderr, "Image read successfully into graph with %d vertices and %d edges\n", no_of_vertices, no_of_edges);
}


////////////////////////////////////////////////
// Perform Our Recursive MST Algorithm
////////////////////////////////////////////////
void HPGMST()
{
	//Make both CUDA grids needed for execution, no_of_vertices and no_of_edges length sizes
	int num_of_blocks, num_of_threads_per_block;

	//Grid and block sizes so each edge has one thread (fit as much threads as possible in one block)
	SetGridThreadLen(no_of_edges, &num_of_blocks, &num_of_threads_per_block);
	dim3 grid_edgelen(num_of_blocks, 1, 1);
	dim3 threads_edgelen(num_of_threads_per_block, 1, 1);

	// Grid and block sizes so each vertex has one thread (fit as much threads as possible in one block)
	SetGridThreadLen(no_of_vertices, &num_of_blocks, &num_of_threads_per_block);
	dim3 grid_vertexlen(num_of_blocks, 1, 1);
	dim3 threads_vertexlen(num_of_threads_per_block, 1, 1);

	/*
	 * A. Find minimum weighted edge
	 */

	// d_edge_flag = F
	//Create the Flag needed for segmented min scan operation, similar operation will also be used at other places
	ClearArray<<< grid_edgelen, threads_edgelen, 0>>>( d_edge_flag, no_of_edges );


	// 2. Divide the edge-list, E, into segments with 1 indicating the start of each segment and 0 otherwise, store this in flag array F.
	// Mark the segments for the segmented min scan
	MakeFlag_3<<< grid_vertexlen, threads_vertexlen, 0>>>( d_edge_flag, d_vertex, no_of_vertices);

	// 10.2 Create vector indicating source vertex u for each edge // DONE: change to thrust
	thrust::inclusive_scan(thrust::device, d_edge_flag, d_edge_flag + no_of_edges, d_old_uIDs);

	// Calculate weights
	CalcWeights<<<grid_edgelen, threads_edgelen, 0>>>(d_avg_color, d_old_uIDs, d_edge, d_edge_strength, d_weight, no_of_edges);

	void printUIntArr(d_weight, 1000);

	// 1. Append weight w and outgoing vertex v per edge into a single array, X.
    // 12 bit for weight, 26 bits for ID.
	//Append in Parallel on the Device itself, call the append kernel
	AppendKernel_1<<< grid_edgelen, threads_edgelen, 0>>>(d_segmented_min_scan_input, d_weight, d_edge, no_of_edges);

	// 3. Perform segmented min scan on X with F indicating segments to find minimum outgoing edge-index per vertex. Min can be found at end of each segment after scan // DONE: change to thrust
	// Prepare key vector for thrust
	thrust::inclusive_scan(thrust::device, d_edge_flag, d_edge_flag + no_of_edges, d_edge_flag_thrust);

	// Min inclusive segmented scan on ints from start to end.
	thrust::equal_to<unsigned int> binaryPred;
	thrust::minimum<unsigned long long int> binaryOp;
	thrust::inclusive_scan_by_key(thrust::device, d_edge_flag_thrust, d_edge_flag_thrust + no_of_edges, d_segmented_min_scan_input, d_segmented_min_scan_output, binaryPred, binaryOp);


	/*
	 * B. Finding and removing cycles
	 */

	// 4. Find the successor of each vertex and add to successor array, S.
	MakeSucessorArray<<< grid_vertexlen, threads_vertexlen, 0>>>(d_successor, d_vertex, d_segmented_min_scan_output, no_of_vertices, no_of_edges);


	// 5. Remove cycle making edges using S, and identify representatives vertices.
	RemoveCycles<<< grid_vertexlen, threads_vertexlen, 0>>>(d_successor,no_of_vertices);


	/*
	 * Can possibly be moved in future once remove pick array stuff
	 */
	//Scan the flag to get u at every edge, use the u to index d_vertex to get the last entry in each segment
	//U at every edge will also be useful later in the algorithm.

	// Set F[0] = 0. F is the same as previous F but first element is 0 instead of 1
	ClearArray<<< grid_edgelen, threads_edgelen, 0>>>( d_edge_flag, no_of_edges );
	MakeFlagForUIds<<< grid_vertexlen, threads_vertexlen, 0>>>(d_edge_flag, d_vertex,no_of_vertices); 


	/*
	 * C. Merging vertices and assigning IDs to supervertices
	 */


	// 7. Propagate Representative Vertex IDs to all vertices iteratively using pointer Doubling until no change occures in Successor Array
	bool succchange;
	do
	{
		succchange=false; //if no thread changes this value, the loop stops
		hipMemcpy( d_succchange, &succchange, sizeof(bool), hipMemcpyHostToDevice);
		//Reusing Vertex Flag
		SuccToCopy<<< grid_vertexlen, threads_vertexlen, 0>>>(d_successor, d_successor_copy, no_of_vertices); // for conflicts
		PropagateRepresentativeID<<< grid_vertexlen, threads_vertexlen, 0>>>(d_successor, d_successor_copy, d_succchange,no_of_vertices);
		CopyToSucc<<< grid_vertexlen, threads_vertexlen, 0>>>(d_successor, d_successor_copy, no_of_vertices); // for conflicts

		hipMemcpy( &succchange, d_succchange, sizeof(bool), hipMemcpyDeviceToHost);
	}
	while(succchange);


	// 8. Append successor array’s entries with its index to form a list, L. Representative left, vertex id right, 64 bit.
	//    Append Vertex Ids with SuperVertexIDs
	AppendVertexIDsForSplit<<< grid_vertexlen, threads_vertexlen, 0>>>(d_vertex_split, d_successor,no_of_vertices);


	//9. Split L, create flag over split output and scan the flag to find new ids per vertex, store new ids in C.
    // 9.1 Split L using representative as key. In parallel using a split of O(V) with log(V) bit key size.
    //     split based on supervertex IDs using 64 bit version of split
	thrust::sort(thrust::device, d_vertex_split, d_vertex_split + no_of_vertices);

	// Sort component sizes
	SuccToCopy<<< grid_vertexlen, threads_vertexlen, 0>>>(d_component_size, d_old_component_size, no_of_vertices);
	SortComponentSizesFromSplit<<< grid_vertexlen, threads_vertexlen, 0>>>(d_component_size, d_component_size_copy, d_vertex_split, no_of_vertices);
	CopyToSucc<<< grid_vertexlen, threads_vertexlen, 0>>>(d_component_size, d_component_size_copy, no_of_vertices);

	// Sort avg colors
	SortAvgColorsFromSplit<<< grid_vertexlen, threads_vertexlen, 0>>>(d_avg_color, d_avg_color_copy, d_vertex_split, no_of_vertices);
	CopyToAvgColor<<< grid_vertexlen, threads_vertexlen, 0>>>(d_avg_color, d_avg_color_copy, no_of_vertices); // for conflicts

	// Sort edge strength
	SortComponentSizesFromSplit<<< grid_vertexlen, threads_vertexlen, 0>>>(d_edge_strength, d_edge_strength_copy, d_vertex_split, no_of_vertices);
	CopyToSucc<<< grid_vertexlen, threads_vertexlen, 0>>>(d_edge_strength, d_edge_strength_copy, no_of_vertices);


	// 9.2 Create flag for assigning new vertex IDs based on difference in supervertex IDs
	//     first element not flagged so that can use simple sum for scan
	ClearArray<<< grid_vertexlen, threads_vertexlen, 0>>>( d_vertex_flag, no_of_vertices);
	MakeFlagForScan<<< grid_vertexlen, threads_vertexlen, 0>>>(d_vertex_flag, d_vertex_split, no_of_vertices);

	// Prepare key vector for thrust
	change_elem<<<1,1>>>(d_vertex_flag, 0, 1); // Set first element 1 for segmented scan
	thrust::inclusive_scan(thrust::device, d_vertex_flag, d_vertex_flag + no_of_vertices, d_vertex_flag_thrust);
	change_elem<<<1,1>>>(d_vertex_flag, 1, 0); // Reset first element to 0 for rest algorithm


	// Perform segmented add scan on component_size with F2 indicating segments to find new component size
	thrust::equal_to<unsigned int> binaryPred2;
	thrust::plus<unsigned int> binaryOp2;
	thrust::inclusive_scan_by_key(thrust::device, d_vertex_flag_thrust, d_vertex_flag_thrust + no_of_vertices, d_component_size, d_component_size_copy, binaryPred2, binaryOp2);
	
	// Extract new component sizes
	ExtractComponentSizes<<< grid_vertexlen, threads_vertexlen, 0>>>(d_component_size_copy, d_component_size, d_vertex_flag_thrust, no_of_vertices);


	// Min inclusive segmented scan on ints from start to end.
	thrust::equal_to<unsigned int> binaryPred4;
	thrust::minimum<unsigned int> binaryOp4;
	thrust::inclusive_scan_by_key(thrust::device, d_vertex_flag_thrust, d_vertex_flag_thrust + no_of_vertices, d_edge_strength, d_edge_strength_copy, binaryPred4, binaryOp4);

	// Extract new edge strengths
	ExtractComponentSizes<<< grid_vertexlen, threads_vertexlen, 0>>>(d_edge_strength_copy, d_edge_strength, d_vertex_flag_thrust, no_of_vertices);

	// 9.3 Scan flag to assign new IDs to supervertices, Using a scan on O(V) elements // DONE: change to thrust
	thrust::inclusive_scan(thrust::device, d_vertex_flag, d_vertex_flag + no_of_vertices, d_new_supervertexIDs);


	// Reweigh colors joining components so their sum when adding them up is the average
	ReweighAndOrganizeColors<<< grid_vertexlen, threads_vertexlen, 0>>>(d_avg_color, d_avg_color_r, d_avg_color_g, d_avg_color_b, d_component_size, d_old_component_size, d_new_supervertexIDs, d_vertex_flag_thrust, no_of_vertices);
	thrust::plus<float> binaryOp3;
	thrust::inclusive_scan_by_key(thrust::device, d_vertex_flag_thrust, d_vertex_flag_thrust + no_of_vertices, d_avg_color_r, d_avg_color_r_copy, binaryPred2, binaryOp3);
	thrust::inclusive_scan_by_key(thrust::device, d_vertex_flag_thrust, d_vertex_flag_thrust + no_of_vertices, d_avg_color_g, d_avg_color_g_copy, binaryPred2, binaryOp3);
	thrust::inclusive_scan_by_key(thrust::device, d_vertex_flag_thrust, d_vertex_flag_thrust + no_of_vertices, d_avg_color_b, d_avg_color_b_copy, binaryPred2, binaryOp3);

	// Extract new colors
	ExtractNewColors<<< grid_vertexlen, threads_vertexlen, 0>>>(d_avg_color_r_copy, d_avg_color_g_copy, d_avg_color_b_copy, d_avg_color, d_vertex_flag_thrust, no_of_vertices);

	/*
	 * D. Removing self edges
	 */

	// 10.1 Create mapping from each original vertex ID to its new supervertex ID so we can lookup supervertex IDs directly
	MakeSuperVertexIDPerVertex<<< grid_vertexlen, threads_vertexlen, 0>>>(d_new_supervertexIDs, d_vertex_split, d_vertex_flag, no_of_vertices);
	CopySuperVertexIDPerVertex<<< grid_vertexlen, threads_vertexlen, 0>>>(d_new_supervertexIDs, d_vertex_flag, no_of_vertices); // for concurrent access problems
	
	//Remove Self Edges from the edge-list
	// 11. Remove edge from edge-list if u, v have same supervertex id (remove self edges)
	CopyEdgeArray<<< grid_edgelen, threads_edgelen, 0>>>(d_edge,d_edge_mapping_copy, no_of_edges); // for conflicts
	RemoveSelfEdges<<< grid_edgelen, threads_edgelen, 0>>>(d_edge, d_old_uIDs, d_new_supervertexIDs, d_edge_mapping_copy, no_of_edges);
	CopyEdgeArrayBack<<< grid_edgelen, threads_edgelen, 0>>>(d_edge,d_edge_mapping_copy, no_of_edges); // for conflicts

	/*
	 * D. Removing duplicate edges. This is not mandatory, however, reduces the edge-list size significantly. You may choose to use it once in the initial 
	 *    iterations of the algorithm, later edge-list size is small anyways so not much is gained by doing this in later iterations
	 */


	// 12. Remove the largest duplicate edges using split over new u,v and w.
	// 12.1 Append supervertex ids of u and v along with weight w into single 64 bit array (u 24 bit, v 24 bit, w 16 bit)
	AppendForDuplicateEdgeRemoval<<< grid_edgelen, threads_edgelen, 0>>>(d_appended_uvw, d_edge, d_old_uIDs, d_weight,d_new_supervertexIDs, no_of_edges);

	//12.2 Split the array using {u,v) as the key. Pick First distinct (u,v) entry as the edge, nullify others
	//     You may also replace the split with sort, but we could not find a 64-bit sort.
	thrust::sort(thrust::device, d_appended_uvw, d_appended_uvw + no_of_edges);
	
	//Pick the first distinct (u,v) combination, mark these edges and compact
	// 12.3 Create flag indicating smallest edges, 0 for larger duplicates
	ClearArray<<< grid_edgelen, threads_edgelen, 0>>>( d_edge_flag, no_of_edges ); // d_edge_flag = F3
	unsigned int dsize=no_of_edges; //just make sure
	hipMemcpy( d_size, &dsize, sizeof(unsigned int), hipMemcpyHostToDevice);
	MarkEdgesUV<<< grid_edgelen, threads_edgelen, 0>>>(d_edge_flag, d_appended_uvw, d_size, no_of_edges);


	// 13. Compact and create new edge and weight list
	// 13.1 Scan the flag array to know where to write the value in new edge and weight lists // DONE: change to thrust
	thrust::inclusive_scan(thrust::device, d_edge_flag, d_edge_flag + no_of_edges, d_old_uIDs);

	// Make sure new locations start from 0 instead of 1.
	thrust::transform(thrust::device,
				  d_old_uIDs,
                  d_old_uIDs + no_of_edges,
                  thrust::make_constant_iterator(1),
                  d_old_uIDs,
                  thrust::minus<unsigned int>());


	// Do some cleanup / clearing
	ClearEdgeStuff<<< grid_edgelen, threads_edgelen, 0>>>((unsigned int*)d_edge, (unsigned int*)d_weight, d_edge_mapping_copy, (unsigned int*)d_pick_array, no_of_edges);
	unsigned int negative=0;
	hipMemcpy( d_edge_list_size, &negative, sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy( d_vertex_list_size, &negative, sizeof(unsigned int), hipMemcpyHostToDevice);
	
	//Compact the edge and weight lists
	unsigned int validsize=0;
	hipMemcpy( &validsize, d_size, sizeof(unsigned int), hipMemcpyDeviceToHost);

	//Make a new grid for valid entries in the d_edge_flag array
	SetGridThreadLen(validsize, &num_of_blocks, &num_of_threads_per_block);
	dim3 grid_validsizelen(num_of_blocks, 1, 1);
	dim3 threads_validsizelen(num_of_threads_per_block, 1, 1);

	// 13.2 Compact and create new edge and weight list
	//      Reusing d_pick_array for storing the u ids
	CompactEdgeList<<< grid_validsizelen, threads_validsizelen, 0>>>(d_edge, d_weight, d_old_uIDs, d_edge_flag, d_appended_uvw, d_pick_array, d_size, d_edge_list_size, d_vertex_list_size);

	// 14. Build the vertex list from the newly formed edge list
	ClearArray<<< grid_edgelen, threads_edgelen, 0>>>( d_edge_flag, no_of_edges);
	ClearArray<<< grid_vertexlen, threads_vertexlen, 0>>>((unsigned int*)d_vertex, no_of_vertices);

	//14.1 Create flag based on difference in u on the new edge list (based on diffference of u ids)
	MakeFlagForVertexList<<< grid_edgelen, threads_edgelen, 0>>>(d_pick_array, d_edge_flag, no_of_edges); // d_edge_flag = F4

	// 14.2 Build the vertex list from the newly formed edge list
	MakeVertexList<<< grid_edgelen, threads_edgelen, 0>>>(d_vertex, d_pick_array, d_edge_flag, no_of_edges);
	
	cur_hierarchy_size = no_of_vertices;
	hipMemcpy( &no_of_edges, d_edge_list_size, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy( &no_of_vertices, d_vertex_list_size, sizeof(unsigned int), hipMemcpyDeviceToHost);

}


void writeComponents(std::vector<unsigned int*>& d_hierarchy_levels, std::vector<int>& hierarchy_level_sizes, std::string outFile) {
	std::chrono::high_resolution_clock::time_point start, end;
	if (TIMING_MODE == TIME_PARTS || TIMING_MODE == TIME_COMPLETE) { // Start write timer
		start = std::chrono::high_resolution_clock::now();
	}

	// Extract filepath without extension
	size_t lastindex = outFile.find_last_of("."); 
	std::string rawOutName = outFile.substr(0, lastindex);

	// Generate random colors for segments
	char *component_colours = (char *) malloc(no_of_vertices_orig * CHANNEL_SIZE * sizeof(char));


	// Generate uniform [0, 1] float
	hiprandGenerator_t gen;
	char* d_component_colours;
	float *d_component_colours_float;
	hipMalloc( (void**) &d_component_colours_float, no_of_vertices_orig * CHANNEL_SIZE * sizeof(float));
	hipMalloc( (void**) &d_component_colours, no_of_vertices_orig * CHANNEL_SIZE * sizeof(char));

	// Generate random floats
	hiprandCreateGenerator(&gen , HIPRAND_RNG_PSEUDO_MTGP32); // Create a Mersenne Twister pseudorandom number generator
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL); // Set seed
	hiprandGenerateUniform(gen, d_component_colours_float, no_of_vertices_orig * CHANNEL_SIZE); // Generate n floats on device

	// Convert floats to RGB char
	int num_of_blocks, num_of_threads_per_block;

	SetGridThreadLen(no_of_vertices_orig * CHANNEL_SIZE, &num_of_blocks, &num_of_threads_per_block);
	dim3 grid_rgb(num_of_blocks, 1, 1);
	dim3 threads_rgb(num_of_threads_per_block, 1, 1);

	RandFloatToRandRGB<<< grid_rgb, threads_rgb, 0>>>(d_component_colours, d_component_colours_float, no_of_vertices_orig * CHANNEL_SIZE);
	hipFree(d_component_colours_float);


	// Create hierarchy
	unsigned int* d_prev_level_component;
	hipMalloc((void**) &d_prev_level_component, sizeof(unsigned int)*no_of_vertices_orig);

	dim3 threads_pixels;
    dim3 grid_pixels;
	SetImageGridThreadLen(no_of_rows, no_of_cols, no_of_vertices_orig, &threads_pixels, &grid_pixels);

    InitPrevLevelComponents<<<grid_pixels, threads_pixels, 0>>>(d_prev_level_component, no_of_rows, no_of_cols);

    char* d_output_image;
	hipMalloc( (void**) &d_output_image, no_of_rows*no_of_cols*CHANNEL_SIZE*sizeof(char));
    char *output = (char*) malloc(no_of_rows*no_of_cols*CHANNEL_SIZE*sizeof(char));

    for (int l = 0; l < d_hierarchy_levels.size(); l++) {
		int level_size = hierarchy_level_sizes[l];
		unsigned int* d_level = d_hierarchy_levels[l];

		CreateLevelOutput<<< grid_pixels, threads_pixels, 0>>>(d_output_image, d_component_colours, d_level, d_prev_level_component, no_of_rows, no_of_cols);
	    hipMemcpy(output, d_output_image, no_of_rows*no_of_cols*CHANNEL_SIZE*sizeof(char), hipMemcpyDeviceToHost);

	    if (!NO_WRITE) {
	    	cv::Mat output_img = cv::Mat(no_of_rows, no_of_cols, CV_8UC3, output);
			std::string outfilename = rawOutName + std::string("_")  + std::to_string(l) + std::string(".png");
			std::string outmessage = std::string("Writing ") + outfilename.c_str() + std::string("\n");

			fprintf(stderr, "%s", outmessage.c_str());
			imwrite(outfilename, output_img);
	    }
	}

	if (TIMING_MODE == TIME_PARTS || TIMING_MODE == TIME_COMPLETE) { // End write timer
		hipDeviceSynchronize();
		end = std::chrono::high_resolution_clock::now();
		int time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
		if (TIMING_MODE == TIME_PARTS) {
			timings.push_back(time);
		} else {
			timings[0] += time;
		}
	}

	// Free memory
	hipFree(d_component_colours);
	hipFree(d_prev_level_component);
	hipFree(d_output_image);
	free(output);
}

void setGraphParams(unsigned int rows, unsigned int cols) {
	no_of_rows = rows;
    no_of_cols = cols;
	no_of_vertices = no_of_rows * no_of_cols;
	no_of_vertices_orig = no_of_vertices;
	no_of_edges = 8 + 6 * (no_of_cols - 2) + 6 * (no_of_rows - 2) + 4 * (no_of_cols - 2) * (no_of_rows - 2);
	no_of_edges_orig = no_of_edges;
}

void clearHierarchy(std::vector<unsigned int*>& d_hierarchy_levels, std::vector<int>& hierarchy_level_sizes) {
	for (int l = 0; l < d_hierarchy_levels.size(); l++) {
			hipFree(d_hierarchy_levels[l]);
		}
        d_hierarchy_levels.clear();
        hierarchy_level_sizes.clear();
}

void segment(Mat image, std::string outFile, bool output) {
	std::chrono::high_resolution_clock::time_point start, end;

	if (TIMING_MODE == TIME_COMPLETE) { // Start whole execution timer
		start = std::chrono::high_resolution_clock::now();
	}


	// Reset num vertices in edges in case of multiple iterations
	no_of_edges = no_of_edges_orig;
	no_of_vertices = no_of_vertices_orig;

	std::vector<unsigned int*> d_hierarchy_levels;	// Vector containing pointers to all hierarchy levels (don't dereference on CPU, device pointers)
	std::vector<int> hierarchy_level_sizes;			// Size of each hierarchy level

	// Graph creation
	createGraph(image);


	if (TIMING_MODE == TIME_PARTS) { // Start segmentation timer
		start = std::chrono::high_resolution_clock::now();
	}
	
	// Segmentation
	do
	{
	    HPGMST();

	    d_hierarchy_levels.push_back(d_new_supervertexIDs);
	    hierarchy_level_sizes.push_back(cur_hierarchy_size);
	    hipMalloc( (void**) &d_new_supervertexIDs, sizeof(unsigned int)*cur_hierarchy_size);

	    fprintf(stderr, "Vertices: %d\n", no_of_vertices);
	}
	while(no_of_vertices>1);

	if (TIMING_MODE == TIME_PARTS) { // End segmentation timer
		hipDeviceSynchronize();
		end = std::chrono::high_resolution_clock::now();
		int time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
		timings.push_back(time);
	}

	if (TIMING_MODE == TIME_COMPLETE) { // End whole execution timer
		hipDeviceSynchronize();
		end = std::chrono::high_resolution_clock::now();
		int time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
		timings.push_back(time);
	}

	// Free GPU segmentation memory
	FreeMem();

	// Write segmentation hierarchy
	writeComponents(d_hierarchy_levels, hierarchy_level_sizes, outFile);

	clearHierarchy(d_hierarchy_levels, hierarchy_level_sizes);
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
void printUsage() {
    puts("Usage: ./felz -i [input image path] -o [output image path]");
    puts("Options:");
    puts("\t-i: Path to input file (default: data/beach.png)");
    puts("\t-o: Path to output file (default: segmented.png)");
    puts("Benchmarking options");
    puts("\t-w: Number of iterations to perform during warmup");
    puts("\t-b: Number of iterations to perform during benchmarking");
    puts("\t-p: If want to do partial timings");
    puts("\t-n: Don't write images to disk (for benchmarking purposes)");
    exit(1);
}

void printCSVHeader() {
	if (TIMING_MODE == TIME_COMPLETE) {
		 printf("total\n"); // Excluding output: gaussian + graph creation + segmentation
	} else {
		printf("gaussian, graph, segmentation, output\n");
	}
}

void printCSVLine() {
	if (timings.size() > 0) {
		printf("%d", timings[0]);
		for (int i = 1; i < timings.size(); i++) {
			printf(", %d", timings[i]);
		}
		printf("\n");
		timings.clear();
	}
	
}

const Options handleParams(int argc, char **argv) {
    Options options = Options();
    TIMING_MODE = TIME_COMPLETE;
    for(;;)
    {
        switch(getopt(argc, argv, "pnhi:o:w:b:"))
        {
            case 'i': {
                options.inFile = std::string(optarg);
                continue;
            }
            case 'o': {
                options.outFile = std::string(optarg);
                continue;
            }
            case 'w': {
                options.warmupIterations = atoi(optarg);
                continue;
            }
            case 'b': {
                options.benchmarkIterations = atoi(optarg);
                continue;
            }
            case 'p': {
                TIMING_MODE = TIME_PARTS;
                continue;
            }
            case 'n': {
            	NO_WRITE = true;
            	continue;
            }
            case '?':
            case 'h':
            default : {
                printUsage();
                break;
            }

            case -1:  {
                break;
            }
        }
        break;
    }
    if (options.inFile == "empty" || options.outFile == "empty") {
    	puts("Provide an input and output image!");
		printUsage();
    }

    return options;
}

int main(int argc, char **argv)
{
    const Options options = handleParams(argc, argv);

    // Read image
    Mat image = imread(options.inFile, IMREAD_COLOR);
    fprintf(stderr, "Size of image obtained is: Rows: %d, Columns: %d, Pixels: %d\n", image.rows, image.cols, image.rows * image.cols);
   	setGraphParams(image.rows, image.cols);

   	printCSVHeader();

	// Warm up
    for (int i = 0; i < options.warmupIterations; i++) {
    	segment(image, options.outFile, false);
    }

    // Benchmark
    timings.clear();
    for (int i = 0; i < options.benchmarkIterations; i++) {
        segment(image, options.outFile, i == options.benchmarkIterations-1);
        printCSVLine();
    }

    return 0;
}

