#include "hip/hip_runtime.h"
/***********************************************************************************
  Implementing Minimum Spanning Tree on CUDA using primitive operations for the 
  algorithm given in "Fast Minimum Spanning Tree Computation", by Pawan Harish, 
  P.J. Narayanan, Vibhav Vineet, and Suryakant Patidar.

  Chapter 7 of Nvidia GPU Computing Gems, Jade Edition, 2011.
  
  Copyright (c) 2011 International Institute of Information Technology - Hyderabad. 
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for 
  educational purpose is hereby granted without fee, provided that the above copyright 
  notice and this permission notice appear in all copies of this software and that you do 
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND, EXPRESSED, IMPLIED OR 
  OTHERWISE.

  Created by: Pawan Harish.
  Split Implementation by: Suryakant Patidar and Parikshit Sakurikar.
 ************************************************************************************/

/***********************************************************************************
  General bit size info
  ---------------------
  Vertex ID 26 bit -> 67.108.864
  - 8K image: 7680 × 4320 = 33.177.600 pixels -> supports 2 8K images

  Weight 12 bit -> Max weight = 4096
  - Max L2 distance RGB: 442 -> can use 3 more bits for extra precision (*8) (SCALE)
  - Could reduce weight precision to support higher resolution images

  1. Segmented min scan: 10 bit weight, 22 bit ID
  -> Changed to long long; 12 bit weight, 26 bit ID
  8. List L: 32 bit vertex ID left, 32 bit vertex ID right
  12. UVW: u.id 24 bit, v.id 24 bit, weight 16 bit
  -> Change to u.id 26 bit, v.id 26 bit, weight 12 bit
************************************************************************************/

////////////////////////////////////////////////
// Variables
////////////////////////////////////////////////

// Standard C stuff
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// C++ stuff
#include <iostream>
#include <vector>

// Command line options
#include <getopt.h>
#include "Options.h"

// Timings
#include <chrono>
#include <sys/time.h>

// Kernels
#include "Kernels.cu"

// Thrust stuff
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/transform.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/functional.h>

// Opencv stuff
#include "opencv2/imgproc.hpp"
#include "opencv2/imgcodecs.hpp"
#include <opencv2/cudafilters.hpp>
using namespace cv;
using namespace cv::cuda;

// Curand stuff
#include <hip/hip_runtime.h>
#include <hiprand.h>


////////////////////////////////////////////////
// Variables
////////////////////////////////////////////////
unsigned int no_of_rows;									// Number of rows in image
unsigned int no_of_cols;									// Number of columns in image

unsigned int no_of_vertices;								//Actual input graph sizes
unsigned int no_of_vertices_orig;							//Original number of vertices graph (constant)

unsigned int no_of_edges;									//Current graph sizes
unsigned int no_of_edges_orig;								//Original number of edges graph (constant)

unsigned int *d_edge;										// Starts as h_edge
unsigned int *d_vertex;										// starts as h_vertex
unsigned int *d_weight;										// starts as h_weight

unsigned long long int *d_segmented_min_scan_input;			//X, Input to the Segmented Min Scan, appended array of weights and edge IDs
unsigned long long int *d_segmented_min_scan_output;		//Output of the Segmented Min Scan, minimum weight outgoing edge as (weight|to_vertex_id elements) can be found at end of each segment
unsigned int *d_edge_flag;									//Flag for the segmented min scan
unsigned int *d_edge_flag_thrust;							//NEW! Flag for the segmented min scan in thrust Needs to be 000111222 instead of 100100100
unsigned int *d_vertex_flag;								//F2, Flag for the scan input for supervertex ID generation
unsigned int *d_pick_array;									//PickArray for each edge. index min weight outgoing edge of u in sorted array if not removed. Else -1 if removed (representative doesn't add edges)
unsigned int *d_successor;									//S, Successor Array
unsigned int *d_successor_copy;								//Helper array for pointer doubling
bool *d_succchange;											//Variable to check if can stop pointer doubling

unsigned int *d_new_supervertexIDs;							//mapping from each original vertex ID to its new supervertex ID so we can lookup supervertex IDs directly
unsigned int *d_old_uIDs;									//expanded old u ids, stored per edge, needed to remove self edges (orig ID of source vertex u for each edge(weight|dest_vertex_id_v))
unsigned long long int *d_appended_uvw;						//Appended u,v,w array for duplicate edge removal

unsigned int *d_size;										//Stores amount of edges
unsigned int *d_edge_mapping_copy;
unsigned int *d_edge_list_size;
unsigned int *d_vertex_list_size;

unsigned long long int *d_vertex_split;						//L, Input to the split function

// Hierarchy output
int cur_hierarchy_size; 									// Size current hierarchy

enum timing_mode {NO_TIME, TIME_COMPLETE, TIME_PARTS};
enum timing_mode TIMING_MODE;
std::vector<int> timings;

////////////////////////////////////////////////
// Debugging helper functions
////////////////////////////////////////////////
void printIntArr(int* d_data, int n_elements) {
	int* h_data = (int *)malloc(sizeof(int)*n_elements);
	hipMemcpy(h_data, d_data, sizeof(int) * n_elements, hipMemcpyDeviceToHost);
	for (int i = 0; i < n_elements; i++) {
		printf("%d ",h_data[i]);
	}
	printf("\n");
	free(h_data);
}

void printXArr(int* d_data, int n_elements) {
	int* h_data = (int *)malloc(sizeof(int)*n_elements);
	hipMemcpy(h_data, d_data, sizeof(int) * n_elements, hipMemcpyDeviceToHost);
	for (int i = 0; i < n_elements; i++) {
		int mask = pow(2.0,MOVEBITS)-1;
		int vertex = h_data[i]&mask;
		int weight = h_data[i]>>MOVEBITS;
		printf("%d|%d ",weight, vertex);
	}
	printf("\n");
	free(h_data);
}

void printUVWArr(unsigned long long int *d_data, int n_elements) {
	unsigned long long int* h_data = (unsigned long long int *)malloc(sizeof(unsigned long long int)*n_elements);
	hipMemcpy(h_data, d_data, sizeof(unsigned long long int) * n_elements, hipMemcpyDeviceToHost);
	for (int i = 0; i < n_elements; i++) {
		unsigned long long int UVW = h_data[i];
		unsigned long long int mask = pow(2.0,64-(NO_OF_BITS_MOVED_FOR_VERTEX_IDS+NO_OF_BITS_MOVED_FOR_VERTEX_IDS))-1;
		unsigned long long int w  = (int) UVW&mask;
		unsigned long long int test = UVW>>(64-(NO_OF_BITS_MOVED_FOR_VERTEX_IDS+NO_OF_BITS_MOVED_FOR_VERTEX_IDS));
		unsigned long long int mask2 = pow(2.0,NO_OF_BITS_MOVED_FOR_VERTEX_IDS)-1;
		unsigned long long int v = test&mask2;
		unsigned long long int u = test>>NO_OF_BITS_MOVED_FOR_VERTEX_IDS;
		printf("%llu|%llu|%llu ",u, v, w);
	}
	printf("\n");
	free(h_data);
}

void printUIntArr(unsigned int* d_data, int n_elements) {
	unsigned int* h_data = (unsigned int *)malloc(sizeof(unsigned int)*n_elements);
	hipMemcpy(h_data, d_data, sizeof(unsigned int) * n_elements, hipMemcpyDeviceToHost);
	for (int i = 0; i < n_elements; i++) {
		printf("%u ",h_data[i]);
	}
	printf("\n");
	free(h_data);
}

void printULongArr(long* d_data, int n_elements) {
	unsigned long* h_data = (unsigned long *)malloc(sizeof(unsigned long)*n_elements);
	hipMemcpy(h_data, d_data, sizeof(unsigned long) * n_elements, hipMemcpyDeviceToHost);
	for (int i = 0; i < n_elements; i++) {
		printf("%lu ",h_data[i]);
	}
	printf("\n");
	free(h_data);
}

void printLongArr(long* d_data, int n_elements) {
	long* h_data = (long *)malloc(sizeof(long)*n_elements);
	hipMemcpy(h_data, d_data, sizeof(long) * n_elements, hipMemcpyDeviceToHost);
	for (int i = 0; i < n_elements; i++) {
		printf("%ld ",h_data[i]);
	}
	printf("\n");
	free(h_data);
}

void printInt(int *d_val) {
	int h_val;
	hipMemcpy( &h_val, d_val, sizeof(int), hipMemcpyDeviceToHost);
	printf("%d", h_val);
}

void printUInt(unsigned int *d_val) {
	unsigned int h_val;
	hipMemcpy( &h_val, d_val, sizeof(unsigned int), hipMemcpyDeviceToHost);
	printf("%u", h_val);
}

////////////////////////////////////////////////
// Helper functions to set the grid sizes
////////////////////////////////////////////////
void SetGridThreadLen(int number, int *num_of_blocks, int *num_of_threads_per_block)
{
	*num_of_blocks = 1;
	*num_of_threads_per_block = number;

	//Make execution Parameters according to the number of nodes
	//Distribute threads across multiple Blocks if necessary
	if(number>MAX_THREADS_PER_BLOCK)
	{
		*num_of_blocks = (int)ceil(number/(double)MAX_THREADS_PER_BLOCK); 
		*num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
	}
}

void SetImageGridThreadLen(int no_of_rows, int no_of_cols, int no_of_vertices, dim3* encode_threads, dim3* encode_blocks)
{    
    if (no_of_vertices < 1024) {
        encode_threads->x = no_of_rows;
        encode_threads->y = no_of_cols;
        encode_blocks->x = 1;
        encode_blocks->y = 1;
    } else {
        encode_threads->x = 32;
        encode_threads->y = 32;
        encode_blocks->x = no_of_rows / 32 + 1;
        encode_blocks->y = no_of_cols / 32 + 1;
    }
}

////////////////////////////////////////////////
// Allocate and Free segmentation Arrays
////////////////////////////////////////////////
void Init()
{

	//Allocate graph device memory
	hipMalloc( (void**) &d_edge, sizeof(unsigned int)*no_of_edges_orig);
	hipMalloc( (void**) &d_vertex, sizeof(unsigned int)*no_of_vertices_orig);
	hipMalloc( (void**) &d_weight, sizeof(unsigned int)*no_of_edges_orig);

	//Allocate memory for other arrays
	hipMalloc( (void**) &d_segmented_min_scan_input, sizeof(unsigned long long int)*no_of_edges_orig);
	hipMalloc( (void**) &d_segmented_min_scan_output, sizeof(unsigned long long int)*no_of_edges_orig);
	hipMalloc( (void**) &d_edge_flag, sizeof(unsigned int)*no_of_edges_orig);
	hipMalloc( (void**) &d_edge_flag_thrust, sizeof(unsigned int)*no_of_edges_orig);
	hipMalloc( (void**) &d_pick_array, sizeof(unsigned int)*no_of_edges_orig);
	hipMalloc( (void**) &d_successor,sizeof(unsigned int)*no_of_vertices_orig);
	hipMalloc( (void**) &d_successor_copy,sizeof(unsigned int)*no_of_vertices_orig);
	
	//Clear Output MST array
	hipMalloc( (void**) &d_succchange, sizeof(bool));
	hipMalloc( (void**) &d_vertex_split, sizeof(unsigned long long int)*no_of_vertices_orig);
	hipMalloc( (void**) &d_vertex_flag, sizeof(unsigned int)*no_of_vertices_orig);
	hipMalloc( (void**) &d_new_supervertexIDs, sizeof(unsigned int)*no_of_vertices_orig);
	hipMalloc( (void**) &d_old_uIDs, sizeof(unsigned int)*no_of_edges_orig);
	hipMalloc( (void**) &d_appended_uvw, sizeof(unsigned long long int)*no_of_edges_orig);
	hipMalloc( (void**) &d_size, sizeof(unsigned int));
	hipMalloc( (void**) &d_edge_mapping_copy, sizeof(unsigned int)*no_of_edges_orig); 

	hipMalloc( (void**) &d_edge_list_size, sizeof(unsigned int));
	hipMalloc( (void**) &d_vertex_list_size, sizeof(unsigned int));
	
}

void FreeMem()
{
	hipFree(d_edge);
	hipFree(d_vertex);
	hipFree(d_weight);
	hipFree(d_segmented_min_scan_input);
	hipFree(d_segmented_min_scan_output);
	hipFree(d_edge_flag);
	hipFree(d_edge_flag_thrust);
	hipFree(d_pick_array);
	hipFree(d_successor);
	hipFree(d_successor_copy);
	hipFree(d_succchange);
	hipFree(d_vertex_split);
	hipFree(d_vertex_flag);
	hipFree(d_new_supervertexIDs);
	hipFree(d_old_uIDs);
	hipFree(d_size);
	hipFree(d_edge_mapping_copy);
	hipFree(d_edge_list_size);
	hipFree(d_vertex_list_size);
	hipFree(d_appended_uvw);
}

////////////////////////////////////////////////
// Create graph in compressed adjacency list
////////////////////////////////////////////////
void createGraph(Mat image) {
	std::chrono::high_resolution_clock::time_point start, end;

   	GpuMat dev_image, d_blurred;; 	 // Released automatically in destructor
   	cv::Ptr<cv::cuda::Filter> filter;
   	

	if (TIMING_MODE == TIME_PARTS) { // Start gaussian filter timer
		start = std::chrono::high_resolution_clock::now();
	}


	// Apply gaussian filter
    dev_image.upload(image);
    filter = cv::cuda::createGaussianFilter(CV_8UC3, CV_8UC3, cv::Size(5, 5), 1.0);
    filter->apply(dev_image, d_blurred);
	

	if (TIMING_MODE == TIME_PARTS) { // End gaussian filter timer
		hipDeviceSynchronize();
		end = std::chrono::high_resolution_clock::now();
		int time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
		timings.push_back(time);
	}


	if (TIMING_MODE == TIME_PARTS) { // Start graph creation timer
		start = std::chrono::high_resolution_clock::now();
	}

	// Allocate GPU segmentation memory
	Init();


	// Create graphs. Kernels executed in different streams for concurrency
	dim3 encode_threads;
	dim3 encode_blocks;
	SetImageGridThreadLen(no_of_rows, no_of_cols, no_of_vertices_orig, &encode_threads, &encode_blocks);

    int num_of_blocks, num_of_threads_per_block;

	SetGridThreadLen(no_of_cols, &num_of_blocks, &num_of_threads_per_block);
	dim3 grid_row(num_of_blocks, 1, 1);
	dim3 threads_row(num_of_threads_per_block, 1, 1);

	SetGridThreadLen(no_of_rows, &num_of_blocks, &num_of_threads_per_block);
	dim3 grid_col(num_of_blocks, 1, 1);
	dim3 threads_col(num_of_threads_per_block, 1, 1);

    dim3 grid_corner(1, 1, 1);
	dim3 threads_corner(4, 1, 1);

    size_t pitch = d_blurred.step;

    // Create inner graph
    createInnerGraphKernel<<< encode_blocks, encode_threads, 0>>>((unsigned char*) d_blurred.cudaPtr(), d_vertex, d_edge, d_weight, no_of_rows, no_of_cols, pitch);

    // Create outer graph
   	createFirstRowGraphKernel<<< grid_row, threads_row, 1>>>((unsigned char*) d_blurred.cudaPtr(), d_vertex, d_edge, d_weight, no_of_rows, no_of_cols, pitch);
   	createLastRowGraphKernel<<< grid_row, threads_row, 2>>>((unsigned char*) d_blurred.cudaPtr(), d_vertex, d_edge, d_weight, no_of_rows, no_of_cols, pitch);

   	createFirstColumnGraphKernel<<< grid_col, threads_col, 3>>>((unsigned char*) d_blurred.cudaPtr(), d_vertex, d_edge, d_weight, no_of_rows, no_of_cols, pitch);
   	createLastColumnGraphKernel<<< grid_col, threads_col, 4>>>((unsigned char*) d_blurred.cudaPtr(), d_vertex, d_edge, d_weight, no_of_rows, no_of_cols, pitch);

    // Create corners
	createCornerGraphKernel<<< grid_corner, threads_corner, 5>>>((unsigned char*) d_blurred.cudaPtr(), d_vertex, d_edge, d_weight, no_of_rows, no_of_cols, pitch);
	
	hipDeviceSynchronize(); // Needed to synchronise streams!

	if (TIMING_MODE == TIME_PARTS) {
		end = std::chrono::high_resolution_clock::now();
		int time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
		timings.push_back(time);
	}


	fprintf(stderr, "Image read successfully into graph with %d vertices and %d edges\n", no_of_vertices, no_of_edges);
}


////////////////////////////////////////////////
// Perform Our Recursive MST Algorithm
////////////////////////////////////////////////
void HPGMST()
{
	//Make both CUDA grids needed for execution, no_of_vertices and no_of_edges length sizes
	int num_of_blocks, num_of_threads_per_block;

	//Grid and block sizes so each edge has one thread (fit as much threads as possible in one block)
	SetGridThreadLen(no_of_edges, &num_of_blocks, &num_of_threads_per_block);
	dim3 grid_edgelen(num_of_blocks, 1, 1);
	dim3 threads_edgelen(num_of_threads_per_block, 1, 1);

	// Grid and block sizes so each vertex has one thread (fit as much threads as possible in one block)
	SetGridThreadLen(no_of_vertices, &num_of_blocks, &num_of_threads_per_block);
	dim3 grid_vertexlen(num_of_blocks, 1, 1);
	dim3 threads_vertexlen(num_of_threads_per_block, 1, 1);

	/*
	 * A. Find minimum weighted edge
	 */

	// 1. Append weight w and outgoing vertex v per edge into a single array, X.
    // 12 bit for weight, 26 bits for ID.
	//Append in Parallel on the Device itself, call the append kernel
	AppendKernel_1<<< grid_edgelen, threads_edgelen, 0>>>(d_segmented_min_scan_input, d_weight, d_edge, no_of_edges);

	// d_edge_flag = F
	//Create the Flag needed for segmented min scan operation, similar operation will also be used at other places
	ClearArray<<< grid_edgelen, threads_edgelen, 0>>>( d_edge_flag, no_of_edges );


	// 2. Divide the edge-list, E, into segments with 1 indicating the start of each segment and 0 otherwise, store this in flag array F.
	// Mark the segments for the segmented min scan
	MakeFlag_3<<< grid_vertexlen, threads_vertexlen, 0>>>( d_edge_flag, d_vertex, no_of_vertices);


	// 3. Perform segmented min scan on X with F indicating segments to find minimum outgoing edge-index per vertex. Min can be found at end of each segment after scan // DONE: change to thrust
	// Prepare key vector for thrust
	thrust::inclusive_scan(thrust::device, d_edge_flag, d_edge_flag + no_of_edges, d_edge_flag_thrust);

	// Min inclusive segmented scan on ints from start to end.
	thrust::equal_to<unsigned int> binaryPred;
	thrust::minimum<unsigned long long int> binaryOp;
	thrust::inclusive_scan_by_key(thrust::device, d_edge_flag_thrust, d_edge_flag_thrust + no_of_edges, d_segmented_min_scan_input, d_segmented_min_scan_output, binaryPred, binaryOp);


	/*
	 * B. Finding and removing cycles
	 */

	// 4. Find the successor of each vertex and add to successor array, S.
	MakeSucessorArray<<< grid_vertexlen, threads_vertexlen, 0>>>(d_successor, d_vertex, d_segmented_min_scan_output, no_of_vertices, no_of_edges);


	// 5. Remove cycle making edges using S, and identify representatives vertices.
	RemoveCycles<<< grid_vertexlen, threads_vertexlen, 0>>>(d_successor,no_of_vertices);


	/*
	 * Can possibly be moved in future once remove pick array stuff
	 */
	//Scan the flag to get u at every edge, use the u to index d_vertex to get the last entry in each segment
	//U at every edge will also be useful later in the algorithm.

	// Set F[0] = 0. F is the same as previous F but first element is 0 instead of 1
	ClearArray<<< grid_edgelen, threads_edgelen, 0>>>( d_edge_flag, no_of_edges );
	MakeFlagForUIds<<< grid_vertexlen, threads_vertexlen, 0>>>(d_edge_flag, d_vertex,no_of_vertices); 

	// 10.2 Create vector indicating source vertex u for each edge // DONE: change to thrust
	thrust::inclusive_scan(thrust::device, d_edge_flag, d_edge_flag + no_of_edges, d_old_uIDs);


	/*
	 * C. Merging vertices and assigning IDs to supervertices
	 */


	// 7. Propagate Representative Vertex IDs to all vertices iteratively using pointer Doubling until no change occures in Successor Array
	bool succchange;
	do
	{
		succchange=false; //if no thread changes this value, the loop stops
		hipMemcpy( d_succchange, &succchange, sizeof(bool), hipMemcpyHostToDevice);
		//Reusing Vertex Flag
		SuccToCopy<<< grid_vertexlen, threads_vertexlen, 0>>>(d_successor, d_successor_copy, no_of_vertices); // for conflicts
		PropagateRepresentativeID<<< grid_vertexlen, threads_vertexlen, 0>>>(d_successor, d_successor_copy, d_succchange,no_of_vertices);
		CopyToSucc<<< grid_vertexlen, threads_vertexlen, 0>>>(d_successor, d_successor_copy, no_of_vertices); // for conflicts

		hipMemcpy( &succchange, d_succchange, sizeof(bool), hipMemcpyDeviceToHost);
	}
	while(succchange);


	// 8. Append successor array’s entries with its index to form a list, L. Representative left, vertex id right, 64 bit.
	//    Append Vertex Ids with SuperVertexIDs
	AppendVertexIDsForSplit<<< grid_vertexlen, threads_vertexlen, 0>>>(d_vertex_split, d_successor,no_of_vertices);


	//9. Split L, create flag over split output and scan the flag to find new ids per vertex, store new ids in C.
    // 9.1 Split L using representative as key. In parallel using a split of O(V) with log(V) bit key size.
    //     split based on supervertex IDs using 64 bit version of split
	thrust::sort(thrust::device, d_vertex_split, d_vertex_split + no_of_vertices);


	// 9.2 Create flag for assigning new vertex IDs based on difference in supervertex IDs
	//     first element not flagged so that can use simple sum for scan
	ClearArray<<< grid_vertexlen, threads_vertexlen, 0>>>( d_vertex_flag, no_of_vertices);
	MakeFlagForScan<<< grid_vertexlen, threads_vertexlen, 0>>>(d_vertex_flag, d_vertex_split, no_of_vertices);

	// 9.3 Scan flag to assign new IDs to supervertices, Using a scan on O(V) elements // DONE: change to thrust
	thrust::inclusive_scan(thrust::device, d_vertex_flag, d_vertex_flag + no_of_vertices, d_new_supervertexIDs);


	/*
	 * D. Removing self edges
	 */

	// 10.1 Create mapping from each original vertex ID to its new supervertex ID so we can lookup supervertex IDs directly
	MakeSuperVertexIDPerVertex<<< grid_vertexlen, threads_vertexlen, 0>>>(d_new_supervertexIDs, d_vertex_split, d_vertex_flag, no_of_vertices);
	CopySuperVertexIDPerVertex<<< grid_vertexlen, threads_vertexlen, 0>>>(d_new_supervertexIDs, d_vertex_flag, no_of_vertices); // for concurrent access problems
	
	//Remove Self Edges from the edge-list
	// 11. Remove edge from edge-list if u, v have same supervertex id (remove self edges)
	CopyEdgeArray<<< grid_edgelen, threads_edgelen, 0>>>(d_edge,d_edge_mapping_copy, no_of_edges); // for conflicts
	RemoveSelfEdges<<< grid_edgelen, threads_edgelen, 0>>>(d_edge, d_old_uIDs, d_new_supervertexIDs, d_edge_mapping_copy, no_of_edges);
	CopyEdgeArrayBack<<< grid_edgelen, threads_edgelen, 0>>>(d_edge,d_edge_mapping_copy, no_of_edges); // for conflicts

	/*
	 * D. Removing duplicate edges. This is not mandatory, however, reduces the edge-list size significantly. You may choose to use it once in the initial 
	 *    iterations of the algorithm, later edge-list size is small anyways so not much is gained by doing this in later iterations
	 */


	// 12. Remove the largest duplicate edges using split over new u,v and w.
	// 12.1 Append supervertex ids of u and v along with weight w into single 64 bit array (u 24 bit, v 24 bit, w 16 bit)
	AppendForDuplicateEdgeRemoval<<< grid_edgelen, threads_edgelen, 0>>>(d_appended_uvw, d_edge, d_old_uIDs, d_weight,d_new_supervertexIDs, no_of_edges);

	//12.2 Split the array using {u,v) as the key. Pick First distinct (u,v) entry as the edge, nullify others
	//     You may also replace the split with sort, but we could not find a 64-bit sort.
	thrust::sort(thrust::device, d_appended_uvw, d_appended_uvw + no_of_edges);
	
	//Pick the first distinct (u,v) combination, mark these edges and compact
	// 12.3 Create flag indicating smallest edges, 0 for larger duplicates
	ClearArray<<< grid_edgelen, threads_edgelen, 0>>>( d_edge_flag, no_of_edges ); // d_edge_flag = F3
	unsigned int dsize=no_of_edges; //just make sure
	hipMemcpy( d_size, &dsize, sizeof(unsigned int), hipMemcpyHostToDevice);
	MarkEdgesUV<<< grid_edgelen, threads_edgelen, 0>>>(d_edge_flag, d_appended_uvw, d_size, no_of_edges);


	// 13. Compact and create new edge and weight list
	// 13.1 Scan the flag array to know where to write the value in new edge and weight lists // DONE: change to thrust
	thrust::inclusive_scan(thrust::device, d_edge_flag, d_edge_flag + no_of_edges, d_old_uIDs);

	// Make sure new locations start from 0 instead of 1.
	thrust::transform(thrust::device,
				  d_old_uIDs,
                  d_old_uIDs + no_of_edges,
                  thrust::make_constant_iterator(1),
                  d_old_uIDs,
                  thrust::minus<unsigned int>());


	// Do some cleanup / clearing
	ClearEdgeStuff<<< grid_edgelen, threads_edgelen, 0>>>((unsigned int*)d_edge, (unsigned int*)d_weight, d_edge_mapping_copy, (unsigned int*)d_pick_array, no_of_edges);
	unsigned int negative=0;
	hipMemcpy( d_edge_list_size, &negative, sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy( d_vertex_list_size, &negative, sizeof(unsigned int), hipMemcpyHostToDevice);
	
	//Compact the edge and weight lists
	unsigned int validsize=0;
	hipMemcpy( &validsize, d_size, sizeof(unsigned int), hipMemcpyDeviceToHost);

	//Make a new grid for valid entries in the d_edge_flag array
	SetGridThreadLen(validsize, &num_of_blocks, &num_of_threads_per_block);
	dim3 grid_validsizelen(num_of_blocks, 1, 1);
	dim3 threads_validsizelen(num_of_threads_per_block, 1, 1);

	// 13.2 Compact and create new edge and weight list
	//      Reusing d_pick_array for storing the u ids
	CompactEdgeList<<< grid_validsizelen, threads_validsizelen, 0>>>(d_edge, d_weight, d_old_uIDs, d_edge_flag, d_appended_uvw, d_pick_array, d_size, d_edge_list_size, d_vertex_list_size);

	// 14. Build the vertex list from the newly formed edge list
	ClearArray<<< grid_edgelen, threads_edgelen, 0>>>( d_edge_flag, no_of_edges);
	ClearArray<<< grid_vertexlen, threads_vertexlen, 0>>>((unsigned int*)d_vertex, no_of_vertices);

	//14.1 Create flag based on difference in u on the new edge list (based on diffference of u ids)
	MakeFlagForVertexList<<< grid_edgelen, threads_edgelen, 0>>>(d_pick_array, d_edge_flag, no_of_edges); // d_edge_flag = F4

	// 14.2 Build the vertex list from the newly formed edge list
	MakeVertexList<<< grid_edgelen, threads_edgelen, 0>>>(d_vertex, d_pick_array, d_edge_flag, no_of_edges);
	
	cur_hierarchy_size = no_of_vertices;
	hipMemcpy( &no_of_edges, d_edge_list_size, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy( &no_of_vertices, d_vertex_list_size, sizeof(unsigned int), hipMemcpyDeviceToHost);

}


void writeComponents(std::vector<unsigned int*>& d_hierarchy_levels, std::vector<int>& hierarchy_level_sizes, std::string outFile) {
	// Extract filepath without extension
	size_t lastindex = outFile.find_last_of("."); 
	std::string rawOutName = outFile.substr(0, lastindex);

	// Generate random colors for segments
	char *component_colours = (char *) malloc(no_of_vertices_orig * CHANNEL_SIZE * sizeof(char));


	// Generate uniform [0, 1] float
	hiprandGenerator_t gen;
	char* d_component_colours;
	float *d_component_colours_float;
	hipMalloc( (void**) &d_component_colours_float, no_of_vertices_orig * CHANNEL_SIZE * sizeof(float));
	hipMalloc( (void**) &d_component_colours, no_of_vertices_orig * CHANNEL_SIZE * sizeof(char));

	// Generate random floats
	hiprandCreateGenerator(&gen , HIPRAND_RNG_PSEUDO_MTGP32); // Create a Mersenne Twister pseudorandom number generator
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL); // Set seed
	hiprandGenerateUniform(gen, d_component_colours_float, no_of_vertices_orig * CHANNEL_SIZE); // Generate n floats on device

	// Convert floats to RGB char
	int num_of_blocks, num_of_threads_per_block;

	SetGridThreadLen(no_of_vertices_orig * CHANNEL_SIZE, &num_of_blocks, &num_of_threads_per_block);
	dim3 grid_rgb(num_of_blocks, 1, 1);
	dim3 threads_rgb(num_of_threads_per_block, 1, 1);

	RandFloatToRandRGB<<< grid_rgb, threads_rgb, 0>>>(d_component_colours, d_component_colours_float, no_of_vertices_orig * CHANNEL_SIZE);
	hipFree(d_component_colours_float);


	// Create hierarchy
	unsigned int* d_prev_level_component;
	hipMalloc((void**) &d_prev_level_component, sizeof(unsigned int)*no_of_vertices_orig);

	dim3 threads_pixels;
    dim3 grid_pixels;
	SetImageGridThreadLen(no_of_rows, no_of_cols, no_of_vertices_orig, &threads_pixels, &grid_pixels);

    InitPrevLevelComponents<<<grid_pixels, threads_pixels, 0>>>(d_prev_level_component, no_of_rows, no_of_cols);

    char* d_output_image;
	hipMalloc( (void**) &d_output_image, no_of_rows*no_of_cols*CHANNEL_SIZE*sizeof(char));
    char *output = (char*) malloc(no_of_rows*no_of_cols*CHANNEL_SIZE*sizeof(char));

    for (int l = 0; l < d_hierarchy_levels.size(); l++) {
		int level_size = hierarchy_level_sizes[l];
		unsigned int* d_level = d_hierarchy_levels[l];

		CreateLevelOutput<<< grid_pixels, threads_pixels, 0>>>(d_output_image, d_component_colours, d_level, d_prev_level_component, no_of_rows, no_of_cols);
	    hipMemcpy(output, d_output_image, no_of_rows*no_of_cols*CHANNEL_SIZE*sizeof(char), hipMemcpyDeviceToHost);

		cv::Mat output_img = cv::Mat(no_of_rows, no_of_cols, CV_8UC3, output);
		std::string outfilename = rawOutName + std::string("_")  + std::to_string(l) + std::string(".png");
		std::string outmessage = std::string("Writing ") + outfilename.c_str() + std::string("\n");

		fprintf(stderr, "%s", outmessage.c_str());
		imwrite(outfilename, output_img);
	}


	// Free memory
	hipFree(d_component_colours);
	hipFree(d_prev_level_component);
	hipFree(d_output_image);
	free(output);
}

void setGraphParams(unsigned int rows, unsigned int cols) {
	no_of_rows = rows;
    no_of_cols = cols;
	no_of_vertices = no_of_rows * no_of_cols;
	no_of_vertices_orig = no_of_vertices;
	no_of_edges = 8 + 6 * (no_of_cols - 2) + 6 * (no_of_rows - 2) + 4 * (no_of_cols - 2) * (no_of_rows - 2);
	no_of_edges_orig = no_of_edges;
}

void clearHierarchy(std::vector<unsigned int*>& d_hierarchy_levels, std::vector<int>& hierarchy_level_sizes) {
	for (int l = 0; l < d_hierarchy_levels.size(); l++) {
			hipFree(d_hierarchy_levels[l]);
		}
        d_hierarchy_levels.clear();
        hierarchy_level_sizes.clear();
}

void segment(Mat image, std::string outFile, bool output) {
	std::chrono::high_resolution_clock::time_point start, end;

	if (TIMING_MODE == TIME_COMPLETE) { // Start whole execution timer
		start = std::chrono::high_resolution_clock::now();
	}


	// Reset num vertices in edges in case of multiple iterations
	no_of_edges = no_of_edges_orig;
	no_of_vertices = no_of_vertices_orig;

	std::vector<unsigned int*> d_hierarchy_levels;	// Vector containing pointers to all hierarchy levels (don't dereference on CPU, device pointers)
	std::vector<int> hierarchy_level_sizes;			// Size of each hierarchy level

	// Graph creation
	createGraph(image);


	if (TIMING_MODE == TIME_PARTS) { // Start segmentation timer
		start = std::chrono::high_resolution_clock::now();
	}
	
	// Segmentation
	do
	{
	    HPGMST();

	    d_hierarchy_levels.push_back(d_new_supervertexIDs);
	    hierarchy_level_sizes.push_back(cur_hierarchy_size);
	    hipMalloc( (void**) &d_new_supervertexIDs, sizeof(unsigned int)*cur_hierarchy_size);

	    fprintf(stderr, "Vertices: %d\n", no_of_vertices);
	}
	while(no_of_vertices>1);

	if (TIMING_MODE == TIME_PARTS) { // End segmentation timer
		hipDeviceSynchronize();
		end = std::chrono::high_resolution_clock::now();
		int time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
		timings.push_back(time);
	}

	if (TIMING_MODE == TIME_COMPLETE) { // End whole execution timer
		hipDeviceSynchronize();
		end = std::chrono::high_resolution_clock::now();
		int time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
		timings.push_back(time);
	}

	// Free GPU segmentation memory
	FreeMem();

	if (output) {
		// Write segmentation hierarchy
		writeComponents(d_hierarchy_levels, hierarchy_level_sizes, outFile);
	}

	clearHierarchy(d_hierarchy_levels, hierarchy_level_sizes);
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
void printUsage() {
    puts("Usage: ./felz -i [input image path] -o [output image path]");
    puts("Options:");
    puts("\t-i: Path to input file (default: data/beach.png)");
    puts("\t-o: Path to output file (default: segmented.png)");
    puts("Benchmarking options");
    puts("\t-w: Number of iterations to perform during warmup");
    puts("\t-b: Number of iterations to perform during benchmarking");
    puts("\t-t: Timing mode: complete / parts (default complete)");
    exit(1);
}

void printCSVHeader() {
	if (TIMING_MODE == TIME_COMPLETE) {
		 printf("total\n"); // Excluding output: gaussian + graph creation + segmentation
	} else {
		printf("gaussian, graph creation, segmentation\n");
	}
}

void printCSVLine() {
	if (timings.size() > 0) {
		printf("%d", timings[0]);
		for (int i = 1; i < timings.size(); i++) {
			printf(",%d", timings[i]);
		}
		printf("\n");
		timings.clear();
	}
	
}

const Options handleParams(int argc, char **argv) {
    Options options = Options();
    TIMING_MODE = TIME_COMPLETE;
    for(;;)
    {
        switch(getopt(argc, argv, "hi:o:w:b:t:"))
        {
            case 'i': {
                options.inFile = std::string(optarg);
                continue;
            }
            case 'o': {
                options.outFile = std::string(optarg);
                continue;
            }
            case 'w': {
                options.warmupIterations = atoi(optarg);
                continue;
            }
            case 'b': {
                options.benchmarkIterations = atoi(optarg);
                continue;
            }
             case 'p': {
                TIMING_MODE = TIME_PARTS;
                continue;
            }
            case '?':
            case 'h':
            default : {
                printUsage();
                break;
            }

            case -1:  {
                break;
            }
        }
        break;
    }
    if (options.inFile == "empty" || options.outFile == "empty") {
    	puts("Provide an input and output image!");
		printUsage();
    }

    return options;
}

int main(int argc, char **argv)
{
    const Options options = handleParams(argc, argv);

    // Read image
    Mat image = imread(options.inFile, IMREAD_COLOR);
    fprintf(stderr, "Size of image obtained is: Rows: %d, Columns: %d, Pixels: %d\n", image.rows, image.cols, image.rows * image.cols);
   	setGraphParams(image.rows, image.cols);

   	printCSVHeader();

	// Warm up
    for (int i = 0; i < options.warmupIterations; i++) {
    	segment(image, options.outFile, false);
    }

    // Benchmark
    timings.clear();
    for (int i = 0; i < options.benchmarkIterations; i++) {
        segment(image, options.outFile, i == options.benchmarkIterations-1);
        printCSVLine();
    }

    return 0;
}

